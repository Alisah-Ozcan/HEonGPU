#include "hip/hip_runtime.h"
// Copyright 2024-2025 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "heongpu.cuh"
#include <gtest/gtest.h>

template <typename T>
bool fix_point_equal(T input1, T input2, T epsilon = static_cast<T>(1e-4))
{
    return std::fabs(input1 - input2) < epsilon;
}

template <typename T>
bool fix_point_array_check(const std::vector<T>& array1,
                           const std::vector<T>& array2,
                           T epsilon = static_cast<T>(1e-4))
{
    if (array1.size() != array2.size())
    {
        return false;
    }

    for (size_t i = 0; i < array1.size(); ++i)
    {
        if (!fix_point_equal(array1[i], array2[i], epsilon))
        {
            return false;
        }
    }

    return true;
}

TEST(HEonGPU, CKKS_Ciphertext_Rotation_Keyswitching_Method_II_Part_I)
{
    hipSetDevice(0);
    {
        size_t poly_modulus_degree = 4096;
        heongpu::HEContext<heongpu::Scheme::CKKS> context(
            heongpu::keyswitching_type::KEYSWITCHING_METHOD_II,
            heongpu::sec_level_type::none);
        context.set_poly_modulus_degree(poly_modulus_degree);
        context.set_coeff_modulus_bit_sizes({40, 30, 30}, {40, 40});
        context.generate();

        heongpu::HEKeyGenerator<heongpu::Scheme::CKKS> keygen(context);
        heongpu::Secretkey<heongpu::Scheme::CKKS> secret_key(context);
        keygen.generate_secret_key(secret_key);

        heongpu::Publickey<heongpu::Scheme::CKKS> public_key(context);
        keygen.generate_public_key(public_key, secret_key);

        heongpu::HEEncoder<heongpu::Scheme::CKKS> encoder(context);
        heongpu::HEEncryptor<heongpu::Scheme::CKKS> encryptor(context,
                                                              public_key);
        heongpu::HEDecryptor<heongpu::Scheme::CKKS> decryptor(context,
                                                              secret_key);
        heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS> operators(context,
                                                                       encoder);

        std::vector<int> shift_key_index = {-5, -2, 31};
        heongpu::Galoiskey<heongpu::Scheme::CKKS> galois_key(context,
                                                             shift_key_index);
        keygen.generate_galois_key(galois_key, secret_key);

        for (size_t j = 0; j < shift_key_index.size(); j++)
        {
            std::random_device rd;
            std::mt19937 gen(rd());
            std::uniform_real_distribution<> dis(0.0, 1.0);
            const int row_size = poly_modulus_degree / 2;
            std::vector<double> message1(row_size, 0);
            for (int i = 0; i < row_size; i++)
            {
                message1[i] = dis(gen);
            }

            int shift_count = shift_key_index[j];
            std::vector<double> message_rotation_result(row_size, 0);
            for (int i = 0; i < row_size; i++)
            {
                int index = ((i + shift_count) < 0)
                                ? ((i + shift_count) + row_size)
                                : ((i + shift_count) % row_size);
                message_rotation_result[i] = message1[index];
            }

            double scale = pow(2.0, 30);
            heongpu::Plaintext<heongpu::Scheme::CKKS> P1(context);
            encoder.encode(P1, message1, scale);

            heongpu::Ciphertext<heongpu::Scheme::CKKS> C1(context);
            encryptor.encrypt(C1, P1);

            operators.rotate_rows(C1, C1, galois_key, shift_count);

            heongpu::Plaintext<heongpu::Scheme::CKKS> P3(context);
            decryptor.decrypt(P3, C1);

            std::vector<double> gpu_result;
            encoder.decode(gpu_result, P3);

            hipDeviceSynchronize();

            EXPECT_EQ(fix_point_array_check(message_rotation_result, gpu_result,
                                            static_cast<double>(1e-1)),
                      true);

            // Leveled Test
            operators.mod_drop_inplace(C1);
            heongpu::Plaintext<heongpu::Scheme::CKKS> P4(context);
            decryptor.decrypt(P4, C1);

            std::vector<double> gpu_result2;
            encoder.decode(gpu_result2, P4);

            hipDeviceSynchronize();

            EXPECT_EQ(fix_point_array_check(message_rotation_result,
                                            gpu_result2,
                                            static_cast<double>(1e-1)),
                      true);
        }
    }

    hipDeviceSynchronize();

    {
        size_t poly_modulus_degree = 8192;
        heongpu::HEContext<heongpu::Scheme::CKKS> context(
            heongpu::keyswitching_type::KEYSWITCHING_METHOD_II,
            heongpu::sec_level_type::none);
        context.set_poly_modulus_degree(poly_modulus_degree);
        context.set_coeff_modulus_bit_sizes({40, 30, 30, 30, 30}, {40, 40});
        context.generate();

        heongpu::HEKeyGenerator<heongpu::Scheme::CKKS> keygen(context);
        heongpu::Secretkey<heongpu::Scheme::CKKS> secret_key(context);
        keygen.generate_secret_key(secret_key);

        heongpu::Publickey<heongpu::Scheme::CKKS> public_key(context);
        keygen.generate_public_key(public_key, secret_key);

        heongpu::HEEncoder<heongpu::Scheme::CKKS> encoder(context);
        heongpu::HEEncryptor<heongpu::Scheme::CKKS> encryptor(context,
                                                              public_key);
        heongpu::HEDecryptor<heongpu::Scheme::CKKS> decryptor(context,
                                                              secret_key);
        heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS> operators(context,
                                                                       encoder);

        std::vector<int> shift_key_index = {-5, -2, 31};
        heongpu::Galoiskey<heongpu::Scheme::CKKS> galois_key(context,
                                                             shift_key_index);
        keygen.generate_galois_key(galois_key, secret_key);

        for (size_t j = 0; j < shift_key_index.size(); j++)
        {
            std::random_device rd;
            std::mt19937 gen(rd());
            std::uniform_real_distribution<> dis(0.0, 1.0);
            const int row_size = poly_modulus_degree / 2;
            std::vector<double> message1(row_size, 0);
            for (int i = 0; i < row_size; i++)
            {
                message1[i] = dis(gen);
            }

            int shift_count = shift_key_index[j];
            std::vector<double> message_rotation_result(row_size, 0);
            for (int i = 0; i < row_size; i++)
            {
                int index = ((i + shift_count) < 0)
                                ? ((i + shift_count) + row_size)
                                : ((i + shift_count) % row_size);
                message_rotation_result[i] = message1[index];
            }

            double scale = pow(2.0, 30);
            heongpu::Plaintext<heongpu::Scheme::CKKS> P1(context);
            encoder.encode(P1, message1, scale);

            heongpu::Ciphertext<heongpu::Scheme::CKKS> C1(context);
            encryptor.encrypt(C1, P1);

            operators.rotate_rows(C1, C1, galois_key, shift_count);

            heongpu::Plaintext<heongpu::Scheme::CKKS> P3(context);
            decryptor.decrypt(P3, C1);

            std::vector<double> gpu_result;
            encoder.decode(gpu_result, P3);

            hipDeviceSynchronize();

            EXPECT_EQ(fix_point_array_check(message_rotation_result, gpu_result,
                                            static_cast<double>(1e-1)),
                      true);

            // Leveled Test
            operators.mod_drop_inplace(C1);
            heongpu::Plaintext<heongpu::Scheme::CKKS> P4(context);
            decryptor.decrypt(P4, C1);

            std::vector<double> gpu_result2;
            encoder.decode(gpu_result2, P4);

            hipDeviceSynchronize();

            EXPECT_EQ(fix_point_array_check(message_rotation_result,
                                            gpu_result2,
                                            static_cast<double>(1e-1)),
                      true);
        }
    }

    hipDeviceSynchronize();

    {
        size_t poly_modulus_degree = 16384;
        heongpu::HEContext<heongpu::Scheme::CKKS> context(
            heongpu::keyswitching_type::KEYSWITCHING_METHOD_II,
            heongpu::sec_level_type::none);
        context.set_poly_modulus_degree(poly_modulus_degree);
        context.set_coeff_modulus_bit_sizes(
            {45, 35, 35, 35, 35, 35, 35, 35, 35, 35, 35, 35}, {45, 45});
        context.generate();

        heongpu::HEKeyGenerator<heongpu::Scheme::CKKS> keygen(context);
        heongpu::Secretkey<heongpu::Scheme::CKKS> secret_key(context);
        keygen.generate_secret_key(secret_key);

        heongpu::Publickey<heongpu::Scheme::CKKS> public_key(context);
        keygen.generate_public_key(public_key, secret_key);

        heongpu::HEEncoder<heongpu::Scheme::CKKS> encoder(context);
        heongpu::HEEncryptor<heongpu::Scheme::CKKS> encryptor(context,
                                                              public_key);
        heongpu::HEDecryptor<heongpu::Scheme::CKKS> decryptor(context,
                                                              secret_key);
        heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS> operators(context,
                                                                       encoder);

        std::vector<int> shift_key_index = {-5, -2, 31};
        heongpu::Galoiskey<heongpu::Scheme::CKKS> galois_key(context,
                                                             shift_key_index);
        keygen.generate_galois_key(galois_key, secret_key);

        for (size_t j = 0; j < shift_key_index.size(); j++)
        {
            std::random_device rd;
            std::mt19937 gen(rd());
            std::uniform_real_distribution<> dis(0.0, 1.0);
            const int row_size = poly_modulus_degree / 2;
            std::vector<double> message1(row_size, 0);
            for (int i = 0; i < row_size; i++)
            {
                message1[i] = dis(gen);
            }

            int shift_count = shift_key_index[j];
            std::vector<double> message_rotation_result(row_size, 0);
            for (int i = 0; i < row_size; i++)
            {
                int index = ((i + shift_count) < 0)
                                ? ((i + shift_count) + row_size)
                                : ((i + shift_count) % row_size);
                message_rotation_result[i] = message1[index];
            }

            double scale = pow(2.0, 30);
            heongpu::Plaintext<heongpu::Scheme::CKKS> P1(context);
            encoder.encode(P1, message1, scale);

            heongpu::Ciphertext<heongpu::Scheme::CKKS> C1(context);
            encryptor.encrypt(C1, P1);

            operators.rotate_rows(C1, C1, galois_key, shift_count);

            heongpu::Plaintext<heongpu::Scheme::CKKS> P3(context);
            decryptor.decrypt(P3, C1);

            std::vector<double> gpu_result;
            encoder.decode(gpu_result, P3);

            hipDeviceSynchronize();

            EXPECT_EQ(fix_point_array_check(message_rotation_result, gpu_result,
                                            static_cast<double>(1e-1)),
                      true);

            // Leveled Test
            operators.mod_drop_inplace(C1);
            heongpu::Plaintext<heongpu::Scheme::CKKS> P4(context);
            decryptor.decrypt(P4, C1);

            std::vector<double> gpu_result2;
            encoder.decode(gpu_result2, P4);

            hipDeviceSynchronize();

            EXPECT_EQ(fix_point_array_check(message_rotation_result,
                                            gpu_result2,
                                            static_cast<double>(1e-1)),
                      true);
        }
    }

    hipDeviceSynchronize();

    {
        size_t poly_modulus_degree = 32768;
        heongpu::HEContext<heongpu::Scheme::CKKS> context(
            heongpu::keyswitching_type::KEYSWITCHING_METHOD_II,
            heongpu::sec_level_type::none);
        context.set_poly_modulus_degree(poly_modulus_degree);
        context.set_coeff_modulus_bit_sizes({59, 40, 40, 40, 40, 40, 40, 40, 40,
                                             40, 40, 40, 40, 40, 40, 40, 40, 40,
                                             40},
                                            {59, 59});
        context.generate();

        heongpu::HEKeyGenerator<heongpu::Scheme::CKKS> keygen(context);
        heongpu::Secretkey<heongpu::Scheme::CKKS> secret_key(context);
        keygen.generate_secret_key(secret_key);

        heongpu::Publickey<heongpu::Scheme::CKKS> public_key(context);
        keygen.generate_public_key(public_key, secret_key);

        heongpu::HEEncoder<heongpu::Scheme::CKKS> encoder(context);
        heongpu::HEEncryptor<heongpu::Scheme::CKKS> encryptor(context,
                                                              public_key);
        heongpu::HEDecryptor<heongpu::Scheme::CKKS> decryptor(context,
                                                              secret_key);
        heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS> operators(context,
                                                                       encoder);

        std::vector<int> shift_key_index = {-5, -2, 31};
        heongpu::Galoiskey<heongpu::Scheme::CKKS> galois_key(context,
                                                             shift_key_index);
        keygen.generate_galois_key(galois_key, secret_key);

        for (size_t j = 0; j < shift_key_index.size(); j++)
        {
            std::random_device rd;
            std::mt19937 gen(rd());
            std::uniform_real_distribution<> dis(0.0, 1.0);
            const int row_size = poly_modulus_degree / 2;
            std::vector<double> message1(row_size, 0);
            for (int i = 0; i < row_size; i++)
            {
                message1[i] = dis(gen);
            }

            int shift_count = shift_key_index[j];
            std::vector<double> message_rotation_result(row_size, 0);
            for (int i = 0; i < row_size; i++)
            {
                int index = ((i + shift_count) < 0)
                                ? ((i + shift_count) + row_size)
                                : ((i + shift_count) % row_size);
                message_rotation_result[i] = message1[index];
            }

            double scale = pow(2.0, 30);
            heongpu::Plaintext<heongpu::Scheme::CKKS> P1(context);
            encoder.encode(P1, message1, scale);

            heongpu::Ciphertext<heongpu::Scheme::CKKS> C1(context);
            encryptor.encrypt(C1, P1);

            operators.rotate_rows(C1, C1, galois_key, shift_count);

            heongpu::Plaintext<heongpu::Scheme::CKKS> P3(context);
            decryptor.decrypt(P3, C1);

            std::vector<double> gpu_result;
            encoder.decode(gpu_result, P3);

            hipDeviceSynchronize();

            EXPECT_EQ(fix_point_array_check(message_rotation_result, gpu_result,
                                            static_cast<double>(1e-1)),
                      true);

            // Leveled Test
            operators.mod_drop_inplace(C1);
            heongpu::Plaintext<heongpu::Scheme::CKKS> P4(context);
            decryptor.decrypt(P4, C1);

            std::vector<double> gpu_result2;
            encoder.decode(gpu_result2, P4);

            hipDeviceSynchronize();

            EXPECT_EQ(fix_point_array_check(message_rotation_result,
                                            gpu_result2,
                                            static_cast<double>(1e-1)),
                      true);
        }
    }

    hipDeviceSynchronize();

    {
        size_t poly_modulus_degree = 65536;
        // TODO: find optimal way to store huge galois key, maybe store it in
        // CPU RAM.
        // heongpu::HEContext<heongpu::Scheme::CKKS>
        // context(heongpu::scheme_type::ckks,
        // heongpu::keyswitching_type::KEYSWITCHING_METHOD_II,
        // heongpu::sec_level_type::none);
        // context.set_poly_modulus_degree(poly_modulus_degree);
        // context.set_coeff_modulus_bit_sizes({59, 45, 45, 45, 45, 45, 45, 45,
        // 45, 45, 45, 45, 45,
        //     45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45,
        //     45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45},
        //    {59});
        // context.generate();
        heongpu::HEContext<heongpu::Scheme::CKKS> context(
            heongpu::keyswitching_type::KEYSWITCHING_METHOD_II,
            heongpu::sec_level_type::none);
        context.set_poly_modulus_degree(poly_modulus_degree);
        context.set_coeff_modulus_bit_sizes(
            {59, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40, 40},
            {59, 59});
        context.generate();

        heongpu::HEKeyGenerator<heongpu::Scheme::CKKS> keygen(context);
        heongpu::Secretkey<heongpu::Scheme::CKKS> secret_key(context);
        keygen.generate_secret_key(secret_key);

        heongpu::Publickey<heongpu::Scheme::CKKS> public_key(context);
        keygen.generate_public_key(public_key, secret_key);

        heongpu::HEEncoder<heongpu::Scheme::CKKS> encoder(context);
        heongpu::HEEncryptor<heongpu::Scheme::CKKS> encryptor(context,
                                                              public_key);
        heongpu::HEDecryptor<heongpu::Scheme::CKKS> decryptor(context,
                                                              secret_key);
        heongpu::HEArithmeticOperator<heongpu::Scheme::CKKS> operators(context,
                                                                       encoder);

        std::vector<int> shift_key_index = {-5, -2, 31};
        heongpu::Galoiskey<heongpu::Scheme::CKKS> galois_key(context,
                                                             shift_key_index);
        keygen.generate_galois_key(galois_key, secret_key);

        for (size_t j = 0; j < shift_key_index.size(); j++)
        {
            std::random_device rd;
            std::mt19937 gen(rd());
            std::uniform_real_distribution<> dis(0.0, 1.0);
            const int row_size = poly_modulus_degree / 2;
            std::vector<double> message1(row_size, 0);
            for (int i = 0; i < row_size; i++)
            {
                message1[i] = dis(gen);
            }

            int shift_count = shift_key_index[j];
            std::vector<double> message_rotation_result(row_size, 0);
            for (int i = 0; i < row_size; i++)
            {
                int index = ((i + shift_count) < 0)
                                ? ((i + shift_count) + row_size)
                                : ((i + shift_count) % row_size);
                message_rotation_result[i] = message1[index];
            }

            double scale = pow(2.0, 30);
            heongpu::Plaintext<heongpu::Scheme::CKKS> P1(context);
            encoder.encode(P1, message1, scale);

            heongpu::Ciphertext<heongpu::Scheme::CKKS> C1(context);
            encryptor.encrypt(C1, P1);

            operators.rotate_rows(C1, C1, galois_key, shift_count);

            heongpu::Plaintext<heongpu::Scheme::CKKS> P3(context);
            decryptor.decrypt(P3, C1);

            std::vector<double> gpu_result;
            encoder.decode(gpu_result, P3);

            hipDeviceSynchronize();

            EXPECT_EQ(fix_point_array_check(message_rotation_result, gpu_result,
                                            static_cast<double>(1e-1)),
                      true);

            // Leveled Test
            operators.mod_drop_inplace(C1);
            heongpu::Plaintext<heongpu::Scheme::CKKS> P4(context);
            decryptor.decrypt(P4, C1);

            std::vector<double> gpu_result2;
            encoder.decode(gpu_result2, P4);

            hipDeviceSynchronize();

            EXPECT_EQ(fix_point_array_check(message_rotation_result,
                                            gpu_result2,
                                            static_cast<double>(1e-1)),
                      true);
        }
    }

    hipDeviceSynchronize();
}

int main(int argc, char** argv)
{
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}