﻿// Copyright 2024-2025 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "bfv/keygenerator.cuh"

namespace heongpu
{
    __host__
    HEKeyGenerator<Scheme::BFV>::HEKeyGenerator(HEContext<Scheme::BFV>& context)
    {
        if (!context.context_generated_)
        {
            throw std::invalid_argument("HEContext is not generated!");
        }

        scheme = context.scheme_;

        std::random_device rd;
        std::mt19937 gen(rd());
        seed_ = gen();
        offset_ = gen();

        new_seed_ = RNGSeed();

        n = context.n;
        n_power = context.n_power;

        Q_prime_size_ = context.Q_prime_size;
        Q_size_ = context.Q_size;
        P_size_ = context.P_size;

        modulus_ = context.modulus_;
        ntt_table_ = context.ntt_table_;
        intt_table_ = context.intt_table_;
        n_inverse_ = context.n_inverse_;
        factor_ = context.factor_;

        d_ = context.d;
        d_tilda_ = context.d_tilda;
        r_prime_ = context.r_prime;

        B_prime_ = context.B_prime_;
        B_prime_ntt_tables_ = context.B_prime_ntt_tables_;
        B_prime_intt_tables_ = context.B_prime_intt_tables_;
        B_prime_n_inverse_ = context.B_prime_n_inverse_;

        base_change_matrix_D_to_B_ = context.base_change_matrix_D_to_B_;
        base_change_matrix_B_to_D_ = context.base_change_matrix_B_to_D_;
        Mi_inv_D_to_B_ = context.Mi_inv_D_to_B_;
        Mi_inv_B_to_D_ = context.Mi_inv_B_to_D_;
        prod_D_to_B_ = context.prod_D_to_B_;
        prod_B_to_D_ = context.prod_B_to_D_;

        I_j_ = context.I_j_;
        I_location_ = context.I_location_;
        Sk_pair_ = context.Sk_pair_;
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_secret_key(
        Secretkey<Scheme::BFV>& sk, const ExecutionOptions& options)
    {
        if (sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                DeviceVector<int> secret_key_without_rns((n), options.stream_);

                secretkey_gen_kernel<<<dim3((n >> 8), 1, 1), 256, 0,
                                       options.stream_>>>(
                    secret_key_without_rns.data(), sk_.hamming_weight_, n_power,
                    seed_);
                HEONGPU_CUDA_CHECK(hipGetLastError());

                DeviceVector<Data64> secret_key_rns(
                    (sk_.coeff_modulus_count() * n), options.stream_);

                secretkey_rns_kernel<<<dim3((n >> 8), 1, 1), 256, 0,
                                       options.stream_>>>(
                    secret_key_without_rns.data(), secret_key_rns.data(),
                    modulus_->data(), n_power, Q_prime_size_);
                HEONGPU_CUDA_CHECK(hipGetLastError());

                gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                    .n_power = n_power,
                    .ntt_type = gpuntt::FORWARD,
                    .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                    .zero_padding = false,
                    .stream = options.stream_};

                gpuntt::GPU_NTT_Inplace(secret_key_rns.data(),
                                        ntt_table_->data(), modulus_->data(),
                                        cfg_ntt, Q_prime_size_, Q_prime_size_);
                HEONGPU_CUDA_CHECK(hipGetLastError());

                sk_.in_ntt_domain_ = true;
                sk_.secret_key_generated_ = true;

                sk_.memory_set(std::move(secret_key_rns));
            },
            options, true);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_public_key(
        Publickey<Scheme::BFV>& pk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (pk.public_key_generated_)
        {
            throw std::logic_error("Publickey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                output_storage_manager(
                    pk,
                    [&](Publickey<Scheme::BFV>& pk_)
                    {
                        DeviceVector<Data64> output_memory(
                            (2 * Q_prime_size_ * n), options.stream_);

                        DeviceVector<Data64> errors_a(2 * Q_prime_size_ * n,
                                                      options.stream_);
                        Data64* error_poly = errors_a.data();
                        Data64* a_poly = error_poly + (Q_prime_size_ * n);

                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, 1, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, 1, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(errors_a.data(),
                                                ntt_table_->data(),
                                                modulus_->data(), cfg_ntt,
                                                Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        publickey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                               256, 0, options.stream_>>>(
                            output_memory.data(), sk_.data(), error_poly,
                            a_poly, modulus_->data(), n_power, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        pk_.memory_set(std::move(output_memory));

                        pk_.in_ntt_domain_ = true;
                        pk_.public_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_relin_key_method_I(
        Relinkey<Scheme::BFV>& rk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (rk.relin_key_generated_)
        {
            throw std::logic_error("Relinkey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                output_storage_manager(
                    rk,
                    [&](Relinkey<Scheme::BFV>& rk_)
                    {
                        DeviceVector<Data64> errors_a(
                            2 * Q_prime_size_ * Q_size_ * n, options.stream_);
                        Data64* error_poly = errors_a.data();
                        Data64* a_poly =
                            error_poly + (Q_prime_size_ * Q_size_ * n);

                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, Q_size_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, Q_size_,
                                options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        DeviceVector<Data64> output_memory(rk_.relinkey_size_,
                                                           options.stream_);

                        relinkey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                              256, 0, options.stream_>>>(
                            output_memory.data(), sk_.data(), error_poly,
                            a_poly, modulus_->data(), factor_->data(), n_power,
                            Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        rk_.memory_set(std::move(output_memory));

                        rk_.relin_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_bfv_relin_key_method_II(
        Relinkey<Scheme::BFV>& rk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (rk.relin_key_generated_)
        {
            throw std::logic_error("Relinkey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                output_storage_manager(
                    rk,
                    [&](Relinkey<Scheme::BFV>& rk_)
                    {
                        DeviceVector<Data64> errors_a(
                            2 * Q_prime_size_ * d_ * n, options.stream_);
                        Data64* error_poly = errors_a.data();
                        Data64* a_poly = error_poly + (Q_prime_size_ * d_ * n);

                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, d_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, d_, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, d_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        DeviceVector<Data64> output_memory(rk_.relinkey_size_,
                                                           options.stream_);

                        relinkey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_,
                                                      1),
                                                 256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), Sk_pair_->data(),
                            n_power, Q_prime_size_, d_, Q_size_, P_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        rk_.memory_set(std::move(output_memory));

                        rk_.relin_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void
    HEKeyGenerator<Scheme::BFV>::generate_bfv_relin_key_method_III(
        Relinkey<Scheme::BFV>& rk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (rk.relin_key_generated_)
        {
            throw std::logic_error("Relinkey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                output_storage_manager(
                    rk,
                    [&](Relinkey<Scheme::BFV>& rk_)
                    {
                        DeviceVector<Data64> errors_a(
                            2 * Q_prime_size_ * d_ * n, options.stream_);
                        Data64* error_poly = errors_a.data();
                        Data64* a_poly = error_poly + (Q_prime_size_ * d_ * n);

                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, d_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, d_, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, d_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        DeviceVector<Data64> temp_calculation(
                            2 * Q_prime_size_ * d_ * n, options.stream_);

                        relinkey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_,
                                                      1),
                                                 256, 0, options.stream_>>>(
                            temp_calculation.data(), sk.data(), error_poly,
                            a_poly, modulus_->data(), factor_->data(),
                            Sk_pair_->data(), n_power, Q_prime_size_, d_,
                            Q_size_, P_size_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::INVERSE,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .mod_inverse = n_inverse_->data(),
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            temp_calculation.data(), intt_table_->data(),
                            modulus_->data(), cfg_intt, 2 * Q_prime_size_ * d_,
                            Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        DeviceVector<Data64> output_memory(rk_.relinkey_size_,
                                                           options.stream_);

                        relinkey_DtoB_kernel<<<dim3((n >> 8), d_tilda_,
                                                    (d_ << 1)),
                                               256, 0, options.stream_>>>(
                            temp_calculation.data(), output_memory.data(),
                            modulus_->data(), B_prime_->data(),
                            base_change_matrix_D_to_B_->data(),
                            Mi_inv_D_to_B_->data(), prod_D_to_B_->data(),
                            I_j_->data(), I_location_->data(), n_power,
                            Q_prime_size_, d_tilda_, d_, r_prime_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        gpuntt::GPU_NTT_Inplace(
                            output_memory.data(), B_prime_ntt_tables_->data(),
                            B_prime_->data(), cfg_ntt,
                            2 * d_tilda_ * d_ * r_prime_, r_prime_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        rk_.memory_set(std::move(output_memory));

                        rk_.relin_key_generated_ = true;
                    },
                    options);
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_galois_key_method_I(
        Galoiskey<Scheme::BFV>& gk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (gk.galois_key_generated_)
        {
            throw std::logic_error("Galoiskey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                DeviceVector<Data64> errors_a(2 * Q_prime_size_ * Q_size_ * n,
                                              options.stream_);
                Data64* error_poly = errors_a.data();
                Data64* a_poly = error_poly + (Q_prime_size_ * Q_size_ * n);

                if (!gk.customized)
                {
                    // Positive Row Shift
                    for (auto& galois : gk.galois_elt)
                    {
                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, Q_size_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, Q_size_,
                                options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        int inv_galois = modInverse(galois.second, 2 * n);

                        DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                           options.stream_);

                        galoiskey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                               256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), inv_galois,
                            n_power, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        if (options.storage_ == storage_type::DEVICE)
                        {
                            gk.device_location_[galois.second] =
                                std::move(output_memory);
                        }
                        else
                        {
                            gk.host_location_[galois.second] =
                                HostVector<Data64>(gk.galoiskey_size_);
                            hipMemcpyAsync(
                                gk.host_location_[galois.second].data(),
                                output_memory.data(),
                                gk.galoiskey_size_ * sizeof(Data64),
                                hipMemcpyDeviceToHost, options.stream_);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }
                    }

                    // Columns Rotate
                    RandomNumberGenerator::instance()
                        .modular_uniform_random_number_generation(
                            a_poly, modulus_->data(), n_power, Q_prime_size_,
                            Q_size_, options.stream_);

                    RandomNumberGenerator::instance()
                        .modular_gaussian_random_number_generation(
                            error_std_dev, error_poly, modulus_->data(),
                            n_power, Q_prime_size_, Q_size_, options.stream_);

                    gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                        .n_power = n_power,
                        .ntt_type = gpuntt::FORWARD,
                        .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                        .zero_padding = false,
                        .stream = options.stream_};

                    gpuntt::GPU_NTT_Inplace(
                        error_poly, ntt_table_->data(), modulus_->data(),
                        cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                       options.stream_);

                    galoiskey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                           256, 0, options.stream_>>>(
                        output_memory.data(), sk.data(), error_poly, a_poly,
                        modulus_->data(), factor_->data(), gk.galois_elt_zero,
                        n_power, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    if (options.storage_ == storage_type::DEVICE)
                    {
                        gk.zero_device_location_ = std::move(output_memory);
                    }
                    else
                    {
                        gk.zero_host_location_ =
                            HostVector<Data64>(gk.galoiskey_size_);
                        hipMemcpyAsync(
                            gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());
                    }
                }
                else
                {
                    for (auto& galois_ : gk.custom_galois_elt)
                    {
                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, Q_size_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, Q_size_,
                                options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        int inv_galois = modInverse(galois_, 2 * n);

                        DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                           options.stream_);

                        galoiskey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                               256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), inv_galois,
                            n_power, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        if (options.storage_ == storage_type::DEVICE)
                        {
                            gk.device_location_[galois_] =
                                std::move(output_memory);
                        }
                        else
                        {
                            gk.host_location_[galois_] =
                                HostVector<Data64>(gk.galoiskey_size_);
                            hipMemcpyAsync(gk.host_location_[galois_].data(),
                                            output_memory.data(),
                                            gk.galoiskey_size_ * sizeof(Data64),
                                            hipMemcpyDeviceToHost,
                                            options.stream_);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }
                    }

                    // Columns Rotate
                    RandomNumberGenerator::instance()
                        .modular_uniform_random_number_generation(
                            a_poly, modulus_->data(), n_power, Q_prime_size_,
                            Q_size_, options.stream_);

                    RandomNumberGenerator::instance()
                        .modular_gaussian_random_number_generation(
                            error_std_dev, error_poly, modulus_->data(),
                            n_power, Q_prime_size_, Q_size_, options.stream_);

                    gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                        .n_power = n_power,
                        .ntt_type = gpuntt::FORWARD,
                        .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                        .zero_padding = false,
                        .stream = options.stream_};

                    gpuntt::GPU_NTT_Inplace(
                        error_poly, ntt_table_->data(), modulus_->data(),
                        cfg_ntt, Q_size_ * Q_prime_size_, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                       options.stream_);

                    galoiskey_gen_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                           256, 0, options.stream_>>>(
                        output_memory.data(), sk.data(), error_poly, a_poly,
                        modulus_->data(), factor_->data(), gk.galois_elt_zero,
                        n_power, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    if (options.storage_ == storage_type::DEVICE)
                    {
                        gk.zero_device_location_ = std::move(output_memory);
                    }
                    else
                    {
                        gk.zero_host_location_ =
                            HostVector<Data64>(gk.galoiskey_size_);
                        hipMemcpyAsync(
                            gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());
                    }
                }

                gk.galois_key_generated_ = true;
                gk.storage_type_ = options.storage_;
            },
            options, false);
    }

    __host__ void
    HEKeyGenerator<Scheme::BFV>::generate_bfv_galois_key_method_II(
        Galoiskey<Scheme::BFV>& gk, Secretkey<Scheme::BFV>& sk,
        const ExecutionOptions& options)
    {
        if (!sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (gk.galois_key_generated_)
        {
            throw std::logic_error("Galoiskey is already generated!");
        }

        input_storage_manager(
            sk,
            [&](Secretkey<Scheme::BFV>& sk_)
            {
                DeviceVector<Data64> errors_a(2 * Q_prime_size_ * d_ * n,
                                              options.stream_);
                Data64* error_poly = errors_a.data();
                Data64* a_poly = error_poly + (Q_prime_size_ * d_ * n);

                if (!gk.customized)
                {
                    // Positive Row Shift
                    for (auto& galois : gk.galois_elt)
                    {
                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, d_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, d_, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, d_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        int inv_galois = modInverse(galois.second, 2 * n);

                        DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                           options.stream_);

                        galoiskey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_,
                                                       1),
                                                  256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), inv_galois,
                            Sk_pair_->data(), n_power, Q_prime_size_, d_,
                            Q_size_, P_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        if (options.storage_ == storage_type::DEVICE)
                        {
                            gk.device_location_[galois.second] =
                                std::move(output_memory);
                        }
                        else
                        {
                            gk.host_location_[galois.second] =
                                HostVector<Data64>(gk.galoiskey_size_);
                            hipMemcpyAsync(
                                gk.host_location_[galois.second].data(),
                                output_memory.data(),
                                gk.galoiskey_size_ * sizeof(Data64),
                                hipMemcpyDeviceToHost, options.stream_);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }
                    }

                    // Columns Rotate
                    RandomNumberGenerator::instance()
                        .modular_uniform_random_number_generation(
                            a_poly, modulus_->data(), n_power, Q_prime_size_,
                            d_, options.stream_);

                    RandomNumberGenerator::instance()
                        .modular_gaussian_random_number_generation(
                            error_std_dev, error_poly, modulus_->data(),
                            n_power, Q_prime_size_, d_, options.stream_);

                    gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                        .n_power = n_power,
                        .ntt_type = gpuntt::FORWARD,
                        .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                        .zero_padding = false,
                        .stream = options.stream_};

                    gpuntt::GPU_NTT_Inplace(error_poly, ntt_table_->data(),
                                            modulus_->data(), cfg_ntt,
                                            d_ * Q_prime_size_, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                       options.stream_);

                    galoiskey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                              256, 0, options.stream_>>>(
                        output_memory.data(), sk.data(), error_poly, a_poly,
                        modulus_->data(), factor_->data(), gk.galois_elt_zero,
                        Sk_pair_->data(), n_power, Q_prime_size_, d_, Q_size_,
                        P_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    if (options.storage_ == storage_type::DEVICE)
                    {
                        gk.zero_device_location_ = std::move(output_memory);
                    }
                    else
                    {
                        gk.zero_host_location_ =
                            HostVector<Data64>(gk.galoiskey_size_);
                        hipMemcpyAsync(
                            gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());
                    }
                }
                else
                {
                    for (auto& galois_ : gk.custom_galois_elt)
                    {
                        RandomNumberGenerator::instance()
                            .modular_uniform_random_number_generation(
                                a_poly, modulus_->data(), n_power,
                                Q_prime_size_, d_, options.stream_);

                        RandomNumberGenerator::instance()
                            .modular_gaussian_random_number_generation(
                                error_std_dev, error_poly, modulus_->data(),
                                n_power, Q_prime_size_, d_, options.stream_);

                        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                            .n_power = n_power,
                            .ntt_type = gpuntt::FORWARD,
                            .reduction_poly =
                                gpuntt::ReductionPolynomial::X_N_plus,
                            .zero_padding = false,
                            .stream = options.stream_};

                        gpuntt::GPU_NTT_Inplace(
                            error_poly, ntt_table_->data(), modulus_->data(),
                            cfg_ntt, d_ * Q_prime_size_, Q_prime_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        int inv_galois = modInverse(galois_, 2 * n);

                        DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                           options.stream_);

                        galoiskey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_,
                                                       1),
                                                  256, 0, options.stream_>>>(
                            output_memory.data(), sk.data(), error_poly, a_poly,
                            modulus_->data(), factor_->data(), inv_galois,
                            Sk_pair_->data(), n_power, Q_prime_size_, d_,
                            Q_size_, P_size_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        if (options.storage_ == storage_type::DEVICE)
                        {
                            gk.device_location_[galois_] =
                                std::move(output_memory);
                        }
                        else
                        {
                            gk.host_location_[galois_] =
                                HostVector<Data64>(gk.galoiskey_size_);
                            hipMemcpyAsync(gk.host_location_[galois_].data(),
                                            output_memory.data(),
                                            gk.galoiskey_size_ * sizeof(Data64),
                                            hipMemcpyDeviceToHost,
                                            options.stream_);
                            HEONGPU_CUDA_CHECK(hipGetLastError());
                        }
                    }

                    // Columns Rotate
                    RandomNumberGenerator::instance()
                        .modular_uniform_random_number_generation(
                            a_poly, modulus_->data(), n_power, Q_prime_size_,
                            d_, options.stream_);

                    RandomNumberGenerator::instance()
                        .modular_gaussian_random_number_generation(
                            error_std_dev, error_poly, modulus_->data(),
                            n_power, Q_prime_size_, d_, options.stream_);

                    gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
                        .n_power = n_power,
                        .ntt_type = gpuntt::FORWARD,
                        .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
                        .zero_padding = false,
                        .stream = options.stream_};

                    gpuntt::GPU_NTT_Inplace(error_poly, ntt_table_->data(),
                                            modulus_->data(), cfg_ntt,
                                            d_ * Q_prime_size_, Q_prime_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    DeviceVector<Data64> output_memory(gk.galoiskey_size_,
                                                       options.stream_);

                    galoiskey_gen_II_kernel<<<dim3((n >> 8), Q_prime_size_, 1),
                                              256, 0, options.stream_>>>(
                        output_memory.data(), sk.data(), error_poly, a_poly,
                        modulus_->data(), factor_->data(), gk.galois_elt_zero,
                        Sk_pair_->data(), n_power, Q_prime_size_, d_, Q_size_,
                        P_size_);
                    HEONGPU_CUDA_CHECK(hipGetLastError());

                    if (options.storage_ == storage_type::DEVICE)
                    {
                        gk.zero_device_location_ = std::move(output_memory);
                    }
                    else
                    {
                        gk.zero_host_location_ =
                            HostVector<Data64>(gk.galoiskey_size_);
                        hipMemcpyAsync(
                            gk.zero_host_location_.data(), output_memory.data(),
                            gk.galoiskey_size_ * sizeof(Data64),
                            hipMemcpyDeviceToHost, options.stream_);
                        HEONGPU_CUDA_CHECK(hipGetLastError());
                    }
                }

                gk.galois_key_generated_ = true;
                gk.storage_type_ = options.storage_;
            },
            options, false);
    }

    __host__ void HEKeyGenerator<Scheme::BFV>::generate_switch_key_method_I(
        Switchkey<Scheme::BFV>& swk, Secretkey<Scheme::BFV>& new_sk,
        Secretkey<Scheme::BFV>& old_sk, const ExecutionOptions& options)
    {
        if (!old_sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (!new_sk.secret_key_generated_)
        {
            throw std::logic_error("Ner Secretkey is not generated!");
        }

        if (swk.switch_key_generated_)
        {
            throw std::logic_error("Switchkey is already generated!");
        }

        input_storage_manager(
            old_sk,
            [&](Secretkey<Scheme::BFV>& old_sk_)
            {
                input_storage_manager(
                    new_sk,
                    [&](Secretkey<Scheme::BFV>& new_sk_)
                    {
                        output_storage_manager(
                            swk,
                            [&](Switchkey<Scheme::BFV>& swk_)
                            {
                                DeviceVector<Data64> errors_a(
                                    2 * Q_prime_size_ * Q_size_ * n,
                                    options.stream_);
                                Data64* error_poly = errors_a.data();
                                Data64* a_poly =
                                    error_poly + (Q_prime_size_ * Q_size_ * n);

                                RandomNumberGenerator::instance()
                                    .modular_uniform_random_number_generation(
                                        a_poly, modulus_->data(), n_power,
                                        Q_prime_size_, Q_size_,
                                        options.stream_);

                                RandomNumberGenerator::instance()
                                    .modular_gaussian_random_number_generation(
                                        error_std_dev, error_poly,
                                        modulus_->data(), n_power,
                                        Q_prime_size_, Q_size_,
                                        options.stream_);

                                gpuntt::ntt_rns_configuration<Data64> cfg_ntt =
                                    {.n_power = n_power,
                                     .ntt_type = gpuntt::FORWARD,
                                     .reduction_poly =
                                         gpuntt::ReductionPolynomial::X_N_plus,
                                     .zero_padding = false,
                                     .stream = options.stream_};

                                gpuntt::GPU_NTT_Inplace(
                                    error_poly, ntt_table_->data(),
                                    modulus_->data(), cfg_ntt,
                                    Q_size_ * Q_prime_size_, Q_prime_size_);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                DeviceVector<Data64> output_memory(
                                    swk_.switchkey_size_, options.stream_);

                                switchkey_gen_kernel<<<
                                    dim3((n >> 8), Q_prime_size_, 1), 256, 0,
                                    options.stream_>>>(
                                    output_memory.data(), new_sk.data(),
                                    old_sk.data(), error_poly, a_poly,
                                    modulus_->data(), factor_->data(), n_power,
                                    Q_prime_size_);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                swk.memory_set(std::move(output_memory));

                                swk.switch_key_generated_ = true;
                            },
                            options);
                    },
                    options, false);
            },
            options, false);
    }

    __host__ void
    HEKeyGenerator<Scheme::BFV>::generate_bfv_switch_key_method_II(
        Switchkey<Scheme::BFV>& swk, Secretkey<Scheme::BFV>& new_sk,
        Secretkey<Scheme::BFV>& old_sk, const ExecutionOptions& options)
    {
        if (!old_sk.secret_key_generated_)
        {
            throw std::logic_error("Secretkey is not generated!");
        }

        if (!new_sk.secret_key_generated_)
        {
            throw std::logic_error("Ner Secretkey is not generated!");
        }

        if (swk.switch_key_generated_)
        {
            throw std::logic_error("Switchkey is already generated!");
        }

        input_storage_manager(
            old_sk,
            [&](Secretkey<Scheme::BFV>& old_sk_)
            {
                input_storage_manager(
                    new_sk,
                    [&](Secretkey<Scheme::BFV>& new_sk_)
                    {
                        output_storage_manager(
                            swk,
                            [&](Switchkey<Scheme::BFV>& swk_)
                            {
                                DeviceVector<Data64> errors_a(
                                    2 * Q_prime_size_ * d_ * n,
                                    options.stream_);
                                Data64* error_poly = errors_a.data();
                                Data64* a_poly =
                                    error_poly + (Q_prime_size_ * d_ * n);

                                RandomNumberGenerator::instance()
                                    .modular_uniform_random_number_generation(
                                        a_poly, modulus_->data(), n_power,
                                        Q_prime_size_, d_, options.stream_);

                                RandomNumberGenerator::instance()
                                    .modular_gaussian_random_number_generation(
                                        error_std_dev, error_poly,
                                        modulus_->data(), n_power,
                                        Q_prime_size_, d_, options.stream_);

                                gpuntt::ntt_rns_configuration<Data64> cfg_ntt =
                                    {.n_power = n_power,
                                     .ntt_type = gpuntt::FORWARD,
                                     .reduction_poly =
                                         gpuntt::ReductionPolynomial::X_N_plus,
                                     .zero_padding = false,
                                     .stream = options.stream_};

                                gpuntt::GPU_NTT_Inplace(
                                    error_poly, ntt_table_->data(),
                                    modulus_->data(), cfg_ntt,
                                    d_ * Q_prime_size_, Q_prime_size_);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                DeviceVector<Data64> output_memory(
                                    swk_.switchkey_size_, options.stream_);

                                switchkey_gen_II_kernel<<<
                                    dim3((n >> 8), Q_prime_size_, 1), 256, 0,
                                    options.stream_>>>(
                                    output_memory.data(), new_sk.data(),
                                    old_sk.data(), error_poly, a_poly,
                                    modulus_->data(), factor_->data(),
                                    Sk_pair_->data(), n_power, Q_prime_size_,
                                    d_, Q_size_, P_size_);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                swk.memory_set(std::move(output_memory));

                                swk.switch_key_generated_ = true;
                            },
                            options);
                    },
                    options, false);
            },
            options, false);
    }

} // namespace heongpu