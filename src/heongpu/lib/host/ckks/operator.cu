#include "hip/hip_runtime.h"
// Copyright 2024-2025 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "ckks/operator.cuh"

namespace heongpu
{
    __host__
    HEOperator<Scheme::CKKS>::HEOperator(HEContext<Scheme::CKKS>& context,
                                         HEEncoder<Scheme::CKKS>& encoder)
    {
        if (!context.context_generated_)
        {
            throw std::invalid_argument("HEContext is not generated!");
        }

        scheme_ = context.scheme_;

        n = context.n;

        n_power = context.n_power;

        Q_prime_size_ = context.Q_prime_size;
        Q_size_ = context.Q_size;
        P_size_ = context.P_size;

        modulus_ = context.modulus_;

        ntt_table_ = context.ntt_table_;

        intt_table_ = context.intt_table_;

        n_inverse_ = context.n_inverse_;

        last_q_modinv_ = context.last_q_modinv_;

        half_p_ = context.half_p_;

        half_mod_ = context.half_mod_;

        //////

        l_leveled_ = context.l_leveled;
        l_tilda_leveled_ = context.l_tilda_leveled;
        d_leveled_ = context.d_leveled;
        d_tilda_leveled_ = context.d_tilda_leveled;
        r_prime_leveled_ = context.r_prime_leveled;

        B_prime_leveled_ = context.B_prime_leveled;
        B_prime_ntt_tables_leveled_ = context.B_prime_ntt_tables_leveled;
        B_prime_intt_tables_leveled_ = context.B_prime_intt_tables_leveled;
        B_prime_n_inverse_leveled_ = context.B_prime_n_inverse_leveled;

        Mi_inv_B_to_D_leveled_ = context.Mi_inv_B_to_D_leveled;
        base_change_matrix_D_to_B_leveled_ =
            context.base_change_matrix_D_to_B_leveled;
        base_change_matrix_B_to_D_leveled_ =
            context.base_change_matrix_B_to_D_leveled;
        Mi_inv_D_to_B_leveled_ = context.Mi_inv_D_to_B_leveled;
        prod_D_to_B_leveled_ = context.prod_D_to_B_leveled;
        prod_B_to_D_leveled_ = context.prod_B_to_D_leveled;

        // Method2
        base_change_matrix_D_to_Qtilda_leveled_ =
            context.base_change_matrix_D_to_Qtilda_leveled;
        Mi_inv_D_to_Qtilda_leveled_ = context.Mi_inv_D_to_Qtilda_leveled;
        prod_D_to_Qtilda_leveled_ = context.prod_D_to_Qtilda_leveled;

        I_j_leveled_ = context.I_j_leveled;
        I_location_leveled_ = context.I_location_leveled;
        Sk_pair_leveled_ = context.Sk_pair_leveled;

        prime_location_leveled_ = context.prime_location_leveled;

        // Leveled Rescale
        rescaled_last_q_modinv_ = context.rescaled_last_q_modinv_;
        rescaled_half_ = context.rescaled_half_;
        rescaled_half_mod_ = context.rescaled_half_mod_;

        prime_vector_ = context.prime_vector_;

        std::vector<int> prime_loc;
        std::vector<int> input_loc;

        int counter = Q_size_;
        for (int i = 0; i < Q_size_ - 1; i++)
        {
            for (int j = 0; j < counter; j++)
            {
                prime_loc.push_back(j);
            }
            counter--;
            for (int j = 0; j < P_size_; j++)
            {
                prime_loc.push_back(Q_size_ + j);
            }
        }

        counter = Q_prime_size_;
        for (int i = 0; i < Q_prime_size_ - 1; i++)
        {
            int sum = counter - 1;
            for (int j = 0; j < 2; j++)
            {
                input_loc.push_back(sum);
                sum += counter;
            }
            counter--;
        }

        new_prime_locations_ = DeviceVector<int>(prime_loc);
        new_input_locations_ = DeviceVector<int>(input_loc);
        new_prime_locations = new_prime_locations_.data();
        new_input_locations = new_input_locations_.data();

        // Encode params
        slot_count_ = encoder.slot_count_;
        log_slot_count_ = encoder.log_slot_count_;
        two_pow_64_ = encoder.two_pow_64;
        reverse_order_ = encoder.reverse_order;
        special_ifft_roots_table_ = encoder.special_ifft_roots_table_;
    }

    __host__ void HEOperator<Scheme::CKKS>::add(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& input2,
        Ciphertext<Scheme::CKKS>& output, const ExecutionOptions& options)
    {
        if (input1.depth_ != input2.depth_)
        {
            throw std::logic_error("Ciphertexts leveled are not equal");
        }

        if (input1.relinearization_required_ !=
            input2.relinearization_required_)
        {
            throw std::invalid_argument("Ciphertexts can not be added because "
                                        "ciphertext sizes have to be equal!");
        }

        if (input1.in_ntt_domain_ != input2.in_ntt_domain_)
        {
            throw std::invalid_argument(
                "Both Ciphertexts should be in same domain");
        }

        int cipher_size = input1.relinearization_required_ ? 3 : 2;

        int current_decomp_count = Q_size_ - input1.depth_;

        if (input1.memory_size() < (cipher_size * n * current_decomp_count) ||
            input2.memory_size() < (cipher_size * n * current_decomp_count))
        {
            throw std::invalid_argument("Invalid Ciphertexts size!");
        }

        input_storage_manager(
            input1,
            [&](Ciphertext<Scheme::CKKS>& input1_)
            {
                input_storage_manager(
                    input2,
                    [&](Ciphertext<Scheme::CKKS>& input2_)
                    {
                        output_storage_manager(
                            output,
                            [&](Ciphertext<Scheme::CKKS>& output_)
                            {
                                DeviceVector<Data64> output_memory(
                                    (cipher_size * n * current_decomp_count),
                                    options.stream_);

                                addition<<<dim3((n >> 8), current_decomp_count,
                                                cipher_size),
                                           256, 0, options.stream_>>>(
                                    input1_.data(), input2_.data(),
                                    output_memory.data(), modulus_->data(),
                                    n_power);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                output_.scheme_ = scheme_;
                                output_.ring_size_ = n;
                                output_.coeff_modulus_count_ = Q_size_;
                                output_.cipher_size_ = cipher_size;
                                output_.depth_ = input1_.depth_;
                                output_.in_ntt_domain_ = input1_.in_ntt_domain_;
                                output_.scale_ = input1_.scale_;
                                output_.rescale_required_ =
                                    (input1_.rescale_required_ ||
                                     input2_.rescale_required_);
                                output_.relinearization_required_ =
                                    input1_.relinearization_required_;
                                output_.ciphertext_generated_ = true;

                                output_.memory_set(std::move(output_memory));
                            },
                            options);
                    },
                    options, (&input2 == &output));
            },
            options, (&input1 == &output));
    }

    __host__ void HEOperator<Scheme::CKKS>::sub(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& input2,
        Ciphertext<Scheme::CKKS>& output, const ExecutionOptions& options)
    {
        if (input1.depth_ != input2.depth_)
        {
            throw std::logic_error("Ciphertexts leveled are not equal");
        }

        if (input1.relinearization_required_ !=
            input2.relinearization_required_)
        {
            throw std::invalid_argument("Ciphertexts can not be added because "
                                        "ciphertext sizes have to be equal!");
        }

        if (input1.in_ntt_domain_ != input2.in_ntt_domain_)
        {
            throw std::invalid_argument(
                "Both Ciphertexts should be in same domain");
        }

        int cipher_size = input1.relinearization_required_ ? 3 : 2;

        int current_decomp_count = Q_size_ - input1.depth_;

        if (input1.memory_size() < (cipher_size * n * current_decomp_count) ||
            input2.memory_size() < (cipher_size * n * current_decomp_count))
        {
            throw std::invalid_argument("Invalid Ciphertexts size!");
        }

        input_storage_manager(
            input1,
            [&](Ciphertext<Scheme::CKKS>& input1_)
            {
                input_storage_manager(
                    input2,
                    [&](Ciphertext<Scheme::CKKS>& input2_)
                    {
                        output_storage_manager(
                            output,
                            [&](Ciphertext<Scheme::CKKS>& output_)
                            {
                                DeviceVector<Data64> output_memory(
                                    (cipher_size * n * current_decomp_count),
                                    options.stream_);

                                substraction<<<dim3((n >> 8),
                                                    current_decomp_count,
                                                    cipher_size),
                                               256, 0, options.stream_>>>(
                                    input1_.data(), input2_.data(),
                                    output_memory.data(), modulus_->data(),
                                    n_power);
                                HEONGPU_CUDA_CHECK(hipGetLastError());

                                output_.scheme_ = scheme_;
                                output_.ring_size_ = n;
                                output_.coeff_modulus_count_ = Q_size_;
                                output_.cipher_size_ = cipher_size;
                                output_.depth_ = input1_.depth_;
                                output_.in_ntt_domain_ = input1_.in_ntt_domain_;
                                output_.scale_ = input1_.scale_;
                                output_.rescale_required_ =
                                    (input1_.rescale_required_ ||
                                     input2_.rescale_required_);
                                output_.relinearization_required_ =
                                    input1_.relinearization_required_;
                                output_.ciphertext_generated_ = true;

                                output_.memory_set(std::move(output_memory));
                            },
                            options);
                    },
                    options, (&input2 == &output));
            },
            options, (&input1 == &output));
    }

    __host__ void
    HEOperator<Scheme::CKKS>::negate(Ciphertext<Scheme::CKKS>& input1,
                                     Ciphertext<Scheme::CKKS>& output,
                                     const ExecutionOptions& options)
    {
        int cipher_size = input1.relinearization_required_ ? 3 : 2;

        int current_decomp_count = Q_size_ - input1.depth_;

        if (input1.memory_size() < (cipher_size * n * current_decomp_count))
        {
            throw std::invalid_argument("Invalid Ciphertexts size!");
        }

        input_storage_manager(
            input1,
            [&](Ciphertext<Scheme::CKKS>& input1_)
            {
                output_storage_manager(
                    output,
                    [&](Ciphertext<Scheme::CKKS>& output_)
                    {
                        DeviceVector<Data64> output_memory(
                            (cipher_size * n * current_decomp_count),
                            options.stream_);

                        negation<<<dim3((n >> 8), current_decomp_count,
                                        cipher_size),
                                   256, 0, options.stream_>>>(
                            input1_.data(), output_memory.data(),
                            modulus_->data(), n_power);
                        HEONGPU_CUDA_CHECK(hipGetLastError());

                        output_.scheme_ = scheme_;
                        output_.ring_size_ = n;
                        output_.coeff_modulus_count_ = Q_size_;
                        output_.cipher_size_ = cipher_size;
                        output_.depth_ = input1_.depth_;
                        output_.in_ntt_domain_ = input1_.in_ntt_domain_;
                        output_.scale_ = input1_.scale_;
                        output_.rescale_required_ = input1_.rescale_required_;
                        output_.relinearization_required_ =
                            input1_.relinearization_required_;
                        output_.ciphertext_generated_ = true;

                        output_.memory_set(std::move(output_memory));
                    },
                    options);
            },
            options, (&input1 == &output));
    }

    __host__ void HEOperator<Scheme::CKKS>::add_plain_ckks(
        Ciphertext<Scheme::CKKS>& input1, Plaintext<Scheme::CKKS>& input2,
        Ciphertext<Scheme::CKKS>& output, const hipStream_t stream)
    {
        if (input1.depth_ != input2.depth_)
        {
            throw std::logic_error("Ciphertexts leveled are not equal");
        }

        int current_decomp_count = Q_size_ - input1.depth_;

        int cipher_size = input1.relinearization_required_ ? 3 : 2;

        if (input1.memory_size() < (cipher_size * n * current_decomp_count))
        {
            throw std::invalid_argument("Invalid Ciphertexts size!");
        }

        if (input2.size() < (n * current_decomp_count))
        {
            throw std::invalid_argument("Invalid Plaintext size!");
        }

        DeviceVector<Data64> output_memory(
            (cipher_size * n * current_decomp_count), stream);

        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count,
                                        cipher_size),
                                   256, 0, stream>>>(
            input1.data(), input2.data(), output_memory.data(),
            modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        output.cipher_size_ = cipher_size;

        output.memory_set(std::move(output_memory));
    }

    __host__ void HEOperator<Scheme::CKKS>::add_plain_ckks_inplace(
        Ciphertext<Scheme::CKKS>& input1, Plaintext<Scheme::CKKS>& input2,
        const hipStream_t stream)
    {
        if (input1.depth_ != input2.depth_)
        {
            throw std::logic_error("Ciphertexts leveled are not equal");
        }

        int current_decomp_count = Q_size_ - input1.depth_;

        int cipher_size = input1.relinearization_required_ ? 3 : 2;

        if (input1.memory_size() < (cipher_size * n * current_decomp_count))
        {
            throw std::invalid_argument("Invalid Ciphertexts size!");
        }

        if (input2.size() < (n * current_decomp_count))
        {
            throw std::invalid_argument("Invalid Plaintext size!");
        }

        addition<<<dim3((n >> 8), current_decomp_count, 1), 256, 0, stream>>>(
            input1.data(), input2.data(), input1.data(), modulus_->data(),
            n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    __host__ void HEOperator<Scheme::CKKS>::sub_plain_ckks(
        Ciphertext<Scheme::CKKS>& input1, Plaintext<Scheme::CKKS>& input2,
        Ciphertext<Scheme::CKKS>& output, const hipStream_t stream)
    {
        if (input1.depth_ != input2.depth_)
        {
            throw std::logic_error("Ciphertexts leveled are not equal");
        }

        int current_decomp_count = Q_size_ - input1.depth_;

        int cipher_size = input1.relinearization_required_ ? 3 : 2;

        if (input1.memory_size() < (cipher_size * n * current_decomp_count))
        {
            throw std::invalid_argument("Invalid Ciphertexts size!");
        }

        if (input2.size() < (n * current_decomp_count))
        {
            throw std::invalid_argument("Invalid Plaintext size!");
        }

        DeviceVector<Data64> output_memory(
            (cipher_size * n * current_decomp_count), stream);

        substraction_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count,
                                            cipher_size),
                                       256, 0, stream>>>(
            input1.data(), input2.data(), output_memory.data(),
            modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        output.cipher_size_ = cipher_size;

        output.memory_set(std::move(output_memory));
    }

    __host__ void HEOperator<Scheme::CKKS>::sub_plain_ckks_inplace(
        Ciphertext<Scheme::CKKS>& input1, Plaintext<Scheme::CKKS>& input2,
        const hipStream_t stream)
    {
        if (input1.depth_ != input2.depth_)
        {
            throw std::logic_error("Ciphertexts leveled are not equal");
        }

        int current_decomp_count = Q_size_ - input1.depth_;

        int cipher_size = input1.relinearization_required_ ? 3 : 2;

        if (input1.memory_size() < (cipher_size * n * current_decomp_count))
        {
            throw std::invalid_argument("Invalid Ciphertexts size!");
        }

        if (input2.size() < (n * current_decomp_count))
        {
            throw std::invalid_argument("Invalid Plaintext size!");
        }

        substraction<<<dim3((n >> 8), current_decomp_count, 1), 256, 0,
                       stream>>>(input1.data(), input2.data(), input1.data(),
                                 modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    __host__ void HEOperator<Scheme::CKKS>::multiply_ckks(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& input2,
        Ciphertext<Scheme::CKKS>& output, const hipStream_t stream)
    {
        if (input1.depth_ != input2.depth_)
        {
            throw std::logic_error("Ciphertexts leveled are not equal");
        }

        int current_decomp_count = Q_size_ - input1.depth_;

        if (input1.memory_size() < (2 * n * current_decomp_count) ||
            input2.memory_size() < (2 * n * current_decomp_count))
        {
            throw std::invalid_argument("Invalid Ciphertexts size!");
        }

        DeviceVector<Data64> output_memory((3 * n * current_decomp_count),
                                           stream);

        cross_multiplication<<<dim3((n >> 8), (current_decomp_count), 1), 256,
                               0, stream>>>(
            input1.data(), input2.data(), output_memory.data(),
            modulus_->data(), n_power, current_decomp_count);

        HEONGPU_CUDA_CHECK(hipGetLastError());

        output.memory_set(std::move(output_memory));

        if (scheme_ == scheme_type::ckks)
        {
            output.scale_ = input1.scale_ * input2.scale_;
        }
    }

    __host__ void HEOperator<Scheme::CKKS>::multiply_plain_ckks(
        Ciphertext<Scheme::CKKS>& input1, Plaintext<Scheme::CKKS>& input2,
        Ciphertext<Scheme::CKKS>& output, const hipStream_t stream)
    {
        if (input1.depth_ != input2.depth_)
        {
            throw std::logic_error("Ciphertexts leveled are not equal");
        }

        int current_decomp_count = Q_size_ - input1.depth_;
        DeviceVector<Data64> output_memory((2 * n * current_decomp_count),
                                           stream);

        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            input1.data(), input2.data(), output_memory.data(),
            modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        if (scheme_ == scheme_type::ckks)
        {
            output.scale_ = input1.scale_ * input2.scale_;
        }

        output.memory_set(std::move(output_memory));
    }

    __host__ void
    HEOperator<Scheme::CKKS>::relinearize_seal_method_inplace_ckks(
        Ciphertext<Scheme::CKKS>& input1, Relinkey<Scheme::CKKS>& relin_key,
        const hipStream_t stream)
    {
        int first_rns_mod_count = Q_prime_size_;
        int current_rns_mod_count = Q_prime_size_ - input1.depth_;

        int first_decomp_count = Q_size_;
        int current_decomp_count = Q_size_ - input1.depth_;

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = stream};

        gpuntt::GPU_NTT_Inplace(input1.data() +
                                    (current_decomp_count << (n_power + 1)),
                                intt_table_->data(), modulus_->data(), cfg_intt,
                                current_decomp_count, current_decomp_count);

        DeviceVector<Data64> temp_relin(
            (n * Q_size_ * Q_prime_size_) + (2 * n * Q_prime_size_), stream);
        Data64* temp1_relin = temp_relin.data();
        Data64* temp2_relin = temp1_relin + (n * Q_size_ * Q_prime_size_);

        cipher_broadcast_leveled_kernel<<<
            dim3((n >> 8), current_decomp_count, 1), 256, 0, stream>>>(
            input1.data() + (current_decomp_count << (n_power + 1)),
            temp1_relin, modulus_->data(), first_rns_mod_count,
            current_rns_mod_count, n_power);

        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        int counter = first_rns_mod_count;
        int location = 0;
        for (int i = 0; i < input1.depth_; i++)
        {
            location += counter;
            counter--;
        }
        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp1_relin, ntt_table_->data(), modulus_->data(), cfg_ntt,
            current_decomp_count * current_rns_mod_count, current_rns_mod_count,
            new_prime_locations + location);

        // TODO: make it efficient
        int iteration_count_1 = current_decomp_count / 4;
        int iteration_count_2 = current_decomp_count % 4;
        if (relin_key.storage_type_ == storage_type::DEVICE)
        {
            keyswitch_multiply_accumulate_leveled_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp1_relin, relin_key.data(), temp2_relin, modulus_->data(),
                first_rns_mod_count, current_decomp_count, iteration_count_1,
                iteration_count_2, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }
        else
        {
            DeviceVector<Data64> key_location(relin_key.host_location_, stream);
            keyswitch_multiply_accumulate_leveled_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp1_relin, key_location.data(), temp2_relin, modulus_->data(),
                first_rns_mod_count, current_decomp_count, iteration_count_1,
                iteration_count_2, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }

        gpuntt::ntt_rns_configuration<Data64> cfg_intt2 = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data() + first_decomp_count,
            .stream = stream};

        gpuntt::GPU_NTT_Poly_Ordered_Inplace(
            temp2_relin, intt_table_->data() + (first_decomp_count << n_power),
            modulus_->data() + first_decomp_count, cfg_intt2, 2, 1,
            new_input_locations + (input1.depth_ * 2));

        divide_round_lastq_leveled_stage_one_kernel<<<dim3((n >> 8), 2, 1), 256,
                                                      0, stream>>>(
            temp2_relin, temp1_relin, modulus_->data(), half_p_->data(),
            half_mod_->data(), n_power, first_decomp_count,
            current_decomp_count);

        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(temp1_relin, ntt_table_->data(),
                                modulus_->data(), cfg_ntt,
                                2 * current_decomp_count, current_decomp_count);

        divide_round_lastq_leveled_stage_two_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp1_relin, temp2_relin, input1.data(), input1.data(),
            modulus_->data(), last_q_modinv_->data(), n_power,
            current_decomp_count);

        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    __host__ void
    HEOperator<Scheme::CKKS>::relinearize_external_product_method_inplace_ckks(
        Ciphertext<Scheme::CKKS>& input1, Relinkey<Scheme::CKKS>& relin_key,
        const hipStream_t stream)
    {
        int first_rns_mod_count = Q_prime_size_;
        int current_rns_mod_count = Q_prime_size_ - input1.depth_;

        int first_decomp_count = Q_size_;
        int current_decomp_count = Q_size_ - input1.depth_;

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = stream};

        int counter = first_rns_mod_count;
        int location = 0;
        for (int j = 0; j < input1.depth_; j++)
        {
            location += counter;
            counter--;
        }

        DeviceVector<Data64> temp_relin_new(
            (n * d_leveled_->operator[](0) * r_prime_leveled_) +
                (2 * n * d_tilda_leveled_->operator[](0) * r_prime_leveled_) +
                (2 * n * Q_prime_size_),
            stream);
        Data64* temp1_relin_new = temp_relin_new.data();
        Data64* temp2_relin_new =
            temp1_relin_new +
            (n * d_leveled_->operator[](0) * r_prime_leveled_);
        Data64* temp3_relin_new =
            temp2_relin_new +
            (2 * n * d_tilda_leveled_->operator[](0) * r_prime_leveled_);

        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            input1.data() + (current_decomp_count << (n_power + 1)),
            intt_table_->data(), modulus_->data(), cfg_intt,
            current_decomp_count, current_decomp_count,
            prime_location_leveled_->data() + location);

        base_conversion_DtoB_relin_leveled_kernel<<<
            dim3((n >> 8), d_leveled_->operator[](input1.depth_), 1), 256, 0,
            stream>>>(
            input1.data() + (current_decomp_count << (n_power + 1)),
            temp1_relin_new, modulus_->data(), B_prime_leveled_->data(),
            base_change_matrix_D_to_B_leveled_->operator[](input1.depth_)
                .data(),
            Mi_inv_D_to_B_leveled_->operator[](input1.depth_).data(),
            prod_D_to_B_leveled_->operator[](input1.depth_).data(),
            I_j_leveled_->operator[](input1.depth_).data(),
            I_location_leveled_->operator[](input1.depth_).data(), n_power,
            d_tilda_leveled_->operator[](input1.depth_),
            d_leveled_->operator[](input1.depth_), r_prime_leveled_,
            prime_location_leveled_->data() + location);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        gpuntt::GPU_NTT_Inplace(
            temp1_relin_new, B_prime_ntt_tables_leveled_->data(),
            B_prime_leveled_->data(), cfg_ntt,
            d_leveled_->operator[](input1.depth_) * r_prime_leveled_,
            r_prime_leveled_);

        // TODO: make it efficient
        if (relin_key.storage_type_ == storage_type::DEVICE)
        {
            multiply_accumulate_extended_kernel<<<
                dim3((n >> 8), r_prime_leveled_,
                     d_tilda_leveled_->operator[](input1.depth_)),
                256, 0, stream>>>(
                temp1_relin_new, relin_key.data(input1.depth_), temp2_relin_new,
                B_prime_leveled_->data(), n_power,
                d_tilda_leveled_->operator[](input1.depth_),
                d_leveled_->operator[](input1.depth_), r_prime_leveled_);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }
        else
        {
            DeviceVector<Data64> key_location(
                relin_key.host_location_leveled_[input1.depth_], stream);
            multiply_accumulate_extended_kernel<<<
                dim3((n >> 8), r_prime_leveled_,
                     d_tilda_leveled_->operator[](input1.depth_)),
                256, 0, stream>>>(
                temp1_relin_new, key_location.data(), temp2_relin_new,
                B_prime_leveled_->data(), n_power,
                d_tilda_leveled_->operator[](input1.depth_),
                d_leveled_->operator[](input1.depth_), r_prime_leveled_);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }

        gpuntt::ntt_rns_configuration<Data64> cfg_intt2 = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = B_prime_n_inverse_leveled_->data(),
            .stream = stream};

        gpuntt::GPU_NTT_Inplace(
            temp2_relin_new, B_prime_intt_tables_leveled_->data(),
            B_prime_leveled_->data(), cfg_intt2,
            2 * r_prime_leveled_ * d_tilda_leveled_->operator[](input1.depth_),
            r_prime_leveled_);

        base_conversion_BtoD_relin_leveled_kernel<<<
            dim3((n >> 8), d_tilda_leveled_->operator[](input1.depth_), 2), 256,
            0, stream>>>(
            temp2_relin_new, temp3_relin_new, modulus_->data(),
            B_prime_leveled_->data(),
            base_change_matrix_B_to_D_leveled_->operator[](input1.depth_)
                .data(),
            Mi_inv_B_to_D_leveled_->data(),
            prod_B_to_D_leveled_->operator[](input1.depth_).data(),
            I_j_leveled_->operator[](input1.depth_).data(),
            I_location_leveled_->operator[](input1.depth_).data(), n_power,
            current_rns_mod_count, d_tilda_leveled_->operator[](input1.depth_),
            d_leveled_->operator[](input1.depth_), r_prime_leveled_,
            prime_location_leveled_->data() + location);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        divide_round_lastq_extended_leveled_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp3_relin_new, temp2_relin_new, modulus_->data(), half_p_->data(),
            half_mod_->data(), last_q_modinv_->data(), n_power,
            current_rns_mod_count, current_decomp_count, first_rns_mod_count,
            first_decomp_count, P_size_);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(temp2_relin_new, ntt_table_->data(),
                                modulus_->data(), cfg_ntt,
                                2 * current_decomp_count, current_decomp_count);

        addition<<<dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp2_relin_new, input1.data(), input1.data(), modulus_->data(),
            n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    __host__ void
    HEOperator<Scheme::CKKS>::relinearize_external_product_method2_inplace_ckks(
        Ciphertext<Scheme::CKKS>& input1, Relinkey<Scheme::CKKS>& relin_key,
        const hipStream_t stream)
    {
        int first_rns_mod_count = Q_prime_size_;
        int current_rns_mod_count = Q_prime_size_ - input1.depth_;

        int first_decomp_count = Q_size_;
        int current_decomp_count = Q_size_ - input1.depth_;

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = stream};

        int counter = first_rns_mod_count;
        int location = 0;
        for (int j = 0; j < input1.depth_; j++)
        {
            location += counter;
            counter--;
        }

        gpuntt::GPU_NTT_Inplace(input1.data() +
                                    (current_decomp_count << (n_power + 1)),
                                intt_table_->data(), modulus_->data(), cfg_intt,
                                current_decomp_count, current_decomp_count);

        DeviceVector<Data64> temp_relin(
            (n * Q_size_ * Q_prime_size_) + (2 * n * Q_prime_size_), stream);
        Data64* temp1_relin = temp_relin.data();
        Data64* temp2_relin = temp1_relin + (n * Q_size_ * Q_prime_size_);

        base_conversion_DtoQtilde_relin_leveled_kernel<<<
            dim3((n >> 8), d_leveled_->operator[](input1.depth_), 1), 256, 0,
            stream>>>(
            input1.data() + (current_decomp_count << (n_power + 1)),
            temp1_relin, modulus_->data(),
            base_change_matrix_D_to_Qtilda_leveled_->operator[](input1.depth_)
                .data(),
            Mi_inv_D_to_Qtilda_leveled_->operator[](input1.depth_).data(),
            prod_D_to_Qtilda_leveled_->operator[](input1.depth_).data(),
            I_j_leveled_->operator[](input1.depth_).data(),
            I_location_leveled_->operator[](input1.depth_).data(), n_power,
            d_leveled_->operator[](input1.depth_), current_rns_mod_count,
            current_decomp_count, input1.depth_,
            prime_location_leveled_->data() + location);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp1_relin, ntt_table_->data(), modulus_->data(), cfg_ntt,
            d_leveled_->operator[](input1.depth_) * current_rns_mod_count,
            current_rns_mod_count, new_prime_locations + location);

        // TODO: make it efficient
        int iteration_count_1 = d_leveled_->operator[](input1.depth_) / 4;
        int iteration_count_2 = d_leveled_->operator[](input1.depth_) % 4;
        if (relin_key.storage_type_ == storage_type::DEVICE)
        {
            keyswitch_multiply_accumulate_leveled_method_II_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp1_relin, relin_key.data(), temp2_relin, modulus_->data(),
                first_rns_mod_count, current_decomp_count,
                current_rns_mod_count, iteration_count_1, iteration_count_2,
                input1.depth_, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }
        else
        {
            DeviceVector<Data64> key_location(relin_key.host_location_, stream);
            keyswitch_multiply_accumulate_leveled_method_II_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp1_relin, key_location.data(), temp2_relin, modulus_->data(),
                first_rns_mod_count, current_decomp_count,
                current_rns_mod_count, iteration_count_1, iteration_count_2,
                input1.depth_, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp2_relin, intt_table_->data(), modulus_->data(), cfg_intt,
            2 * current_rns_mod_count, current_rns_mod_count,
            new_prime_locations + location);

        divide_round_lastq_extended_leveled_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp2_relin, temp1_relin, modulus_->data(), half_p_->data(),
            half_mod_->data(), last_q_modinv_->data(), n_power,
            current_rns_mod_count, current_decomp_count, first_rns_mod_count,
            first_decomp_count, P_size_);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(temp1_relin, ntt_table_->data(),
                                modulus_->data(), cfg_ntt,
                                2 * current_decomp_count, current_decomp_count);

        addition<<<dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp1_relin, input1.data(), input1.data(), modulus_->data(),
            n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    __host__ void HEOperator<Scheme::CKKS>::rescale_inplace_ckks_leveled(
        Ciphertext<Scheme::CKKS>& input1, const hipStream_t stream)
    {
        int first_decomp_count = Q_size_;
        int current_decomp_count = Q_size_ - input1.depth_;

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data() + (current_decomp_count - 1),
            .stream = stream};

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        // int counter = first_rns_mod_count - 2;
        int counter = first_decomp_count - 1;
        int location = 0;
        for (int i = 0; i < input1.depth_; i++)
        {
            location += counter;
            counter--;
        }

        DeviceVector<Data64> temp_rescale(
            (2 * n * Q_prime_size_) + (2 * n * Q_prime_size_), stream);
        Data64* temp1_rescale = temp_rescale.data();
        Data64* temp2_rescale = temp1_rescale + (2 * n * Q_prime_size_);

        gpuntt::GPU_NTT_Poly_Ordered_Inplace(
            input1.data(),
            intt_table_->data() + ((current_decomp_count - 1) << n_power),
            modulus_->data() + (current_decomp_count - 1), cfg_intt, 2, 1,
            new_input_locations + ((input1.depth_ + P_size_) * 2));

        divide_round_lastq_leveled_stage_one_kernel<<<dim3((n >> 8), 2, 1), 256,
                                                      0, stream>>>(
            input1.data(), temp1_rescale, modulus_->data(),
            rescaled_half_->data() + input1.depth_,
            rescaled_half_mod_->data() + location, n_power,
            current_decomp_count - 1, current_decomp_count - 1);

        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(
            temp1_rescale, ntt_table_->data(), modulus_->data(), cfg_ntt,
            2 * (current_decomp_count - 1), (current_decomp_count - 1));

        move_cipher_leveled_kernel<<<
            dim3((n >> 8), current_decomp_count - 1, 2), 256, 0, stream>>>(
            input1.data(), temp2_rescale, n_power, current_decomp_count - 1);

        divide_round_lastq_rescale_kernel<<<
            dim3((n >> 8), current_decomp_count - 1, 2), 256, 0, stream>>>(
            temp1_rescale, temp2_rescale, input1.data(), modulus_->data(),
            rescaled_last_q_modinv_->data() + location, n_power,
            current_decomp_count - 1);

        HEONGPU_CUDA_CHECK(hipGetLastError());

        if (scheme_ == scheme_type::ckks)
        {
            input1.scale_ = input1.scale_ /
                            static_cast<double>(
                                prime_vector_[current_decomp_count - 1].value);
        }

        input1.depth_++;
    }

    __host__ void HEOperator<Scheme::CKKS>::mod_drop_ckks_leveled_inplace(
        Ciphertext<Scheme::CKKS>& input1, const hipStream_t stream)
    {
        if (input1.depth_ >= (Q_size_ - 1))
        {
            throw std::logic_error("Ciphertext modulus can not be dropped!");
        }

        int current_decomp_count = Q_size_ - input1.depth_;

        int offset1 = current_decomp_count << n_power;
        int offset2 = (current_decomp_count - 1) << n_power;

        DeviceVector<Data64> temp_mod_drop_(n * Q_size_, stream);
        Data64* temp_mod_drop = temp_mod_drop_.data();

        // TODO: do with efficient way!
        global_memory_replace_kernel<<<
            dim3((n >> 8), current_decomp_count - 1, 1), 256, 0, stream>>>(
            input1.data() + offset1, temp_mod_drop, n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        global_memory_replace_kernel<<<
            dim3((n >> 8), current_decomp_count - 1, 1), 256, 0, stream>>>(
            temp_mod_drop, input1.data() + offset2, n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        input1.depth_++;
    }

    __host__ void HEOperator<Scheme::CKKS>::mod_drop_ckks_leveled(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& output,
        const hipStream_t stream)
    {
        if (input1.depth_ >= (Q_size_ - 1))
        {
            throw std::logic_error("Ciphertext modulus can not be dropped!");
        }

        int current_decomp_count = Q_size_ - input1.depth_;
        DeviceVector<Data64> output_memory(
            (current_decomp_count * n * current_decomp_count), stream);

        global_memory_replace_offset_kernel<<<
            dim3((n >> 8), current_decomp_count - 1, 2), 256, 0, stream>>>(
            input1.data(), output_memory.data(), current_decomp_count, n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        output.memory_set(std::move(output_memory));
    }

    __host__ void HEOperator<Scheme::CKKS>::mod_drop_ckks_plaintext(
        Plaintext<Scheme::CKKS>& input1, Plaintext<Scheme::CKKS>& output,
        const hipStream_t stream)
    {
        if (input1.depth_ >= (Q_size_ - 1))
        {
            throw std::logic_error("Plaintext modulus can not be dropped!");
        }

        int current_decomp_count = Q_size_ - input1.depth_;
        DeviceVector<Data64> output_memory(n * (current_decomp_count - 1),
                                           stream);

        global_memory_replace_kernel<<<
            dim3((n >> 8), current_decomp_count - 1, 1), 256, 0, stream>>>(
            input1.data(), output_memory.data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        output.depth_ = input1.depth_ + 1;

        output.memory_set(std::move(output_memory));
    }

    __host__ void HEOperator<Scheme::CKKS>::mod_drop_ckks_plaintext_inplace(
        Plaintext<Scheme::CKKS>& input1, const hipStream_t stream)
    {
        if (input1.depth_ >= (Q_size_ - 1))
        {
            throw std::logic_error("Plaintext modulus can not be dropped!");
        }

        input1.depth_++;
    }

    __host__ void HEOperator<Scheme::CKKS>::rotate_ckks_method_I(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& output,
        Galoiskey<Scheme::CKKS>& galois_key, int shift,
        const hipStream_t stream)
    {
        int galoiselt = steps_to_galois_elt(shift, n, galois_key.group_order_);
        bool key_exist = (galois_key.storage_type_ == storage_type::DEVICE)
                             ? (galois_key.device_location_.find(galoiselt) !=
                                galois_key.device_location_.end())
                             : (galois_key.host_location_.find(galoiselt) !=
                                galois_key.host_location_.end());
        if (key_exist)
        {
            apply_galois_ckks_method_I(input1, output, galois_key, galoiselt,
                                       stream);
        }
        else
        {
            std::vector<int> required_galoiselt;
            int shift_num = abs(shift);
            int negative = (shift < 0) ? (-1) : 1;
            while (shift_num != 0)
            {
                int power = int(log2(shift_num));
                int power_2 = pow(2, power);
                shift_num = shift_num - power_2;

                int index_in = power_2 * negative;

                if (!(galois_key.galois_elt.find(index_in) !=
                      galois_key.galois_elt.end()))
                {
                    throw std::logic_error("Galois key not present!");
                }
                galoiselt = galois_key.galois_elt[index_in];
                required_galoiselt.push_back(galoiselt);
            }

            Ciphertext<Scheme::CKKS>& in_data = input1;
            for (auto& galois_elt : required_galoiselt)
            {
                apply_galois_ckks_method_I(in_data, output, galois_key,
                                           galois_elt, stream);
                in_data = output;
            }
        }
    }

    __host__ void HEOperator<Scheme::CKKS>::rotate_ckks_method_II(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& output,
        Galoiskey<Scheme::CKKS>& galois_key, int shift,
        const hipStream_t stream)
    {
        int galoiselt = steps_to_galois_elt(shift, n, galois_key.group_order_);
        bool key_exist = (galois_key.storage_type_ == storage_type::DEVICE)
                             ? (galois_key.device_location_.find(galoiselt) !=
                                galois_key.device_location_.end())
                             : (galois_key.host_location_.find(galoiselt) !=
                                galois_key.host_location_.end());
        if (key_exist)
        {
            apply_galois_ckks_method_II(input1, output, galois_key, galoiselt,
                                        stream);
        }
        else
        {
            std::vector<int> required_galoiselt;
            int shift_num = abs(shift);
            int negative = (shift < 0) ? (-1) : 1;
            while (shift_num != 0)
            {
                int power = int(log2(shift_num));
                int power_2 = pow(2, power);
                shift_num = shift_num - power_2;

                int index_in = power_2 * negative;

                if (!(galois_key.galois_elt.find(index_in) !=
                      galois_key.galois_elt.end()))
                {
                    throw std::logic_error("Galois key not present!");
                }
                galoiselt = galois_key.galois_elt[index_in];
                required_galoiselt.push_back(galoiselt);
            }

            Ciphertext<Scheme::CKKS>& in_data = input1;
            for (auto& galois_elt : required_galoiselt)
            {
                apply_galois_ckks_method_II(in_data, output, galois_key,
                                            galois_elt, stream);
                in_data = output;
            }
        }
    }

    __host__ void HEOperator<Scheme::CKKS>::apply_galois_ckks_method_I(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& output,
        Galoiskey<Scheme::CKKS>& galois_key, int galois_elt,
        const hipStream_t stream)
    {
        int first_rns_mod_count = Q_prime_size_;
        int current_rns_mod_count = Q_prime_size_ - input1.depth_;

        int first_decomp_count = Q_size_;
        int current_decomp_count = Q_size_ - input1.depth_;

        DeviceVector<Data64> output_memory((2 * n * current_decomp_count),
                                           stream);

        DeviceVector<Data64> temp_rotation(
            (2 * n * Q_size_) + (2 * n * Q_size_) +
                (n * Q_size_ * Q_prime_size_) + (2 * n * Q_prime_size_),
            stream);

        Data64* temp0_rotation = temp_rotation.data();
        Data64* temp1_rotation = temp0_rotation + (2 * n * Q_size_);
        Data64* temp2_rotation = temp1_rotation + (2 * n * Q_size_);
        Data64* temp3_rotation = temp2_rotation + (n * Q_size_ * Q_prime_size_);

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = stream};

        gpuntt::GPU_NTT(input1.data(), temp0_rotation, intt_table_->data(),
                        modulus_->data(), cfg_intt, 2 * current_decomp_count,
                        current_decomp_count);

        // TODO: make it efficient
        ckks_duplicate_kernel<<<dim3((n >> 8), current_decomp_count, 1), 256, 0,
                                stream>>>(
            temp0_rotation, temp2_rotation, modulus_->data(), n_power,
            first_rns_mod_count, current_rns_mod_count, current_decomp_count);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        int counter = first_rns_mod_count;
        int location = 0;
        for (int i = 0; i < input1.depth_; i++)
        {
            location += counter;
            counter--;
        }
        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp2_rotation, ntt_table_->data(), modulus_->data(), cfg_ntt,
            current_decomp_count * current_rns_mod_count, current_rns_mod_count,
            new_prime_locations + location);

        // MultSum
        // TODO: make it efficient
        int iteration_count_1 = current_decomp_count / 4;
        int iteration_count_2 = current_decomp_count % 4;
        if (galois_key.storage_type_ == storage_type::DEVICE)
        {
            keyswitch_multiply_accumulate_leveled_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp2_rotation, galois_key.device_location_[galois_elt].data(),
                temp3_rotation, modulus_->data(), first_rns_mod_count,
                current_decomp_count, iteration_count_1, iteration_count_2,
                n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }
        else
        {
            DeviceVector<Data64> key_location(
                galois_key.host_location_[galois_elt], stream);
            keyswitch_multiply_accumulate_leveled_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp2_rotation, key_location.data(), temp3_rotation,
                modulus_->data(), first_rns_mod_count, current_decomp_count,
                iteration_count_1, iteration_count_2, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp3_rotation, intt_table_->data(), modulus_->data(), cfg_intt,
            2 * current_rns_mod_count, current_rns_mod_count,
            new_prime_locations + location);

        // ModDown + Permute
        divide_round_lastq_permute_ckks_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp3_rotation, temp0_rotation, output_memory.data(),
            modulus_->data(), half_p_->data(), half_mod_->data(),
            last_q_modinv_->data(), galois_elt, n_power, current_rns_mod_count,
            current_decomp_count, first_rns_mod_count, first_decomp_count,
            P_size_);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(output_memory.data(), ntt_table_->data(),
                                modulus_->data(), cfg_ntt,
                                2 * current_decomp_count, current_decomp_count);

        output.memory_set(std::move(output_memory));
    }

    __host__ void HEOperator<Scheme::CKKS>::apply_galois_ckks_method_II(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& output,
        Galoiskey<Scheme::CKKS>& galois_key, int galois_elt,
        const hipStream_t stream)
    {
        int first_rns_mod_count = Q_prime_size_;
        int current_rns_mod_count = Q_prime_size_ - input1.depth_;

        int first_decomp_count = Q_size_;
        int current_decomp_count = Q_size_ - input1.depth_;

        DeviceVector<Data64> output_memory((2 * n * current_decomp_count),
                                           stream);

        DeviceVector<Data64> temp_rotation(
            (2 * n * Q_size_) + (2 * n * Q_size_) + (n * Q_size_) +
                (2 * n * d_leveled_->operator[](0) * Q_prime_size_) +
                (2 * n * Q_prime_size_),
            stream);

        Data64* temp0_rotation = temp_rotation.data();
        Data64* temp1_rotation = temp0_rotation + (2 * n * Q_size_);
        Data64* temp2_rotation = temp1_rotation + (2 * n * Q_size_);
        Data64* temp3_rotation = temp2_rotation + (n * Q_size_);
        Data64* temp4_rotation =
            temp3_rotation +
            (2 * n * d_leveled_->operator[](0) * Q_prime_size_);

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = stream};

        gpuntt::GPU_NTT(input1.data(), temp0_rotation, intt_table_->data(),
                        modulus_->data(), cfg_intt, 2 * current_decomp_count,
                        current_decomp_count);

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        int counter = first_rns_mod_count;
        int location = 0;
        for (int i = 0; i < input1.depth_; i++)
        {
            location += counter;
            counter--;
        }

        base_conversion_DtoQtilde_relin_leveled_kernel<<<
            dim3((n >> 8), d_leveled_->operator[](input1.depth_), 1), 256, 0,
            stream>>>(
            temp0_rotation + (current_decomp_count << n_power), temp3_rotation,
            modulus_->data(),
            base_change_matrix_D_to_Qtilda_leveled_->operator[](input1.depth_)
                .data(),
            Mi_inv_D_to_Qtilda_leveled_->operator[](input1.depth_).data(),
            prod_D_to_Qtilda_leveled_->operator[](input1.depth_).data(),
            I_j_leveled_->operator[](input1.depth_).data(),
            I_location_leveled_->operator[](input1.depth_).data(), n_power,
            d_leveled_->operator[](input1.depth_), current_rns_mod_count,
            current_decomp_count, input1.depth_,
            prime_location_leveled_->data() + location);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp3_rotation, ntt_table_->data(), modulus_->data(), cfg_ntt,
            d_leveled_->operator[](input1.depth_) * current_rns_mod_count,
            current_rns_mod_count, new_prime_locations + location);

        // MultSum
        // TODO: make it efficient
        int iteration_count_1 = d_leveled_->operator[](input1.depth_) / 4;
        int iteration_count_2 = d_leveled_->operator[](input1.depth_) % 4;
        if (galois_key.storage_type_ == storage_type::DEVICE)
        {
            keyswitch_multiply_accumulate_leveled_method_II_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp3_rotation, galois_key.device_location_[galois_elt].data(),
                temp4_rotation, modulus_->data(), first_rns_mod_count,
                current_decomp_count, current_rns_mod_count, iteration_count_1,
                iteration_count_2, input1.depth_, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }
        else
        {
            DeviceVector<Data64> key_location(
                galois_key.host_location_[galois_elt], stream);
            keyswitch_multiply_accumulate_leveled_method_II_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp3_rotation, key_location.data(), temp4_rotation,
                modulus_->data(), first_rns_mod_count, current_decomp_count,
                current_rns_mod_count, iteration_count_1, iteration_count_2,
                input1.depth_, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp4_rotation, intt_table_->data(), modulus_->data(), cfg_intt,
            2 * current_rns_mod_count, current_rns_mod_count,
            new_prime_locations + location);

        // ModDown + Permute
        divide_round_lastq_permute_ckks_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp4_rotation, temp0_rotation, output_memory.data(),
            modulus_->data(), half_p_->data(), half_mod_->data(),
            last_q_modinv_->data(), galois_elt, n_power, current_rns_mod_count,
            current_decomp_count, first_rns_mod_count, first_decomp_count,
            P_size_);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(output_memory.data(), ntt_table_->data(),
                                modulus_->data(), cfg_ntt,
                                2 * current_decomp_count, current_decomp_count);

        output.memory_set(std::move(output_memory));
    }

    __host__ void HEOperator<Scheme::CKKS>::switchkey_ckks_method_I(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& output,
        Switchkey<Scheme::CKKS>& switch_key, const hipStream_t stream)
    {
        int first_rns_mod_count = Q_prime_size_;
        int current_rns_mod_count = Q_prime_size_ - input1.depth_;

        int first_decomp_count = Q_size_;
        int current_decomp_count = Q_size_ - input1.depth_;

        DeviceVector<Data64> output_memory((2 * n * current_decomp_count),
                                           stream);

        DeviceVector<Data64> temp_rotation(
            (2 * n * Q_size_) + (2 * n * Q_size_) +
                (n * Q_size_ * Q_prime_size_) + (2 * n * Q_prime_size_),
            stream);

        Data64* temp0_rotation = temp_rotation.data();
        Data64* temp1_rotation = temp0_rotation + (2 * n * Q_size_);
        Data64* temp2_rotation = temp1_rotation + (2 * n * Q_size_);
        Data64* temp3_rotation = temp2_rotation + (n * Q_size_ * Q_prime_size_);

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = stream};

        gpuntt::GPU_NTT(input1.data(), temp0_rotation, intt_table_->data(),
                        modulus_->data(), cfg_intt, 2 * current_decomp_count,
                        current_decomp_count);

        cipher_broadcast_switchkey_leveled_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp0_rotation, temp1_rotation, temp2_rotation, modulus_->data(),
            n_power, first_rns_mod_count, current_rns_mod_count,
            current_decomp_count);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        int counter = first_rns_mod_count;
        int location = 0;
        for (int i = 0; i < input1.depth_; i++)
        {
            location += counter;
            counter--;
        }
        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp2_rotation, ntt_table_->data(), modulus_->data(), cfg_ntt,
            current_decomp_count * current_rns_mod_count, current_rns_mod_count,
            new_prime_locations + location);

        // TODO: make it efficient
        int iteration_count_1 = current_decomp_count / 4;
        int iteration_count_2 = current_decomp_count % 4;
        if (switch_key.storage_type_ == storage_type::DEVICE)
        {
            keyswitch_multiply_accumulate_leveled_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp2_rotation, switch_key.data(), temp3_rotation,
                modulus_->data(), first_rns_mod_count, current_decomp_count,
                iteration_count_1, iteration_count_2, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }
        else
        {
            DeviceVector<Data64> key_location(switch_key.host_location_,
                                              stream);
            keyswitch_multiply_accumulate_leveled_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp2_rotation, key_location.data(), temp3_rotation,
                modulus_->data(), first_rns_mod_count, current_decomp_count,
                iteration_count_1, iteration_count_2, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }

        gpuntt::ntt_rns_configuration<Data64> cfg_intt2 = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data() + first_decomp_count,
            .stream = stream};

        gpuntt::GPU_NTT_Poly_Ordered_Inplace(
            temp3_rotation,
            intt_table_->data() + (first_decomp_count << n_power),
            modulus_->data() + first_decomp_count, cfg_intt2, 2, 1,
            new_input_locations + (input1.depth_ * 2));

        divide_round_lastq_leveled_stage_one_kernel<<<dim3((n >> 8), 2, 1), 256,
                                                      0, stream>>>(
            temp3_rotation, temp2_rotation, modulus_->data(), half_p_->data(),
            half_mod_->data(), n_power, first_decomp_count,
            current_decomp_count);

        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(temp2_rotation, ntt_table_->data(),
                                modulus_->data(), cfg_ntt,
                                2 * current_decomp_count, current_decomp_count);

        // TODO: Merge with previous one
        gpuntt::GPU_NTT_Inplace(temp1_rotation, ntt_table_->data(),
                                modulus_->data(), cfg_ntt, current_decomp_count,
                                current_decomp_count);

        divide_round_lastq_leveled_stage_two_switchkey_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp2_rotation, temp3_rotation, temp1_rotation,
            output_memory.data(), modulus_->data(), last_q_modinv_->data(),
            n_power, current_decomp_count);

        HEONGPU_CUDA_CHECK(hipGetLastError());

        output.memory_set(std::move(output_memory));
    }

    __host__ void HEOperator<Scheme::CKKS>::switchkey_ckks_method_II(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& output,
        Switchkey<Scheme::CKKS>& switch_key, const hipStream_t stream)
    {
        int first_rns_mod_count = Q_prime_size_;
        int current_rns_mod_count = Q_prime_size_ - input1.depth_;

        int first_decomp_count = Q_size_;
        int current_decomp_count = Q_size_ - input1.depth_;

        DeviceVector<Data64> output_memory((2 * n * current_decomp_count),
                                           stream);

        DeviceVector<Data64> temp_rotation(
            (2 * n * Q_size_) + (2 * n * Q_size_) + (n * Q_size_) +
                (2 * n * d_leveled_->operator[](0) * Q_prime_size_) +
                (2 * n * Q_prime_size_),
            stream);

        Data64* temp0_rotation = temp_rotation.data();
        Data64* temp1_rotation = temp0_rotation + (2 * n * Q_size_);
        Data64* temp2_rotation = temp1_rotation + (2 * n * Q_size_);
        Data64* temp3_rotation = temp2_rotation + (n * Q_size_);
        Data64* temp4_rotation =
            temp3_rotation +
            (2 * n * d_leveled_->operator[](0) * Q_prime_size_);

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = stream};

        gpuntt::GPU_NTT(input1.data(), temp0_rotation, intt_table_->data(),
                        modulus_->data(), cfg_intt, 2 * current_decomp_count,
                        current_decomp_count);

        cipher_broadcast_switchkey_method_II_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp0_rotation, temp1_rotation, temp2_rotation, modulus_->data(),
            n_power, current_decomp_count);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        int counter = first_rns_mod_count;
        int location = 0;
        for (int i = 0; i < input1.depth_; i++)
        {
            location += counter;
            counter--;
        }

        base_conversion_DtoQtilde_relin_leveled_kernel<<<
            dim3((n >> 8), d_leveled_->operator[](input1.depth_), 1), 256, 0,
            stream>>>(
            temp2_rotation, temp3_rotation, modulus_->data(),
            base_change_matrix_D_to_Qtilda_leveled_->operator[](input1.depth_)
                .data(),
            Mi_inv_D_to_Qtilda_leveled_->operator[](input1.depth_).data(),
            prod_D_to_Qtilda_leveled_->operator[](input1.depth_).data(),
            I_j_leveled_->operator[](input1.depth_).data(),
            I_location_leveled_->operator[](input1.depth_).data(), n_power,
            d_leveled_->operator[](input1.depth_), current_rns_mod_count,
            current_decomp_count, input1.depth_,
            prime_location_leveled_->data() + location);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp3_rotation, ntt_table_->data(), modulus_->data(), cfg_ntt,
            d_leveled_->operator[](input1.depth_) * current_rns_mod_count,
            current_rns_mod_count, new_prime_locations + location);

        // TODO: make it efficient
        int iteration_count_1 = d_leveled_->operator[](input1.depth_) / 4;
        int iteration_count_2 = d_leveled_->operator[](input1.depth_) % 4;
        if (switch_key.storage_type_ == storage_type::DEVICE)
        {
            keyswitch_multiply_accumulate_leveled_method_II_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp3_rotation, switch_key.data(), temp4_rotation,
                modulus_->data(), first_rns_mod_count, current_decomp_count,
                current_rns_mod_count, iteration_count_1, iteration_count_2,
                input1.depth_, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }
        else
        {
            DeviceVector<Data64> key_location(switch_key.host_location_,
                                              stream);
            keyswitch_multiply_accumulate_leveled_method_II_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp3_rotation, key_location.data(), temp4_rotation,
                modulus_->data(), first_rns_mod_count, current_decomp_count,
                current_rns_mod_count, iteration_count_1, iteration_count_2,
                input1.depth_, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp4_rotation, intt_table_->data(), modulus_->data(), cfg_intt,
            2 * current_rns_mod_count, current_rns_mod_count,
            new_prime_locations + location);

        divide_round_lastq_extended_leveled_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp4_rotation, temp3_rotation, modulus_->data(), half_p_->data(),
            half_mod_->data(), last_q_modinv_->data(), n_power,
            current_rns_mod_count, current_decomp_count, first_rns_mod_count,
            first_decomp_count, P_size_);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(temp3_rotation, ntt_table_->data(),
                                modulus_->data(), cfg_ntt,
                                2 * current_decomp_count, current_decomp_count);

        // TODO: Fused the redundant kernels
        // TODO: Merge with previous one
        gpuntt::GPU_NTT_Inplace(temp1_rotation, ntt_table_->data(),
                                modulus_->data(), cfg_ntt, current_decomp_count,
                                current_decomp_count);

        addition_switchkey<<<dim3((n >> 8), current_decomp_count, 2), 256, 0,
                             stream>>>(temp3_rotation, temp1_rotation,
                                       output_memory.data(), modulus_->data(),
                                       n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        output.memory_set(std::move(output_memory));
    }

    __host__ void HEOperator<Scheme::CKKS>::conjugate_ckks_method_I(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& output,
        Galoiskey<Scheme::CKKS>& conjugate_key, const hipStream_t stream)
    {
        int first_rns_mod_count = Q_prime_size_;
        int current_rns_mod_count = Q_prime_size_ - input1.depth_;

        int first_decomp_count = Q_size_;
        int current_decomp_count = Q_size_ - input1.depth_;

        DeviceVector<Data64> output_memory((2 * n * current_decomp_count),
                                           stream);

        int galois_elt = conjugate_key.galois_elt_zero;

        DeviceVector<Data64> temp_rotation(
            (2 * n * Q_size_) + (2 * n * Q_size_) +
                (n * Q_size_ * Q_prime_size_) + (2 * n * Q_prime_size_),
            stream);

        Data64* temp0_rotation = temp_rotation.data();
        Data64* temp1_rotation = temp0_rotation + (2 * n * Q_size_);
        Data64* temp2_rotation = temp1_rotation + (2 * n * Q_size_);
        Data64* temp3_rotation = temp2_rotation + (n * Q_size_ * Q_prime_size_);

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = stream};

        gpuntt::GPU_NTT(input1.data(), temp0_rotation, intt_table_->data(),
                        modulus_->data(), cfg_intt, 2 * current_decomp_count,
                        current_decomp_count);

        // TODO: make it efficient
        ckks_duplicate_kernel<<<dim3((n >> 8), current_decomp_count, 1), 256, 0,
                                stream>>>(
            temp0_rotation, temp2_rotation, modulus_->data(), n_power,
            first_rns_mod_count, current_rns_mod_count, current_decomp_count);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        int counter = first_rns_mod_count;
        int location = 0;
        for (int i = 0; i < input1.depth_; i++)
        {
            location += counter;
            counter--;
        }
        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp2_rotation, ntt_table_->data(), modulus_->data(), cfg_ntt,
            current_decomp_count * current_rns_mod_count, current_rns_mod_count,
            new_prime_locations + location);

        // MultSum
        // TODO: make it efficient
        int iteration_count_1 = current_decomp_count / 4;
        int iteration_count_2 = current_decomp_count % 4;
        if (conjugate_key.storage_type_ == storage_type::DEVICE)
        {
            keyswitch_multiply_accumulate_leveled_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp2_rotation, conjugate_key.c_data(), temp3_rotation,
                modulus_->data(), first_rns_mod_count, current_decomp_count,
                iteration_count_1, iteration_count_2, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }
        else
        {
            DeviceVector<Data64> key_location(conjugate_key.zero_host_location_,
                                              stream);
            keyswitch_multiply_accumulate_leveled_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp2_rotation, key_location.data(), temp3_rotation,
                modulus_->data(), first_rns_mod_count, current_decomp_count,
                iteration_count_1, iteration_count_2, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp3_rotation, intt_table_->data(), modulus_->data(), cfg_intt,
            2 * current_rns_mod_count, current_rns_mod_count,
            new_prime_locations + location);

        // ModDown + Permute
        divide_round_lastq_permute_ckks_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp3_rotation, temp0_rotation, output_memory.data(),
            modulus_->data(), half_p_->data(), half_mod_->data(),
            last_q_modinv_->data(), galois_elt, n_power, current_rns_mod_count,
            current_decomp_count, first_rns_mod_count, first_decomp_count,
            P_size_);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(output_memory.data(), ntt_table_->data(),
                                modulus_->data(), cfg_ntt,
                                2 * current_decomp_count, current_decomp_count);

        output.memory_set(std::move(output_memory));
    }

    __host__ void HEOperator<Scheme::CKKS>::conjugate_ckks_method_II(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& output,
        Galoiskey<Scheme::CKKS>& conjugate_key, const hipStream_t stream)
    {
        int first_rns_mod_count = Q_prime_size_;
        int current_rns_mod_count = Q_prime_size_ - input1.depth_;

        int first_decomp_count = Q_size_;
        int current_decomp_count = Q_size_ - input1.depth_;

        DeviceVector<Data64> output_memory((2 * n * current_decomp_count),
                                           stream);

        int galois_elt = conjugate_key.galois_elt_zero;

        DeviceVector<Data64> temp_rotation(
            (2 * n * Q_size_) + (2 * n * Q_size_) + (n * Q_size_) +
                (2 * n * d_leveled_->operator[](0) * Q_prime_size_) +
                (2 * n * Q_prime_size_),
            stream);

        Data64* temp0_rotation = temp_rotation.data();
        Data64* temp1_rotation = temp0_rotation + (2 * n * Q_size_);
        Data64* temp2_rotation = temp1_rotation + (2 * n * Q_size_);
        Data64* temp3_rotation = temp2_rotation + (n * Q_size_);
        Data64* temp4_rotation =
            temp3_rotation +
            (2 * n * d_leveled_->operator[](0) * Q_prime_size_);

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = stream};

        gpuntt::GPU_NTT(input1.data(), temp0_rotation, intt_table_->data(),
                        modulus_->data(), cfg_intt, 2 * current_decomp_count,
                        current_decomp_count);

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        int counter = first_rns_mod_count;
        int location = 0;
        for (int i = 0; i < input1.depth_; i++)
        {
            location += counter;
            counter--;
        }

        base_conversion_DtoQtilde_relin_leveled_kernel<<<
            dim3((n >> 8), d_leveled_->operator[](input1.depth_), 1), 256, 0,
            stream>>>(
            temp0_rotation + (current_decomp_count << n_power), temp3_rotation,
            modulus_->data(),
            base_change_matrix_D_to_Qtilda_leveled_->operator[](input1.depth_)
                .data(),
            Mi_inv_D_to_Qtilda_leveled_->operator[](input1.depth_).data(),
            prod_D_to_Qtilda_leveled_->operator[](input1.depth_).data(),
            I_j_leveled_->operator[](input1.depth_).data(),
            I_location_leveled_->operator[](input1.depth_).data(), n_power,
            d_leveled_->operator[](input1.depth_), current_rns_mod_count,
            current_decomp_count, input1.depth_,
            prime_location_leveled_->data() + location);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp3_rotation, ntt_table_->data(), modulus_->data(), cfg_ntt,
            d_leveled_->operator[](input1.depth_) * current_rns_mod_count,
            current_rns_mod_count, new_prime_locations + location);

        // MultSum
        // TODO: make it efficient
        int iteration_count_1 = d_leveled_->operator[](input1.depth_) / 4;
        int iteration_count_2 = d_leveled_->operator[](input1.depth_) % 4;
        if (conjugate_key.storage_type_ == storage_type::DEVICE)
        {
            keyswitch_multiply_accumulate_leveled_method_II_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp3_rotation, conjugate_key.c_data(), temp4_rotation,
                modulus_->data(), first_rns_mod_count, current_decomp_count,
                current_rns_mod_count, iteration_count_1, iteration_count_2,
                input1.depth_, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }
        else
        {
            DeviceVector<Data64> key_location(conjugate_key.zero_host_location_,
                                              stream);
            keyswitch_multiply_accumulate_leveled_method_II_kernel<<<
                dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                temp3_rotation, key_location.data(), temp4_rotation,
                modulus_->data(), first_rns_mod_count, current_decomp_count,
                current_rns_mod_count, iteration_count_1, iteration_count_2,
                input1.depth_, n_power);
            HEONGPU_CUDA_CHECK(hipGetLastError());
        }

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp4_rotation, intt_table_->data(), modulus_->data(), cfg_intt,
            2 * current_rns_mod_count, current_rns_mod_count,
            new_prime_locations + location);

        // ModDown + Permute
        divide_round_lastq_permute_ckks_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
            temp4_rotation, temp0_rotation, output_memory.data(),
            modulus_->data(), half_p_->data(), half_mod_->data(),
            last_q_modinv_->data(), galois_elt, n_power, current_rns_mod_count,
            current_decomp_count, first_rns_mod_count, first_decomp_count,
            P_size_);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(output_memory.data(), ntt_table_->data(),
                                modulus_->data(), cfg_ntt,
                                2 * current_decomp_count, current_decomp_count);

        output.memory_set(std::move(output_memory));
    }

    ////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////
    //                       BOOTSRAPPING                         //
    ////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////

    __host__ Plaintext<Scheme::CKKS>
    HEOperator<Scheme::CKKS>::operator_plaintext(hipStream_t stream)
    {
        Plaintext<Scheme::CKKS> plain;

        plain.scheme_ = scheme_;
        plain.plain_size_ = n * Q_size_; // n
        plain.depth_ = 0;
        plain.scale_ = 0;
        plain.in_ntt_domain_ = true;

        plain.device_locations_ =
            DeviceVector<Data64>(plain.plain_size_, stream);

        return plain;
    }

    __host__ Plaintext<Scheme::CKKS>
    HEOperator<Scheme::CKKS>::operator_from_plaintext(
        Plaintext<Scheme::CKKS>& input, hipStream_t stream)
    {
        Plaintext<Scheme::CKKS> plain;

        plain.scheme_ = input.scheme_;
        plain.plain_size_ = input.plain_size_;
        plain.depth_ = input.depth_;
        plain.scale_ = input.scale_;
        plain.in_ntt_domain_ = input.in_ntt_domain_;

        plain.storage_type_ = storage_type::DEVICE;
        plain.device_locations_ =
            DeviceVector<Data64>(plain.plain_size_, stream);

        return plain;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HEOperator<Scheme::CKKS>::operator_ciphertext(double scale,
                                                  hipStream_t stream)
    {
        Ciphertext<Scheme::CKKS> cipher;

        cipher.coeff_modulus_count_ = Q_size_;
        cipher.cipher_size_ = 2; // default
        cipher.ring_size_ = n; // n
        cipher.depth_ = 0;

        cipher.scheme_ = scheme_;
        cipher.in_ntt_domain_ = true;
        cipher.storage_type_ = storage_type::DEVICE;

        cipher.rescale_required_ = false;
        cipher.relinearization_required_ = false;
        cipher.scale_ = scale;
        cipher.ciphertext_generated_ = true;

        int cipher_memory_size = 2 * (Q_size_ - cipher.depth_) * n;

        cipher.device_locations_ =
            DeviceVector<Data64>(cipher_memory_size, stream);

        return cipher;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HEOperator<Scheme::CKKS>::operator_from_ciphertext(
        Ciphertext<Scheme::CKKS>& input, hipStream_t stream)
    {
        Ciphertext<Scheme::CKKS> cipher;

        cipher.coeff_modulus_count_ = input.coeff_modulus_count_;
        cipher.cipher_size_ = input.cipher_size_;
        cipher.ring_size_ = input.ring_size_;
        cipher.depth_ = input.depth_;

        cipher.scheme_ = input.scheme_;
        cipher.in_ntt_domain_ = input.in_ntt_domain_;

        cipher.storage_type_ = storage_type::DEVICE;

        cipher.rescale_required_ = input.rescale_required_;
        cipher.relinearization_required_ = input.relinearization_required_;
        cipher.scale_ = input.scale_;
        cipher.ciphertext_generated_ = true;

        int cipher_memory_size = 2 * (Q_size_ - cipher.depth_) * n;

        cipher.device_locations_ =
            DeviceVector<Data64>(cipher_memory_size, stream);

        return cipher;
    }

    __host__ void HEOperator<Scheme::CKKS>::quick_ckks_encoder_vec_complex(
        Complex64* input, Data64* output, const double scale,
        bool use_all_bases)
    {
        int rns_count = use_all_bases ? Q_prime_size_ : Q_size_;

        double fix = scale / static_cast<double>(slot_count_);

        gpufft::fft_configuration<Float64> cfg_ifft{};
        cfg_ifft.n_power = log_slot_count_;
        cfg_ifft.fft_type = gpufft::type::INVERSE;
        cfg_ifft.mod_inverse = Complex64(fix, 0.0);
        cfg_ifft.stream = 0;

        gpufft::GPU_Special_FFT(input, special_ifft_roots_table_->data(),
                                cfg_ifft, 1);

        encode_kernel_ckks_conversion<<<dim3(((slot_count_) >> 8), 1, 1),
                                        256>>>(output, input, modulus_->data(),
                                               rns_count, two_pow_64_,
                                               reverse_order_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = 0};

        gpuntt::GPU_NTT_Inplace(output, ntt_table_->data(), modulus_->data(),
                                cfg_ntt, rns_count, rns_count);
    }

    __host__ void HEOperator<Scheme::CKKS>::quick_ckks_encoder_constant_complex(
        Complex64 input, Data64* output, const double scale, bool use_all_bases)
    {
        // std::vector<Complex64> in = {input};
        std::vector<Complex64> in;
        for (int i = 0; i < slot_count_; i++)
        {
            in.push_back(input);
        }
        DeviceVector<Complex64> message_gpu(slot_count_);
        hipMemcpy(message_gpu.data(), in.data(), in.size() * sizeof(Complex64),
                   hipMemcpyHostToDevice);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        double fix = scale / static_cast<double>(slot_count_);

        gpufft::fft_configuration<Float64> cfg_ifft{};
        cfg_ifft.n_power = log_slot_count_;
        cfg_ifft.fft_type = gpufft::type::INVERSE;
        cfg_ifft.mod_inverse = Complex64(fix, 0.0);
        cfg_ifft.stream = 0;

        gpufft::GPU_Special_FFT(message_gpu.data(),
                                special_ifft_roots_table_->data(), cfg_ifft, 1);

        encode_kernel_ckks_conversion<<<dim3(((slot_count_) >> 8), 1, 1),
                                        256>>>(
            output, message_gpu.data(), modulus_->data(), Q_size_, two_pow_64_,
            reverse_order_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = 0};

        gpuntt::GPU_NTT_Inplace(output, ntt_table_->data(), modulus_->data(),
                                cfg_ntt, Q_size_, Q_size_);
    }

    __host__ void HEOperator<Scheme::CKKS>::quick_ckks_encoder_constant_double(
        double input, Data64* output, const double scale, bool use_all_bases)
    {
        double value = input * scale;

        encode_kernel_double_ckks_conversion<<<dim3((n >> 8), 1, 1), 256>>>(
            output, value, modulus_->data(), Q_size_, two_pow_64_, n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    __host__ void HEOperator<Scheme::CKKS>::quick_ckks_encoder_constant_integer(
        std::int64_t input, Data64* output, const double scale,
        bool use_all_bases)
    {
        double value = static_cast<double>(input) * scale;

        encode_kernel_double_ckks_conversion<<<dim3((n >> 8), 1, 1), 256>>>(
            output, value, modulus_->data(), Q_size_, two_pow_64_, n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    __host__ std::vector<heongpu::DeviceVector<Data64>>
    HEOperator<Scheme::CKKS>::encode_V_matrixs(Vandermonde& vandermonde,
                                               const double scale,
                                               bool use_all_bases)
    {
        std::vector<heongpu::DeviceVector<Data64>> result;

        int rns_count = use_all_bases ? Q_prime_size_ : Q_size_;

        for (int m = 0; m < vandermonde.StoC_piece_; m++)
        {
            heongpu::DeviceVector<Data64> temp_encoded(
                (vandermonde.V_matrixs_index_[m].size() * rns_count)
                << (vandermonde.log_num_slots_ + 1));

            for (int i = 0; i < vandermonde.V_matrixs_index_[m].size(); i++)
            {
                int matrix_location = (i << vandermonde.log_num_slots_);
                int plaintext_location =
                    ((i * rns_count) << (vandermonde.log_num_slots_ + 1));

                quick_ckks_encoder_vec_complex(
                    vandermonde.V_matrixs_rotated_[m].data() + matrix_location,
                    temp_encoded.data() + plaintext_location, scale,
                    use_all_bases);
            }

            result.push_back(std::move(temp_encoded));
        }

        return result;
    }

    __host__ std::vector<heongpu::DeviceVector<Data64>>
    HEOperator<Scheme::CKKS>::encode_V_inv_matrixs(Vandermonde& vandermonde,
                                                   const double scale,
                                                   bool use_all_bases)
    {
        std::vector<heongpu::DeviceVector<Data64>> result;

        int rns_count = use_all_bases ? Q_prime_size_ : Q_size_;

        for (int m = 0; m < vandermonde.CtoS_piece_; m++)
        {
            heongpu::DeviceVector<Data64> temp_encoded(
                (vandermonde.V_inv_matrixs_index_[m].size() * rns_count)
                << (vandermonde.log_num_slots_ + 1));

            for (int i = 0; i < vandermonde.V_inv_matrixs_index_[m].size(); i++)
            {
                int matrix_location = (i << vandermonde.log_num_slots_);
                int plaintext_location =
                    ((i * rns_count) << (vandermonde.log_num_slots_ + 1));

                quick_ckks_encoder_vec_complex(
                    vandermonde.V_inv_matrixs_rotated_[m].data() +
                        matrix_location,
                    temp_encoded.data() + plaintext_location, scale,
                    use_all_bases);
            }

            result.push_back(std::move(temp_encoded));
        }

        return result;
    }

    __host__ Ciphertext<Scheme::CKKS> HEOperator<Scheme::CKKS>::multiply_matrix(
        Ciphertext<Scheme::CKKS>& cipher,
        std::vector<heongpu::DeviceVector<Data64>>& matrix,
        std::vector<std::vector<std::vector<int>>>& diags_matrices_bsgs_,
        Galoiskey<Scheme::CKKS>& galois_key, const ExecutionOptions& options)
    {
        hipStream_t old_stream = cipher.stream();
        cipher.switch_stream(
            options.stream_); // TODO: Change copy and assign structure!
        Ciphertext<Scheme::CKKS> result;
        result = cipher;
        cipher.switch_stream(
            old_stream); // TODO: Change copy and assign structure!

        int matrix_count = diags_matrices_bsgs_.size();
        for (int m = (matrix_count - 1); - 1 < m; m--)
        {
            int n1 = diags_matrices_bsgs_[m][0].size();
            int current_level = result.depth_;
            int current_decomp_count = (Q_size_ - current_level);

            DeviceVector<Data64> rotated_result =
                fast_single_hoisting_rotation_ckks(
                    result, diags_matrices_bsgs_[m][0], n1, galois_key,
                    options.stream_);

            int counter = 0;
            for (int j = 0; j < diags_matrices_bsgs_[m].size(); j++)
            {
                int real_shift = diags_matrices_bsgs_[m][j][0];

                Ciphertext<Scheme::CKKS> inner_sum =
                    operator_ciphertext(0, options.stream_);

                int matrix_plaintext_location = (counter * Q_size_) << n_power;
                int inner_n1 = diags_matrices_bsgs_[m][j].size();

                cipherplain_multiply_accumulate_kernel<<<
                    dim3((n >> 8), current_decomp_count, 2), 256, 0,
                    options.stream_>>>(
                    rotated_result.data(),
                    matrix[m].data() + matrix_plaintext_location,
                    inner_sum.data(), modulus_->data(), inner_n1,
                    current_decomp_count, Q_size_, n_power);
                HEONGPU_CUDA_CHECK(hipGetLastError());

                counter = counter + inner_n1;

                inner_sum.scheme_ = scheme_;
                inner_sum.ring_size_ = n;
                inner_sum.coeff_modulus_count_ = Q_size_;
                inner_sum.cipher_size_ = 2;
                inner_sum.depth_ = result.depth_;
                inner_sum.scale_ = result.scale_;
                inner_sum.in_ntt_domain_ = result.in_ntt_domain_;
                inner_sum.rescale_required_ = result.rescale_required_;
                inner_sum.relinearization_required_ =
                    result.relinearization_required_;
                inner_sum.ciphertext_generated_ = true;

                rotate_rows_inplace(inner_sum, galois_key, real_shift, options);

                if (j == 0)
                {
                    hipStream_t old_stream2 = inner_sum.stream();
                    inner_sum.switch_stream(
                        options.stream_); // TODO: Change copy and assign
                                          // structure!
                    result = inner_sum;
                    inner_sum.switch_stream(
                        old_stream2); // TODO: Change copy and assign structure!
                }
                else
                {
                    add(result, inner_sum, result, options);
                }
            }

            result.scale_ = result.scale_ * scale_boot_;
            result.rescale_required_ = true;
            rescale_inplace(result, options);
        }

        return result;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HEOperator<Scheme::CKKS>::multiply_matrix_less_memory(
        Ciphertext<Scheme::CKKS>& cipher,
        std::vector<heongpu::DeviceVector<Data64>>& matrix,
        std::vector<std::vector<std::vector<int>>>& diags_matrices_bsgs_,
        std::vector<std::vector<std::vector<int>>>& real_shift,
        Galoiskey<Scheme::CKKS>& galois_key, const ExecutionOptions& options)
    {
        hipStream_t old_stream = cipher.stream();
        cipher.switch_stream(
            options.stream_); // TODO: Change copy and assign structure!
        Ciphertext<Scheme::CKKS> result;
        result = cipher;
        cipher.switch_stream(
            old_stream); // TODO: Change copy and assign structure!

        int matrix_count = diags_matrices_bsgs_.size();
        for (int m = (matrix_count - 1); - 1 < m; m--)
        {
            int n1 = diags_matrices_bsgs_[m][0].size();
            int current_level = result.depth_;
            int current_decomp_count = (Q_size_ - current_level);

            DeviceVector<Data64> rotated_result =
                fast_single_hoisting_rotation_ckks(
                    result, diags_matrices_bsgs_[m][0], n1, galois_key,
                    options.stream_);

            int counter = 0;
            for (int j = 0; j < diags_matrices_bsgs_[m].size(); j++)
            {
                Ciphertext<Scheme::CKKS> inner_sum =
                    operator_ciphertext(0, options.stream_);

                int matrix_plaintext_location = (counter * Q_size_) << n_power;
                int inner_n1 = diags_matrices_bsgs_[m][j].size();

                cipherplain_multiply_accumulate_kernel<<<
                    dim3((n >> 8), current_decomp_count, 2), 256, 0,
                    options.stream_>>>(
                    rotated_result.data(),
                    matrix[m].data() + matrix_plaintext_location,
                    inner_sum.data(), modulus_->data(), inner_n1,
                    current_decomp_count, Q_size_, n_power);
                HEONGPU_CUDA_CHECK(hipGetLastError());

                counter = counter + inner_n1;

                inner_sum.scheme_ = scheme_;
                inner_sum.ring_size_ = n;
                inner_sum.coeff_modulus_count_ = Q_size_;
                inner_sum.cipher_size_ = 2;
                inner_sum.depth_ = result.depth_;
                inner_sum.scale_ = result.scale_;
                inner_sum.in_ntt_domain_ = result.in_ntt_domain_;
                inner_sum.storage_type_ = result.storage_type_;
                inner_sum.rescale_required_ = result.rescale_required_;
                inner_sum.relinearization_required_ =
                    result.relinearization_required_;
                inner_sum.ciphertext_generated_ = true;

                int real_shift_size = real_shift[m][j].size();
                for (int ss = 0; ss < real_shift_size; ss++)
                {
                    int shift_amount = real_shift[m][j][ss];
                    rotate_rows_inplace(inner_sum, galois_key, shift_amount,
                                        options);
                }

                if (j == 0)
                {
                    hipStream_t old_stream2 = inner_sum.stream();
                    inner_sum.switch_stream(
                        options.stream_); // TODO: Change copy and assign
                                          // structure!
                    result = inner_sum;
                    inner_sum.switch_stream(
                        old_stream2); // TODO: Change copy and assign structure!
                }
                else
                {
                    add(result, inner_sum, result, options);
                }
            }
            result.scale_ = result.scale_ * scale_boot_;
            result.rescale_required_ = true;
            rescale_inplace(result, options);
        }

        return result;
    }

    __host__ std::vector<Ciphertext<Scheme::CKKS>>
    HEOperator<Scheme::CKKS>::coeff_to_slot(Ciphertext<Scheme::CKKS>& cipher,
                                            Galoiskey<Scheme::CKKS>& galois_key,
                                            const ExecutionOptions& options)
    {
        Ciphertext<Scheme::CKKS> c1;
        if (less_key_mode_)
        {
            c1 = multiply_matrix_less_memory(
                cipher, V_inv_matrixs_rotated_encoded_,
                diags_matrices_inv_bsgs_, real_shift_n2_inv_bsgs_, galois_key,
                options);
        }
        else
        {
            c1 = multiply_matrix(cipher, V_inv_matrixs_rotated_encoded_,
                                 diags_matrices_inv_bsgs_, galois_key, options);
        }

        Ciphertext<Scheme::CKKS> c2 = operator_ciphertext(0, options.stream_);
        conjugate(c1, c2, galois_key, options); // conjugate

        Ciphertext<Scheme::CKKS> result0 =
            operator_ciphertext(0, options.stream_);
        add(c1, c2, result0, options);

        int current_decomp_count = Q_size_ - result0.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            result0.data(), encoded_constant_1over2_.data(), result0.data(),
            modulus_->data(), n_power);
        result0.scale_ = result0.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        result0.rescale_required_ = true;
        rescale_inplace(result0, options);

        Ciphertext<Scheme::CKKS> result1 =
            operator_ciphertext(0, options.stream_);
        sub(c1, c2, result1, options);

        current_decomp_count = Q_size_ - result1.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            result1.data(), encoded_complex_minus_iover2_.data(),
            result1.data(), modulus_->data(), n_power);
        result1.scale_ = result1.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        result1.rescale_required_ = true;
        rescale_inplace(result1, options);

        std::vector<Ciphertext<Scheme::CKKS>> result;
        result.push_back(std::move(result0));
        result.push_back(std::move(result1));

        return result;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HEOperator<Scheme::CKKS>::solo_coeff_to_slot(
        Ciphertext<Scheme::CKKS>& cipher, Galoiskey<Scheme::CKKS>& galois_key,
        const ExecutionOptions& options)
    {
        Ciphertext<Scheme::CKKS> c1;
        if (less_key_mode_)
        {
            c1 = multiply_matrix_less_memory(
                cipher, V_inv_matrixs_rotated_encoded_,
                diags_matrices_inv_bsgs_, real_shift_n2_inv_bsgs_, galois_key,
                options);
        }
        else
        {
            c1 = multiply_matrix(cipher, V_inv_matrixs_rotated_encoded_,
                                 diags_matrices_inv_bsgs_, galois_key, options);
        }

        Ciphertext<Scheme::CKKS> c2 = operator_ciphertext(0, options.stream_);
        conjugate(c1, c2, galois_key, options); // conjugate

        Ciphertext<Scheme::CKKS> result =
            operator_ciphertext(0, options.stream_);
        add(c1, c2, result, options);

        int current_decomp_count = Q_size_ - result.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            result.data(), encoded_constant_1over2_.data(), result.data(),
            modulus_->data(), n_power);
        result.scale_ = result.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        result.rescale_required_ = true;
        rescale_inplace(result, options);

        return result;
    }

    __host__ Ciphertext<Scheme::CKKS> HEOperator<Scheme::CKKS>::slot_to_coeff(
        Ciphertext<Scheme::CKKS>& cipher0, Ciphertext<Scheme::CKKS>& cipher1,
        Galoiskey<Scheme::CKKS>& galois_key, const ExecutionOptions& options)
    {
        hipStream_t old_stream = cipher1.stream();
        cipher1.switch_stream(
            options.stream_); // TODO: Change copy and assign structure!
        Ciphertext<Scheme::CKKS> result;
        result = cipher1;
        cipher1.switch_stream(
            old_stream); // TODO: Change copy and assign structure!

        int current_decomp_count = Q_size_ - cipher1.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            result.data(), encoded_complex_i_.data(), result.data(),
            modulus_->data(), n_power);
        result.scale_ = result.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        result.rescale_required_ = true;
        rescale_inplace(result, options);

        mod_drop_inplace(cipher0, options);

        add(result, cipher0, result, options);

        Ciphertext<Scheme::CKKS> c1;
        if (less_key_mode_)
        {
            c1 = multiply_matrix_less_memory(
                result, V_matrixs_rotated_encoded_, diags_matrices_bsgs_,
                real_shift_n2_bsgs_, galois_key, options);
        }
        else
        {
            c1 = multiply_matrix(result, V_matrixs_rotated_encoded_,
                                 diags_matrices_bsgs_, galois_key, options);
        }

        return c1;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HEOperator<Scheme::CKKS>::solo_slot_to_coeff(
        Ciphertext<Scheme::CKKS>& cipher, Galoiskey<Scheme::CKKS>& galois_key,
        const ExecutionOptions& options)
    {
        Ciphertext<Scheme::CKKS> result;
        if (less_key_mode_)
        {
            result = multiply_matrix_less_memory(
                cipher, V_matrixs_rotated_encoded_, diags_matrices_bsgs_,
                real_shift_n2_bsgs_, galois_key, options);
        }
        else
        {
            result = multiply_matrix(cipher, V_matrixs_rotated_encoded_,
                                     diags_matrices_bsgs_, galois_key, options);
        }

        return result;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HEOperator<Scheme::CKKS>::exp_scaled(Ciphertext<Scheme::CKKS>& cipher,
                                         Relinkey<Scheme::CKKS>& relin_key,
                                         const ExecutionOptions& options)
    {
        int current_decomp_count = Q_size_ - cipher.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            cipher.data(), encoded_complex_iscaleoverr_.data(), cipher.data(),
            modulus_->data(), n_power);
        cipher.scale_ = cipher.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        cipher.rescale_required_ = true;
        rescale_inplace(cipher, options);

        Ciphertext<Scheme::CKKS> cipher_taylor =
            exp_taylor_approximation(cipher, relin_key, options);

        for (int i = 0; i < taylor_number_; i++)
        {
            multiply_inplace(cipher_taylor, cipher_taylor, options);
            relinearize_inplace(cipher_taylor, relin_key, options);
            rescale_inplace(cipher_taylor, options);
        }

        return cipher_taylor;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HEOperator<Scheme::CKKS>::exp_taylor_approximation(
        Ciphertext<Scheme::CKKS>& cipher, Relinkey<Scheme::CKKS>& relin_key,
        const ExecutionOptions& options)
    {
        hipStream_t old_stream = cipher.stream();
        cipher.switch_stream(
            options.stream_); // TODO: Change copy and assign structure!
        Ciphertext<Scheme::CKKS> second;
        second = cipher; // 1 - c^1

        Ciphertext<Scheme::CKKS> third =
            operator_ciphertext(0, options.stream_);
        multiply(second, second, third, options);
        relinearize_inplace(third, relin_key, options);
        rescale_inplace(third, options); // 2 - c^2

        mod_drop_inplace(second, options); // 2
        Ciphertext<Scheme::CKKS> forth =
            operator_ciphertext(0, options.stream_);
        multiply(third, second, forth, options);
        relinearize_inplace(forth, relin_key, options);
        rescale_inplace(forth, options); // 3 - c^3

        Ciphertext<Scheme::CKKS> fifth =
            operator_ciphertext(0, options.stream_);
        multiply(third, third, fifth, options);
        relinearize_inplace(fifth, relin_key, options);
        rescale_inplace(fifth, options); // 3 - c^4

        mod_drop_inplace(second, options); // 3
        Ciphertext<Scheme::CKKS> sixth =
            operator_ciphertext(0, options.stream_);
        multiply(fifth, second, sixth, options);
        relinearize_inplace(sixth, relin_key, options);
        rescale_inplace(sixth, options); // 4 - c^5

        Ciphertext<Scheme::CKKS> seventh =
            operator_ciphertext(0, options.stream_);
        multiply(forth, forth, seventh, options);
        relinearize_inplace(seventh, relin_key, options);
        rescale_inplace(seventh, options); // 4 - c^6

        Ciphertext<Scheme::CKKS> eighth =
            operator_ciphertext(0, options.stream_);
        multiply(fifth, forth, eighth, options);
        relinearize_inplace(eighth, relin_key, options);
        rescale_inplace(eighth, options); // 4 - c^7

        //

        int current_decomp_count = Q_size_ - third.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            third.data(), encoded_constant_1over2_.data(), third.data(),
            modulus_->data(), n_power);
        third.scale_ = third.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        third.rescale_required_ = true;
        rescale_inplace(third, options); // 3

        //

        current_decomp_count = Q_size_ - forth.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            forth.data(), encoded_constant_1over6_.data(), forth.data(),
            modulus_->data(), n_power);
        forth.scale_ = forth.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        forth.rescale_required_ = true;
        rescale_inplace(forth, options); // 4

        //

        current_decomp_count = Q_size_ - fifth.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            fifth.data(), encoded_constant_1over24_.data(), fifth.data(),
            modulus_->data(), n_power);
        fifth.scale_ = fifth.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        fifth.rescale_required_ = true;
        rescale_inplace(fifth, options); // 4

        //

        current_decomp_count = Q_size_ - sixth.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            sixth.data(), encoded_constant_1over120_.data(), sixth.data(),
            modulus_->data(), n_power);
        sixth.scale_ = sixth.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        sixth.rescale_required_ = true;
        rescale_inplace(sixth, options); // 5

        //

        current_decomp_count = Q_size_ - seventh.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            seventh.data(), encoded_constant_1over720_.data(), seventh.data(),
            modulus_->data(), n_power);
        seventh.scale_ = seventh.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        seventh.rescale_required_ = true;
        rescale_inplace(seventh, options); // 5

        //

        current_decomp_count = Q_size_ - eighth.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            eighth.data(), encoded_constant_1over5040_.data(), eighth.data(),
            modulus_->data(), n_power);
        eighth.scale_ = eighth.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        eighth.rescale_required_ = true;
        rescale_inplace(eighth, options); // 5

        //

        Ciphertext<Scheme::CKKS> result =
            operator_ciphertext(0, options.stream_);
        current_decomp_count = Q_size_ - second.depth_;
        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options.stream_>>>(
            second.data(), encoded_constant_1_.data(), result.data(),
            modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        result.scheme_ = scheme_;
        result.ring_size_ = n;
        result.coeff_modulus_count_ = Q_size_;
        result.cipher_size_ = 2;
        result.depth_ = second.depth_;
        result.scale_ = second.scale_;
        result.in_ntt_domain_ = second.in_ntt_domain_;
        result.rescale_required_ = second.rescale_required_;
        result.relinearization_required_ = second.relinearization_required_;
        result.ciphertext_generated_ = true;

        //

        add_inplace(result, third, options); // 3

        //

        mod_drop_inplace(result, options); // 4

        //

        add_inplace(result, forth, options); // 4
        add_inplace(result, fifth, options); // 4

        //

        mod_drop_inplace(result, options); // 5

        //

        add_inplace(result, sixth, options); // 5
        add_inplace(result, seventh, options); // 5
        add_inplace(result, eighth, options); // 5

        return result;
    }

    __host__ DeviceVector<Data64>
    HEOperator<Scheme::CKKS>::fast_single_hoisting_rotation_ckks_method_I(
        Ciphertext<Scheme::CKKS>& first_cipher, std::vector<int>& bsgs_shift,
        int n1, Galoiskey<Scheme::CKKS>& galois_key, const hipStream_t stream)
    {
        int current_level = first_cipher.depth_;
        int first_rns_mod_count = Q_prime_size_;
        int current_rns_mod_count = Q_prime_size_ - current_level;
        int current_decomp_count = Q_size_ - current_level;

        DeviceVector<Data64> temp_rotation(
            (2 * n * Q_size_) + (2 * n * Q_size_) +
                (n * Q_size_ * Q_prime_size_) + (2 * n * Q_prime_size_),
            stream);

        Data64* temp0_rotation = temp_rotation.data();
        Data64* temp1_rotation = temp0_rotation + (2 * n * Q_size_);
        Data64* temp2_rotation = temp1_rotation + (2 * n * Q_size_);
        Data64* temp3_rotation = temp2_rotation + (n * Q_size_ * Q_prime_size_);

        DeviceVector<Data64> result((2 * current_decomp_count * n1) << n_power,
                                    stream); // store n1 ciphertext

        // decompose and mult P
        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = stream};

        gpuntt::GPU_NTT(first_cipher.data(), temp0_rotation,
                        intt_table_->data(), modulus_->data(), cfg_intt,
                        2 * current_decomp_count, current_decomp_count);

        // TODO: make it efficient
        ckks_duplicate_kernel<<<dim3((n >> 8), current_decomp_count, 1), 256, 0,
                                stream>>>(
            temp0_rotation, temp2_rotation, modulus_->data(), n_power,
            first_rns_mod_count, current_rns_mod_count, current_decomp_count);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        int counter = first_rns_mod_count;
        int location = 0;
        for (int i = 0; i < current_level; i++)
        {
            location += counter;
            counter--;
        }

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp2_rotation, ntt_table_->data(), modulus_->data(), cfg_ntt,
            current_decomp_count * current_rns_mod_count, current_rns_mod_count,
            new_prime_locations + location);

        //

        global_memory_replace_kernel<<<dim3((n >> 8), current_decomp_count, 2),
                                       256, 0, stream>>>(
            first_cipher.data(), result.data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        //

        for (int i = 1; i < n1; i++)
        {
            int shift_n1 = bsgs_shift[i];
            int galoiselt =
                steps_to_galois_elt(shift_n1, n, galois_key.group_order_);
            int offset = ((2 * current_decomp_count) << n_power) * i;

            // MultSum
            // TODO: make it efficient
            int iteration_count_1 = current_decomp_count / 4;
            int iteration_count_2 = current_decomp_count % 4;
            if (galois_key.storage_type_ == storage_type::DEVICE)
            {
                keyswitch_multiply_accumulate_leveled_kernel<<<
                    dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                    temp2_rotation,
                    galois_key.device_location_[galoiselt].data(),
                    temp3_rotation, modulus_->data(), first_rns_mod_count,
                    current_decomp_count, iteration_count_1, iteration_count_2,
                    n_power);
                HEONGPU_CUDA_CHECK(hipGetLastError());
            }
            else
            {
                DeviceVector<Data64> key_location(
                    galois_key.host_location_[galoiselt], stream);
                keyswitch_multiply_accumulate_leveled_kernel<<<
                    dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                    temp2_rotation, key_location.data(), temp3_rotation,
                    modulus_->data(), first_rns_mod_count, current_decomp_count,
                    iteration_count_1, iteration_count_2, n_power);
                HEONGPU_CUDA_CHECK(hipGetLastError());
            }

            gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
                temp3_rotation, intt_table_->data(), modulus_->data(), cfg_intt,
                2 * current_rns_mod_count, current_rns_mod_count,
                new_prime_locations + location);

            // ModDown + Permute
            divide_round_lastq_permute_ckks_kernel<<<
                dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
                temp3_rotation, temp0_rotation, result.data() + offset,
                modulus_->data(), half_p_->data(), half_mod_->data(),
                last_q_modinv_->data(), galoiselt, n_power,
                current_rns_mod_count, current_decomp_count,
                first_rns_mod_count, Q_size_, P_size_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            gpuntt::GPU_NTT_Inplace(
                result.data() + offset, ntt_table_->data(), modulus_->data(),
                cfg_ntt, 2 * current_decomp_count, current_decomp_count);
        }

        return result;
    }

    __host__ DeviceVector<Data64>
    HEOperator<Scheme::CKKS>::fast_single_hoisting_rotation_ckks_method_II(
        Ciphertext<Scheme::CKKS>& first_cipher, std::vector<int>& bsgs_shift,
        int n1, Galoiskey<Scheme::CKKS>& galois_key, const hipStream_t stream)
    {
        int current_level = first_cipher.depth_;
        int first_rns_mod_count = Q_prime_size_;
        int current_rns_mod_count = Q_prime_size_ - current_level;
        int current_decomp_count = Q_size_ - current_level;

        DeviceVector<Data64> temp_rotation(
            (2 * n * Q_size_) + (2 * n * Q_size_) + (n * Q_size_) +
                (2 * n * d_leveled_->operator[](0) * Q_prime_size_) +
                (2 * n * Q_prime_size_),
            stream);

        Data64* temp0_rotation = temp_rotation.data();
        Data64* temp1_rotation = temp0_rotation + (2 * n * Q_size_);
        Data64* temp2_rotation = temp1_rotation + (2 * n * Q_size_);
        Data64* temp3_rotation = temp2_rotation + (n * Q_size_);
        Data64* temp4_rotation =
            temp3_rotation +
            (2 * n * d_leveled_->operator[](0) * Q_prime_size_);

        DeviceVector<Data64> result((2 * current_decomp_count * n1) << n_power,
                                    stream); // store n1 ciphertext

        // decompose and mult P
        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = stream};

        gpuntt::GPU_NTT(first_cipher.data(), temp0_rotation,
                        intt_table_->data(), modulus_->data(), cfg_intt,
                        2 * current_decomp_count, current_decomp_count);

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = stream};

        int counter = first_rns_mod_count;
        int location = 0;
        for (int i = 0; i < current_level; i++)
        {
            location += counter;
            counter--;
        }

        base_conversion_DtoQtilde_relin_leveled_kernel<<<
            dim3((n >> 8), d_leveled_->operator[](current_level), 1), 256, 0,
            stream>>>(
            temp0_rotation + (current_decomp_count << n_power), temp3_rotation,
            modulus_->data(),
            base_change_matrix_D_to_Qtilda_leveled_->operator[](current_level)
                .data(),
            Mi_inv_D_to_Qtilda_leveled_->operator[](current_level).data(),
            prod_D_to_Qtilda_leveled_->operator[](current_level).data(),
            I_j_leveled_->operator[](current_level).data(),
            I_location_leveled_->operator[](current_level).data(), n_power,
            d_leveled_->operator[](current_level), current_rns_mod_count,
            current_decomp_count, current_level,
            prime_location_leveled_->data() + location);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
            temp3_rotation, ntt_table_->data(), modulus_->data(), cfg_ntt,
            d_leveled_->operator[](current_level) * current_rns_mod_count,
            current_rns_mod_count, new_prime_locations + location);

        global_memory_replace_kernel<<<dim3((n >> 8), current_decomp_count, 2),
                                       256, 0, stream>>>(
            first_cipher.data(), result.data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        for (int i = 1; i < n1; i++)
        {
            int shift_n1 = bsgs_shift[i];
            int galoiselt =
                steps_to_galois_elt(shift_n1, n, galois_key.group_order_);
            int offset = ((2 * current_decomp_count) << n_power) * i;

            // MultSum
            // TODO: make it efficient
            int iteration_count_1 =
                d_leveled_->operator[](first_cipher.depth_) / 4;
            int iteration_count_2 =
                d_leveled_->operator[](first_cipher.depth_) % 4;
            if (galois_key.storage_type_ == storage_type::DEVICE)
            {
                keyswitch_multiply_accumulate_leveled_method_II_kernel<<<
                    dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                    temp3_rotation,
                    galois_key.device_location_[galoiselt].data(),
                    temp4_rotation, modulus_->data(), first_rns_mod_count,
                    current_decomp_count, current_rns_mod_count,
                    iteration_count_1, iteration_count_2, first_cipher.depth_,
                    n_power);
                HEONGPU_CUDA_CHECK(hipGetLastError());
            }
            else
            {
                DeviceVector<Data64> key_location(
                    galois_key.host_location_[galoiselt], stream);
                keyswitch_multiply_accumulate_leveled_method_II_kernel<<<
                    dim3((n >> 8), current_rns_mod_count, 1), 256, 0, stream>>>(
                    temp3_rotation, key_location.data(), temp4_rotation,
                    modulus_->data(), first_rns_mod_count, current_decomp_count,
                    current_rns_mod_count, iteration_count_1, iteration_count_2,
                    first_cipher.depth_, n_power);
                HEONGPU_CUDA_CHECK(hipGetLastError());
            }

            gpuntt::GPU_NTT_Modulus_Ordered_Inplace(
                temp4_rotation, intt_table_->data(), modulus_->data(), cfg_intt,
                2 * current_rns_mod_count, current_rns_mod_count,
                new_prime_locations + location);

            // ModDown + Permute
            divide_round_lastq_permute_ckks_kernel<<<
                dim3((n >> 8), current_decomp_count, 2), 256, 0, stream>>>(
                temp4_rotation, temp0_rotation, result.data() + offset,
                modulus_->data(), half_p_->data(), half_mod_->data(),
                last_q_modinv_->data(), galoiselt, n_power,
                current_rns_mod_count, current_decomp_count,
                first_rns_mod_count, Q_size_, P_size_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            gpuntt::GPU_NTT_Inplace(
                result.data() + offset, ntt_table_->data(), modulus_->data(),
                cfg_ntt, 2 * current_decomp_count, current_decomp_count);
        }

        return result;
    }

    __host__ HEOperator<Scheme::CKKS>::Vandermonde::Vandermonde(
        const int poly_degree, const int CtoS_piece, const int StoC_piece,
        const bool less_key_mode)
    {
        poly_degree_ = poly_degree;
        num_slots_ = poly_degree_ >> 1;
        log_num_slots_ = int(log2l(num_slots_));

        CtoS_piece_ = CtoS_piece;
        StoC_piece_ = StoC_piece;

        generate_E_diagonals_index();
        generate_E_inv_diagonals_index();
        split_E();
        split_E_inv();

        generate_E_diagonals();
        generate_E_inv_diagonals();

        generate_V_n_lists();

        generate_pre_comp_V();
        generate_pre_comp_V_inv();

        generate_key_indexs(less_key_mode);
        key_indexs_ = unique_sort(key_indexs_);
    }

    __host__ void
    HEOperator<Scheme::CKKS>::Vandermonde::generate_E_diagonals_index()
    {
        bool first = true;
        for (int i = 1; i < (log_num_slots_ + 1); i++)
        {
            if (first)
            {
                int block_size = num_slots_ >> i;
                E_index_.push_back(0);
                E_index_.push_back(block_size);
                first = false;

                E_size_.push_back(2);
            }
            else
            {
                int block_size = num_slots_ >> i;
                E_index_.push_back(0);
                E_index_.push_back(block_size);
                E_index_.push_back(num_slots_ - block_size);

                E_size_.push_back(3);
            }
        }
    }

    __host__ void
    HEOperator<Scheme::CKKS>::Vandermonde::generate_E_inv_diagonals_index()
    {
        for (int i = log_num_slots_; 0 < i; i--)
        {
            if (i == 1)
            {
                int block_size = num_slots_ >> i;
                E_inv_index_.push_back(0);
                E_inv_index_.push_back(block_size);

                E_inv_size_.push_back(2);
            }
            else
            {
                int block_size = num_slots_ >> i;
                E_inv_index_.push_back(0);
                E_inv_index_.push_back(block_size);
                E_inv_index_.push_back(num_slots_ - block_size);

                E_inv_size_.push_back(3);
            }
        }
    }

    __host__ void HEOperator<Scheme::CKKS>::Vandermonde::split_E()
    {
        // E_splitted
        int k = log_num_slots_ / StoC_piece_;
        int m = log_num_slots_ % StoC_piece_;

        for (int i = 0; i < StoC_piece_; i++)
        {
            E_splitted_.push_back(k);
        }

        for (int i = 0; i < m; i++)
        {
            E_splitted_[i]++;
        }

        int counter = 0;
        for (int i = 0; i < StoC_piece_; i++)
        {
            std::vector<int> temp;
            for (int j = 0; j < E_splitted_[i]; j++)
            {
                int size = (counter == 0) ? 2 : 3;
                for (int k = 0; k < size; k++)
                {
                    temp.push_back(E_index_[counter]);
                    counter++;
                }
            }
            E_splitted_index_.push_back(temp);
        }

        int num_slots_mask = num_slots_ - 1;
        counter = 0;
        for (int k = 0; k < StoC_piece_; k++)
        {
            int matrix_count = E_splitted_[k];
            int L_m_loc = (k == 0) ? 2 : 3;
            std::vector<int> index_mul;
            std::vector<int> index_mul_sorted;
            std::vector<int> diag_index_temp;
            std::vector<int> iteration_temp;
            for (int m = 0; m < matrix_count - 1; m++)
            {
                if (m == 0)
                {
                    iteration_temp.push_back(E_size_[counter]);
                    for (int i = 0; i < E_size_[counter]; i++)
                    {
                        int R_m_İNDEX = E_splitted_index_[k][i];
                        diag_index_temp.push_back(R_m_İNDEX);
                        for (int j = 0; j < E_size_[counter + 1]; j++)
                        {
                            int L_m_İNDEX = E_splitted_index_[k][L_m_loc + j];
                            index_mul.push_back((L_m_İNDEX + R_m_İNDEX) &
                                                num_slots_mask);
                        }
                    }
                    index_mul_sorted = unique_sort(index_mul);
                    index_mul.clear();
                    L_m_loc += 3;
                }
                else
                {
                    iteration_temp.push_back(index_mul_sorted.size());
                    for (int i = 0; i < index_mul_sorted.size(); i++)
                    {
                        int R_m_İNDEX = index_mul_sorted[i];
                        diag_index_temp.push_back(R_m_İNDEX);
                        for (int j = 0; j < E_size_[counter + 1 + m]; j++)
                        {
                            int L_m_İNDEX = E_splitted_index_[k][L_m_loc + j];
                            index_mul.push_back((L_m_İNDEX + R_m_İNDEX) &
                                                num_slots_mask);
                        }
                    }
                    index_mul_sorted = unique_sort(index_mul);
                    index_mul.clear();
                    L_m_loc += 3;
                }
            }
            V_matrixs_index_.push_back(index_mul_sorted);
            E_splitted_diag_index_gpu_.push_back(diag_index_temp);
            E_splitted_iteration_gpu_.push_back(iteration_temp);
            counter += matrix_count;
        }

        std::vector<std::unordered_map<int, int>> dict_output_index;
        for (int k = 0; k < StoC_piece_; k++)
        {
            std::unordered_map<int, int> temp;
            for (int i = 0; i < V_matrixs_index_[k].size(); i++)
            {
                temp[V_matrixs_index_[k][i]] = i;
            }
            dict_output_index.push_back(temp);
        }

        counter = 0;
        for (int k = 0; k < StoC_piece_; k++)
        {
            int matrix_count = E_splitted_[k];
            int L_m_loc = (k == 0) ? 2 : 3;
            std::vector<int> index_mul;
            std::vector<int> index_mul_sorted;

            std::vector<int> temp_in_index;
            std::vector<int> temp_out_index;
            for (int m = 0; m < matrix_count - 1; m++)
            {
                if (m == 0)
                {
                    for (int i = 0; i < E_size_[counter]; i++)
                    {
                        int R_m_İNDEX = E_splitted_index_[k][i];
                        for (int j = 0; j < E_size_[counter + 1]; j++)
                        {
                            int L_m_İNDEX = E_splitted_index_[k][L_m_loc + j];
                            int indexs =
                                (L_m_İNDEX + R_m_İNDEX) & num_slots_mask;
                            index_mul.push_back(indexs);
                            temp_out_index.push_back(
                                dict_output_index[k][indexs]);
                        }
                    }
                    index_mul_sorted = unique_sort(index_mul);
                    index_mul.clear();
                    L_m_loc += 3;
                }
                else
                {
                    for (int i = 0; i < index_mul_sorted.size(); i++)
                    {
                        int R_m_İNDEX = index_mul_sorted[i];
                        temp_in_index.push_back(
                            dict_output_index[k][R_m_İNDEX]);
                        for (int j = 0; j < E_size_[counter + 1 + m]; j++)
                        {
                            int L_m_İNDEX = E_splitted_index_[k][L_m_loc + j];
                            int indexs =
                                (L_m_İNDEX + R_m_İNDEX) & num_slots_mask;
                            index_mul.push_back(indexs);
                            temp_out_index.push_back(
                                dict_output_index[k][indexs]);
                        }
                    }
                    index_mul_sorted = unique_sort(index_mul);
                    index_mul.clear();
                    L_m_loc += 3;
                }
            }
            counter += matrix_count;
            E_splitted_input_index_gpu_.push_back(temp_in_index);
            E_splitted_output_index_gpu_.push_back(temp_out_index);
        }
    }

    __host__ void HEOperator<Scheme::CKKS>::Vandermonde::split_E_inv()
    {
        // E_inv_splitted
        int k = log_num_slots_ / CtoS_piece_;
        int m = log_num_slots_ % CtoS_piece_;

        for (int i = 0; i < CtoS_piece_; i++)
        {
            E_inv_splitted_.push_back(k);
        }

        for (int i = 0; i < m; i++)
        {
            E_inv_splitted_[i]++;
        }

        int counter = 0;
        for (int i = 0; i < CtoS_piece_; i++)
        {
            std::vector<int> temp;
            for (int j = 0; j < E_inv_splitted_[i]; j++)
            {
                int size = (counter == (E_inv_index_.size() - 2)) ? 2 : 3;
                for (int k = 0; k < size; k++)
                {
                    temp.push_back(E_inv_index_[counter]);
                    counter++;
                }
            }
            E_inv_splitted_index_.push_back(temp);
        }

        int num_slots_mask = num_slots_ - 1;
        counter = 0;
        for (int k = 0; k < CtoS_piece_; k++)
        {
            int matrix_count = E_inv_splitted_[k];

            int L_m_loc = 3;
            std::vector<int> index_mul;
            std::vector<int> index_mul_sorted;
            std::vector<int> diag_index_temp;
            std::vector<int> iteration_temp;
            for (int m = 0; m < matrix_count - 1; m++)
            {
                if (m == 0)
                {
                    iteration_temp.push_back(E_inv_size_[counter]);
                    for (int i = 0; i < E_inv_size_[counter]; i++)
                    {
                        int R_m_İNDEX = E_inv_splitted_index_[k][i];
                        diag_index_temp.push_back(R_m_İNDEX);
                        for (int j = 0; j < E_inv_size_[counter + 1]; j++)
                        {
                            int L_m_İNDEX =
                                E_inv_splitted_index_[k][L_m_loc + j];
                            index_mul.push_back((L_m_İNDEX + R_m_İNDEX) &
                                                num_slots_mask);
                        }
                    }
                    index_mul_sorted = unique_sort(index_mul);
                    index_mul.clear();
                    L_m_loc += 3;
                }
                else
                {
                    iteration_temp.push_back(index_mul_sorted.size());
                    for (int i = 0; i < index_mul_sorted.size(); i++)
                    {
                        int R_m_İNDEX = index_mul_sorted[i];
                        diag_index_temp.push_back(R_m_İNDEX);
                        for (int j = 0; j < E_inv_size_[counter + 1 + m]; j++)
                        {
                            int L_m_İNDEX =
                                E_inv_splitted_index_[k][L_m_loc + j];
                            index_mul.push_back((L_m_İNDEX + R_m_İNDEX) &
                                                num_slots_mask);
                        }
                    }
                    index_mul_sorted = unique_sort(index_mul);
                    index_mul.clear();
                    L_m_loc += 3;
                }
            }
            V_inv_matrixs_index_.push_back(index_mul_sorted);
            E_inv_splitted_diag_index_gpu_.push_back(diag_index_temp);
            E_inv_splitted_iteration_gpu_.push_back(iteration_temp);
            counter += matrix_count;
        }

        std::vector<std::unordered_map<int, int>> dict_output_index;
        for (int k = 0; k < CtoS_piece_; k++)
        {
            std::unordered_map<int, int> temp;
            for (int i = 0; i < V_inv_matrixs_index_[k].size(); i++)
            {
                temp[V_inv_matrixs_index_[k][i]] = i;
            }
            dict_output_index.push_back(temp);
        }

        counter = 0;
        for (int k = 0; k < CtoS_piece_; k++)
        {
            int matrix_count = E_inv_splitted_[k];
            int L_m_loc = 3;
            std::vector<int> index_mul;
            std::vector<int> index_mul_sorted;

            std::vector<int> temp_in_index;
            std::vector<int> temp_out_index;
            for (int m = 0; m < matrix_count - 1; m++)
            {
                if (m == 0)
                {
                    for (int i = 0; i < E_inv_size_[counter]; i++)
                    {
                        int R_m_İNDEX = E_inv_splitted_index_[k][i];
                        for (int j = 0; j < E_inv_size_[counter + 1]; j++)
                        {
                            int L_m_İNDEX =
                                E_inv_splitted_index_[k][L_m_loc + j];
                            int indexs =
                                (L_m_İNDEX + R_m_İNDEX) & num_slots_mask;
                            index_mul.push_back(indexs);
                            temp_out_index.push_back(
                                dict_output_index[k][indexs]);
                        }
                    }
                    index_mul_sorted = unique_sort(index_mul);
                    index_mul.clear();
                    L_m_loc += 3;
                }
                else
                {
                    for (int i = 0; i < index_mul_sorted.size(); i++)
                    {
                        int R_m_İNDEX = index_mul_sorted[i];
                        temp_in_index.push_back(
                            dict_output_index[k][R_m_İNDEX]);
                        for (int j = 0; j < E_inv_size_[counter + 1 + m]; j++)
                        {
                            int L_m_İNDEX =
                                E_inv_splitted_index_[k][L_m_loc + j];
                            int indexs =
                                (L_m_İNDEX + R_m_İNDEX) & num_slots_mask;
                            index_mul.push_back(indexs);
                            temp_out_index.push_back(
                                dict_output_index[k][indexs]);
                        }
                    }
                    index_mul_sorted = unique_sort(index_mul);
                    index_mul.clear();
                    L_m_loc += 3;
                }
            }
            counter += matrix_count;
            E_inv_splitted_input_index_gpu_.push_back(temp_in_index);
            E_inv_splitted_output_index_gpu_.push_back(temp_out_index);
        }
    }

    __host__ void HEOperator<Scheme::CKKS>::Vandermonde::generate_E_diagonals()
    {
        int bloksize = (num_slots_ <= 1024) ? num_slots_ : 1024;
        int blokcount = (num_slots_ + (1023)) / 1024;

        heongpu::DeviceVector<Complex64> V_logn_diagnal(
            ((3 * log_num_slots_) - 1) << log_num_slots_);
        E_diagonal_generate_kernel<<<dim3(blokcount, log_num_slots_, 1),
                                     bloksize>>>(V_logn_diagnal.data(),
                                                 log_num_slots_);

        int matrix_counter = 0;
        for (int i = 0; i < StoC_piece_; i++)
        {
            heongpu::DeviceVector<int> diag_index_gpu(
                E_splitted_diag_index_gpu_[i]);
            heongpu::DeviceVector<int> input_index_gpu(
                E_splitted_input_index_gpu_[i]);
            heongpu::DeviceVector<int> output_index_gpu(
                E_splitted_output_index_gpu_[i]);

            heongpu::DeviceVector<Complex64> V_mul((V_matrixs_index_[i].size())
                                                   << log_num_slots_);
            hipMemset(V_mul.data(), 0, V_mul.size() * sizeof(Complex64));

            int input_loc;
            if (i == 0)
            {
                input_loc = 0;
            }
            else
            {
                input_loc = ((3 * matrix_counter) - 1) << log_num_slots_;
            }

            int R_matrix_counter = 0;
            int output_index_counter = 0;

            for (int j = 0; j < (E_splitted_[i] - 1); j++)
            {
                heongpu::DeviceVector<Complex64> temp_result(
                    (V_matrixs_index_[i].size()) << log_num_slots_);
                hipMemset(temp_result.data(), 0,
                           temp_result.size() * sizeof(Complex64));

                bool first_check1 = (i == 0) ? true : false;
                bool first_check2 = (j == 0) ? true : false;

                E_diagonal_matrix_mult_kernel<<<blokcount, bloksize>>>(
                    V_logn_diagnal.data() + input_loc, temp_result.data(),
                    V_mul.data(), diag_index_gpu.data(), input_index_gpu.data(),
                    output_index_gpu.data(), E_splitted_iteration_gpu_[i][j],
                    R_matrix_counter, output_index_counter, j, first_check1,
                    first_check2, log_num_slots_);

                V_mul = std::move(temp_result);

                R_matrix_counter += E_splitted_iteration_gpu_[i][j];
                output_index_counter += (E_splitted_iteration_gpu_[i][j] * 3);
            }

            V_matrixs_.push_back(std::move(V_mul));
            matrix_counter += E_splitted_[i];
        }
    }

    __host__ void
    HEOperator<Scheme::CKKS>::Vandermonde::generate_E_inv_diagonals()
    {
        int bloksize = (num_slots_ <= 1024) ? num_slots_ : 1024;
        int blokcount = (num_slots_ + (1023)) / 1024;

        heongpu::DeviceVector<Complex64> V_inv_logn_diagnal(
            ((3 * log_num_slots_) - 1) << log_num_slots_);
        E_diagonal_inverse_generate_kernel<<<dim3(blokcount, log_num_slots_, 1),
                                             bloksize>>>(
            V_inv_logn_diagnal.data(), log_num_slots_);

        int matrix_counter = 0;
        for (int i = 0; i < CtoS_piece_; i++)
        {
            heongpu::DeviceVector<int> diag_index_gpu(
                E_inv_splitted_diag_index_gpu_[i]);
            heongpu::DeviceVector<int> input_index_gpu(
                E_inv_splitted_input_index_gpu_[i]);
            heongpu::DeviceVector<int> output_index_gpu(
                E_inv_splitted_output_index_gpu_[i]);

            heongpu::DeviceVector<Complex64> V_mul(
                (V_inv_matrixs_index_[i].size()) << log_num_slots_);
            hipMemset(V_mul.data(), 0, V_mul.size() * sizeof(Complex64));

            int input_loc = (3 * matrix_counter) << log_num_slots_;
            int R_matrix_counter = 0;
            int output_index_counter = 0;

            for (int j = 0; j < (E_inv_splitted_[i] - 1); j++)
            {
                heongpu::DeviceVector<Complex64> temp_result(
                    (V_inv_matrixs_index_[i].size()) << log_num_slots_);
                hipMemset(temp_result.data(), 0,
                           temp_result.size() * sizeof(Complex64));
                bool first_check = (j == 0) ? true : false;
                bool last_check = ((i == (CtoS_piece_ - 1)) &&
                                   (j == (E_inv_splitted_[i] - 2)))
                                      ? true
                                      : false;

                E_diagonal_inverse_matrix_mult_kernel<<<blokcount, bloksize>>>(
                    V_inv_logn_diagnal.data() + input_loc, temp_result.data(),
                    V_mul.data(), diag_index_gpu.data(), input_index_gpu.data(),
                    output_index_gpu.data(),
                    E_inv_splitted_iteration_gpu_[i][j], R_matrix_counter,
                    output_index_counter, j, first_check, last_check,
                    log_num_slots_);

                V_mul = std::move(temp_result);
                R_matrix_counter += E_inv_splitted_iteration_gpu_[i][j];
                output_index_counter +=
                    (E_inv_splitted_iteration_gpu_[i][j] * 3);
            }
            V_inv_matrixs_.push_back(std::move(V_mul));
            matrix_counter += E_inv_splitted_[i];
        }
    }

    __host__ void HEOperator<Scheme::CKKS>::Vandermonde::generate_V_n_lists()
    {
        for (int i = 0; i < StoC_piece_; i++)
        {
            std::vector<std::vector<int>> result =
                heongpu::seperate_func(V_matrixs_index_[i]);

            int sizex = result.size();
            int sizex_2 = (sizex >> 1);

            std::vector<std::vector<int>> real_shift_n2;
            for (size_t l1 = 0; l1 < sizex_2; l1++)
            {
                std::vector<int> temp = {result[l1][0]};
                real_shift_n2.push_back(std::move(temp));
            }

            for (size_t l1 = sizex_2; l1 < sizex; l1++)
            {
                std::vector<int> temp;
                int fisrt_ = result[sizex_2][0];
                int second_ = result[l1][0] - result[sizex_2][0];

                if (second_ == 0)
                {
                    temp.push_back(fisrt_);
                }
                else
                {
                    temp.push_back(fisrt_);
                    temp.push_back(second_);
                }

                real_shift_n2.push_back(std::move(temp));
            }

            diags_matrices_bsgs_.push_back(std::move(result));
            real_shift_n2_bsgs_.push_back(std::move(real_shift_n2));
        }

        for (int i = 0; i < CtoS_piece_; i++)
        {
            std::vector<std::vector<int>> result =
                heongpu::seperate_func(V_inv_matrixs_index_[i]);

            int sizex = result.size();
            int sizex_2 = (sizex >> 1);

            std::vector<std::vector<int>> real_shift_n2;
            for (size_t l1 = 0; l1 < sizex_2; l1++)
            {
                std::vector<int> temp = {result[l1][0]};
                real_shift_n2.push_back(std::move(temp));
            }

            for (size_t l1 = sizex_2; l1 < sizex; l1++)
            {
                std::vector<int> temp;
                int fisrt_ = result[sizex_2][0];
                int second_ = result[l1][0] - result[sizex_2][0];

                if (second_ == 0)
                {
                    temp.push_back(fisrt_);
                }
                else
                {
                    temp.push_back(fisrt_);
                    temp.push_back(second_);
                }

                real_shift_n2.push_back(std::move(temp));
            }

            diags_matrices_inv_bsgs_.push_back(std::move(result));
            real_shift_n2_inv_bsgs_.push_back(std::move(real_shift_n2));
        }
    }

    __host__ void HEOperator<Scheme::CKKS>::Vandermonde::generate_pre_comp_V()
    {
        int bloksize = (num_slots_ <= 1024) ? num_slots_ : 1024;
        int blokcount = (num_slots_ + (1023)) / 1024;

        for (int m = 0; m < StoC_piece_; m++)
        {
            heongpu::DeviceVector<Complex64> temp_rotated(
                (V_matrixs_index_[m].size()) << log_num_slots_);

            int counter = 0;
            for (int j = 0; j < diags_matrices_bsgs_[m].size(); j++)
            {
                int real_shift = -(diags_matrices_bsgs_[m][j][0]);
                for (int i = 0; i < diags_matrices_bsgs_[m][j].size(); i++)
                {
                    int location = (counter << log_num_slots_);

                    vector_rotate_kernel<<<blokcount, bloksize>>>(
                        V_matrixs_[m].data() + location,
                        temp_rotated.data() + location, real_shift,
                        log_num_slots_);

                    counter++;
                }
            }

            V_matrixs_rotated_.push_back(std::move(temp_rotated));
        }
    }

    __host__ void
    HEOperator<Scheme::CKKS>::Vandermonde::generate_pre_comp_V_inv()
    {
        int bloksize = (num_slots_ <= 1024) ? num_slots_ : 1024;
        int blokcount = (num_slots_ + (1023)) / 1024;

        for (int m = 0; m < CtoS_piece_; m++)
        {
            heongpu::DeviceVector<Complex64> temp_rotated(
                (V_inv_matrixs_index_[m].size()) << log_num_slots_);

            int counter = 0;
            for (int j = 0; j < diags_matrices_inv_bsgs_[m].size(); j++)
            {
                int real_shift = -(diags_matrices_inv_bsgs_[m][j][0]);
                for (int i = 0; i < diags_matrices_inv_bsgs_[m][j].size(); i++)
                {
                    int location = (counter << log_num_slots_);

                    vector_rotate_kernel<<<blokcount, bloksize>>>(
                        V_inv_matrixs_[m].data() + location,
                        temp_rotated.data() + location, real_shift,
                        log_num_slots_);

                    counter++;
                }
            }

            V_inv_matrixs_rotated_.push_back(std::move(temp_rotated));
        }
    }

    __host__ void HEOperator<Scheme::CKKS>::Vandermonde::generate_key_indexs(
        const bool less_key_mode)
    {
        if (less_key_mode)
        {
            for (int m = 0; m < CtoS_piece_; m++)
            {
                key_indexs_.insert(key_indexs_.end(),
                                   diags_matrices_inv_bsgs_[m][0].begin(),
                                   diags_matrices_inv_bsgs_[m][0].end());
                for (int j = 0; j < diags_matrices_inv_bsgs_[m].size(); j++)
                {
                    key_indexs_.push_back(real_shift_n2_inv_bsgs_[m][j][0]);
                }
            }

            for (int m = 0; m < StoC_piece_; m++)
            {
                key_indexs_.insert(key_indexs_.end(),
                                   diags_matrices_bsgs_[m][0].begin(),
                                   diags_matrices_bsgs_[m][0].end());
                for (int j = 0; j < diags_matrices_bsgs_[m].size(); j++)
                {
                    key_indexs_.push_back(real_shift_n2_bsgs_[m][j][0]);
                }
            }
        }
        else
        {
            for (int m = 0; m < CtoS_piece_; m++)
            {
                key_indexs_.insert(key_indexs_.end(),
                                   diags_matrices_inv_bsgs_[m][0].begin(),
                                   diags_matrices_inv_bsgs_[m][0].end());
                for (int j = 0; j < diags_matrices_inv_bsgs_[m].size(); j++)
                {
                    key_indexs_.push_back(diags_matrices_inv_bsgs_[m][j][0]);
                }
            }

            for (int m = 0; m < StoC_piece_; m++)
            {
                key_indexs_.insert(key_indexs_.end(),
                                   diags_matrices_bsgs_[m][0].begin(),
                                   diags_matrices_bsgs_[m][0].end());
                for (int j = 0; j < diags_matrices_bsgs_[m].size(); j++)
                {
                    key_indexs_.push_back(diags_matrices_bsgs_[m][j][0]);
                }
            }
        }
    }

    HEArithmeticOperator<Scheme::CKKS>::HEArithmeticOperator(
        HEContext<Scheme::CKKS>& context, HEEncoder<Scheme::CKKS>& encoder)
        : HEOperator<Scheme::CKKS>(context, encoder)
    {
    }

    __host__ void
    HEArithmeticOperator<Scheme::CKKS>::generate_bootstrapping_params(
        const double scale, const BootstrappingConfig& config)
    {
        if (!boot_context_generated_)
        {
            scale_boot_ = scale;
            CtoS_piece_ = config.CtoS_piece_;
            StoC_piece_ = config.StoC_piece_;
            taylor_number_ = config.taylor_number_;
            less_key_mode_ = config.less_key_mode_;

            // TODO: remove it!
            bool use_all_bases = false; // Do not change it!

            Vandermonde matrix_gen(n, CtoS_piece_, StoC_piece_, less_key_mode_);

            V_matrixs_rotated_encoded_ =
                encode_V_matrixs(matrix_gen, scale_boot_, use_all_bases);
            V_inv_matrixs_rotated_encoded_ =
                encode_V_inv_matrixs(matrix_gen, scale_boot_, use_all_bases);

            V_matrixs_index_ = matrix_gen.V_matrixs_index_;
            V_inv_matrixs_index_ = matrix_gen.V_inv_matrixs_index_;

            diags_matrices_bsgs_ = matrix_gen.diags_matrices_bsgs_;
            diags_matrices_inv_bsgs_ = matrix_gen.diags_matrices_inv_bsgs_;

            if (less_key_mode_)
            {
                real_shift_n2_bsgs_ = matrix_gen.real_shift_n2_bsgs_;
                real_shift_n2_inv_bsgs_ = matrix_gen.real_shift_n2_inv_bsgs_;
            }

            key_indexs_ = matrix_gen.key_indexs_;

            // Pre-computed encoded parameters
            // CtoS
            double constant_1over2 = 0.5;
            encoded_constant_1over2_ = DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1over2, encoded_constant_1over2_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            Complex64 complex_minus_iover2(0.0, -0.5);
            encoded_complex_minus_iover2_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_complex(
                complex_minus_iover2, encoded_complex_minus_iover2_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            // StoC
            Complex64 complex_i(0, 1);
            encoded_complex_i_ = DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_complex(
                complex_i, encoded_complex_i_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            // Scale part
            Complex64 complex_minus_iscale(
                0.0, -(((static_cast<double>(prime_vector_[0].value) * 0.25) /
                        (scale_boot_ * M_PI))));
            encoded_complex_minus_iscale_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_complex(
                complex_minus_iscale, encoded_complex_minus_iscale_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            // Exponentiate
            Complex64 complex_iscaleoverr(
                0.0, (((2 * M_PI * scale_boot_) /
                       static_cast<double>(prime_vector_[0].value))) /
                         static_cast<double>(1 << taylor_number_));
            encoded_complex_iscaleoverr_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_complex(
                complex_iscaleoverr, encoded_complex_iscaleoverr_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            // Sinus taylor
            double constant_1 = 1.0;
            encoded_constant_1_ = DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1, encoded_constant_1_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_1over6 = 1.0 / 6.0;
            encoded_constant_1over6_ = DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1over6, encoded_constant_1over6_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_1over24 = 1.0 / 24.0;
            encoded_constant_1over24_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(constant_1over24,
                                               encoded_constant_1over24_.data(),
                                               scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_1over120 = 1.0 / 120.0;
            encoded_constant_1over120_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1over120, encoded_constant_1over120_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_1over720 = 1.0 / 720.0;
            encoded_constant_1over720_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1over720, encoded_constant_1over720_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_1over5040 = 1.0 / 5040.0;
            encoded_constant_1over5040_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1over5040, encoded_constant_1over5040_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());
            boot_context_generated_ = true;
        }
        else
        {
            throw std::runtime_error("Bootstrapping parameters is locked after "
                                     "generation and cannot be modified.");
        }

        hipDeviceSynchronize();
    }

    __host__ Ciphertext<Scheme::CKKS>
    HEArithmeticOperator<Scheme::CKKS>::regular_bootstrapping(
        Ciphertext<Scheme::CKKS>& input1, Galoiskey<Scheme::CKKS>& galois_key,
        Relinkey<Scheme::CKKS>& relin_key, const ExecutionOptions& options)
    {
        if (!boot_context_generated_)
        {
            throw std::invalid_argument(
                "Bootstrapping operation can not be performed before "
                "generating Bootstrapping parameters!");
        }

        // Raise modulus
        int current_decomp_count = Q_size_ - input1.depth_;
        if (current_decomp_count != 1)
        {
            throw std::logic_error("Ciphertexts leveled should be at max!");
        }

        ExecutionOptions options_inner =
            ExecutionOptions()
                .set_stream(options.stream_)
                .set_storage_type(storage_type::DEVICE)
                .set_initial_location(true);

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = options.stream_};

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = options.stream_};

        DeviceVector<Data64> input_intt_poly(2 * n, options.stream_);
        input_storage_manager(
            input1,
            [&](Ciphertext<Scheme::CKKS>& input1_)
            {
                gpuntt::GPU_NTT(input1.data(), input_intt_poly.data(),
                                intt_table_->data(), modulus_->data(), cfg_intt,
                                2, 1);
            },
            options, false);

        Ciphertext<Scheme::CKKS> c_raised =
            operator_ciphertext(scale_boot_, options_inner.stream_);
        mod_raise_kernel<<<dim3((n >> 8), Q_size_, 2), 256, 0,
                           options_inner.stream_>>>(
            input_intt_poly.data(), c_raised.data(), modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(c_raised.data(), ntt_table_->data(),
                                modulus_->data(), cfg_ntt, 2 * Q_size_,
                                Q_size_);

        // Coeff to slot
        std::vector<heongpu::Ciphertext<Scheme::CKKS>> enc_results =
            coeff_to_slot(c_raised, galois_key, options_inner); // c_raised

        // Exponentiate
        Ciphertext<Scheme::CKKS> ciph_neg_exp0 =
            operator_ciphertext(0, options_inner.stream_);
        Ciphertext<Scheme::CKKS> ciph_exp0 =
            exp_scaled(enc_results[0], relin_key, options_inner);

        Ciphertext<Scheme::CKKS> ciph_neg_exp1 =
            operator_ciphertext(0, options_inner.stream_);
        Ciphertext<Scheme::CKKS> ciph_exp1 =
            exp_scaled(enc_results[1], relin_key, options_inner);

        // Compute sine
        Ciphertext<Scheme::CKKS> ciph_sin0 =
            operator_ciphertext(0, options_inner.stream_);
        conjugate(ciph_exp0, ciph_neg_exp0, galois_key,
                  options_inner); // conjugate
        sub(ciph_exp0, ciph_neg_exp0, ciph_sin0, options_inner);

        Ciphertext<Scheme::CKKS> ciph_sin1 =
            operator_ciphertext(0, options_inner.stream_);
        conjugate(ciph_exp1, ciph_neg_exp1, galois_key,
                  options_inner); // conjugate
        sub(ciph_exp1, ciph_neg_exp1, ciph_sin1, options_inner);

        // Scale
        current_decomp_count = Q_size_ - ciph_sin0.depth_;
        cipherplain_multiplication_kernel<<<dim3((n >> 8), current_decomp_count,
                                                 2),
                                            256, 0, options_inner.stream_>>>(
            ciph_sin0.data(), encoded_complex_minus_iscale_.data(),
            ciph_sin0.data(), modulus_->data(), n_power);
        ciph_sin0.scale_ = ciph_sin0.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        ciph_sin0.rescale_required_ = true;
        rescale_inplace(ciph_sin0, options_inner);

        current_decomp_count = Q_size_ - ciph_sin1.depth_;
        cipherplain_multiplication_kernel<<<dim3((n >> 8), current_decomp_count,
                                                 2),
                                            256, 0, options_inner.stream_>>>(
            ciph_sin1.data(), encoded_complex_minus_iscale_.data(),
            ciph_sin1.data(), modulus_->data(), n_power);
        ciph_sin1.scale_ = ciph_sin1.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        ciph_sin1.rescale_required_ = true;
        rescale_inplace(ciph_sin1, options_inner);

        // Slot to coeff
        Ciphertext<Scheme::CKKS> StoC_results =
            slot_to_coeff(ciph_sin0, ciph_sin1, galois_key, options_inner);
        StoC_results.scale_ = scale_boot_;

        return StoC_results;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HEArithmeticOperator<Scheme::CKKS>::slim_bootstrapping(
        Ciphertext<Scheme::CKKS>& input1, Galoiskey<Scheme::CKKS>& galois_key,
        Relinkey<Scheme::CKKS>& relin_key, const ExecutionOptions& options)
    {
        if (!boot_context_generated_)
        {
            throw std::invalid_argument(
                "Bootstrapping operation can not be performed before "
                "generating Bootstrapping parameters!");
        }

        // Raise modulus
        int current_decomp_count = Q_size_ - input1.depth_;
        if (current_decomp_count != (1 + StoC_piece_))
        {
            throw std::logic_error("Ciphertexts leveled should be at max!");
        }

        ExecutionOptions options_inner =
            ExecutionOptions()
                .set_stream(options.stream_)
                .set_storage_type(storage_type::DEVICE)
                .set_initial_location(true);

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = options.stream_};

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = options.stream_};

        // Slot to coeff
        Ciphertext<Scheme::CKKS> StoC_results =
            solo_slot_to_coeff(input1, galois_key, options_inner);

        DeviceVector<Data64> input_intt_poly(2 * n, options.stream_);
        input_storage_manager(
            StoC_results,
            [&](Ciphertext<Scheme::CKKS>& StoC_results_)
            {
                gpuntt::GPU_NTT(StoC_results.data(), input_intt_poly.data(),
                                intt_table_->data(), modulus_->data(), cfg_intt,
                                2, 1);
            },
            options, false);

        Ciphertext<Scheme::CKKS> c_raised =
            operator_ciphertext(scale_boot_, options_inner.stream_);
        mod_raise_kernel<<<dim3((n >> 8), Q_size_, 2), 256, 0,
                           options_inner.stream_>>>(
            input_intt_poly.data(), c_raised.data(), modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(c_raised.data(), ntt_table_->data(),
                                modulus_->data(), cfg_ntt, 2 * Q_size_,
                                Q_size_);

        // Coeff to slot
        Ciphertext<Scheme::CKKS> CtoS_results =
            solo_coeff_to_slot(c_raised, galois_key, options_inner);

        // Exponentiate
        Ciphertext<Scheme::CKKS> ciph_neg_exp =
            operator_ciphertext(0, options_inner.stream_);
        Ciphertext<Scheme::CKKS> ciph_exp =
            exp_scaled(CtoS_results, relin_key, options_inner);

        // Compute sine
        Ciphertext<Scheme::CKKS> ciph_sin =
            operator_ciphertext(0, options_inner.stream_);
        conjugate(ciph_exp, ciph_neg_exp, galois_key,
                  options_inner); // conjugate
        sub(ciph_exp, ciph_neg_exp, ciph_sin, options_inner);

        // Scale
        current_decomp_count = Q_size_ - ciph_sin.depth_;
        cipherplain_multiplication_kernel<<<dim3((n >> 8), current_decomp_count,
                                                 2),
                                            256, 0, options_inner.stream_>>>(
            ciph_sin.data(), encoded_complex_minus_iscale_.data(),
            ciph_sin.data(), modulus_->data(), n_power);
        ciph_sin.scale_ = ciph_sin.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        ciph_sin.rescale_required_ = true;
        rescale_inplace(ciph_sin, options_inner);
        ciph_sin.scale_ = scale_boot_;

        return ciph_sin;
    }

    HELogicOperator<Scheme::CKKS>::HELogicOperator(
        HEContext<Scheme::CKKS>& context, HEEncoder<Scheme::CKKS>& encoder,
        double scale)
        : HEOperator<Scheme::CKKS>(context, encoder)
    {
        if (scale == 0.0)
        {
            throw std::invalid_argument(
                "Scale can not be zero for CKKS Scheme");
        }

        double constant_1 = 1.0;
        encoded_constant_one_ = DeviceVector<Data64>(Q_size_ << n_power);
        quick_ckks_encoder_constant_double(constant_1,
                                           encoded_constant_one_.data(), scale);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    __host__ void HELogicOperator<Scheme::CKKS>::generate_bootstrapping_params(
        const double scale, const BootstrappingConfig& config,
        const logic_bootstrapping_type& boot_type)
    {
        if (!boot_context_generated_)
        {
            int division = static_cast<int>(
                round(static_cast<double>(prime_vector_[0].value) / scale));

            switch (static_cast<int>(boot_type))
            {
                case 1: // BIT_BOOTSTRAPPING
                    if ((division != 2))
                    {
                        throw std::invalid_argument(
                            "Bootstrapping parameters can not be generated, "
                            "because of context is not suitable for Bit "
                            "Bootstrapping. Last modulus should be 2*scale!");
                    }
                    break;
                case 2: // GATE_BOOTSTRAPPING
                    if ((division != 3))
                    {
                        throw std::invalid_argument(
                            "Bootstrapping parameters can not be generated, "
                            "because of context is not suitable for Gate "
                            "Bootstrapping. Last modulus should be 3*scale!");
                    }

                    break;
                default:
                    throw std::invalid_argument("Invalid Key Switching Type");
                    break;
            }

            /*
            enum class logic_bootstrapping_type : std::uint8_t
            {
                NONE = 0x0,
                BIT_BOOTSTRAPPING = 0x1, // scale = q0 / 2. More detail:
                                        // https://eprint.iacr.org/2024/767.pdf
                GATE_BOOTSTRAPPING = 0x2, // scale = q0 / 3. More detail:
                                        // https://eprint.iacr.org/2024/767.pdf
            };
            */

            scale_boot_ = scale;
            CtoS_piece_ = config.CtoS_piece_;
            StoC_piece_ = config.StoC_piece_;
            taylor_number_ = config.taylor_number_;
            less_key_mode_ = config.less_key_mode_;

            // TODO: remove it!
            bool use_all_bases = false; // Do not change it!

            Vandermonde matrix_gen(n, CtoS_piece_, StoC_piece_, less_key_mode_);

            V_matrixs_rotated_encoded_ =
                encode_V_matrixs(matrix_gen, scale_boot_, use_all_bases);
            V_inv_matrixs_rotated_encoded_ =
                encode_V_inv_matrixs(matrix_gen, scale_boot_, use_all_bases);

            V_matrixs_index_ = matrix_gen.V_matrixs_index_;
            V_inv_matrixs_index_ = matrix_gen.V_inv_matrixs_index_;

            diags_matrices_bsgs_ = matrix_gen.diags_matrices_bsgs_;
            diags_matrices_inv_bsgs_ = matrix_gen.diags_matrices_inv_bsgs_;

            if (less_key_mode_)
            {
                real_shift_n2_bsgs_ = matrix_gen.real_shift_n2_bsgs_;
                real_shift_n2_inv_bsgs_ = matrix_gen.real_shift_n2_inv_bsgs_;
            }

            key_indexs_ = matrix_gen.key_indexs_;

            // Pre-computed encoded parameters
            // CtoS
            double constant_1over2 = 0.5;
            encoded_constant_1over2_ = DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1over2, encoded_constant_1over2_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            Complex64 complex_minus_iover2(0.0, -0.5);
            encoded_complex_minus_iover2_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_complex(
                complex_minus_iover2, encoded_complex_minus_iover2_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            // StoC
            Complex64 complex_i(0.0, 1.0);
            encoded_complex_i_ = DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_complex(
                complex_i, encoded_complex_i_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            // Scale part
            Complex64 complex_minus_iscale(
                0.0, -(((static_cast<double>(prime_vector_[0].value) * 0.25) /
                        (scale_boot_ * M_PI))));
            encoded_complex_minus_iscale_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_complex(
                complex_minus_iscale, encoded_complex_minus_iscale_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            // Exponentiate
            Complex64 complex_iscaleoverr(
                0.0, (((2 * M_PI * scale_boot_) /
                       static_cast<double>(prime_vector_[0].value))) /
                         static_cast<double>(1 << taylor_number_));
            encoded_complex_iscaleoverr_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_complex(
                complex_iscaleoverr, encoded_complex_iscaleoverr_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            // Sinus taylor
            double constant_1 = 1.0;
            encoded_constant_1_ = DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1, encoded_constant_1_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_1over6 = 1.0 / 6.0;
            encoded_constant_1over6_ = DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1over6, encoded_constant_1over6_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_1over24 = 1.0 / 24.0;
            encoded_constant_1over24_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(constant_1over24,
                                               encoded_constant_1over24_.data(),
                                               scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_1over120 = 1.0 / 120.0;
            encoded_constant_1over120_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1over120, encoded_constant_1over120_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_1over720 = 1.0 / 720.0;
            encoded_constant_1over720_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1over720, encoded_constant_1over720_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_1over5040 = 1.0 / 5040.0;
            encoded_constant_1over5040_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1over5040, encoded_constant_1over5040_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            // Bit bootstrapping
            double constant_minus_1over4 = -0.25;
            encoded_constant_minus_1over4_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_minus_1over4, encoded_constant_minus_1over4_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            // Gate bootstrapping
            double constant_1over3_ = 1.0 / 3.0;
            encoded_constant_1over3_ = DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_1over3_, encoded_constant_1over3_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_2over3_ = 2.0 / 3.0;
            encoded_constant_2over3_ = DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_2over3_, encoded_constant_2over3_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            Complex64 complex_minus_2over6j_(0.0, (1.0 / 3.0));
            encoded_complex_minus_2over6j_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_complex(
                complex_minus_2over6j_, encoded_complex_minus_2over6j_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_minus_2over6_ = -(1.0 / 3.0);
            encoded_constant_minus_2over6_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_minus_2over6_, encoded_constant_minus_2over6_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            Complex64 complex_2over6j_(0.0, (-1.0 / 3.0));
            encoded_complex_2over6j_ = DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_complex(
                complex_2over6j_, encoded_complex_2over6j_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_2over6_ = 1.0 / 3.0;
            encoded_constant_2over6_ = DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_2over6_, encoded_constant_2over6_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_pioversome_ =
                prime_vector_[0].value / (12.0 * scale_boot_);
            encoded_constant_pioversome_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_pioversome_, encoded_constant_pioversome_.data(),
                scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            double constant_minus_pioversome_ =
                -((prime_vector_[0].value) / (12.0 * scale_boot_));
            encoded_constant_minus_pioversome_ =
                DeviceVector<Data64>(Q_size_ << n_power);
            quick_ckks_encoder_constant_double(
                constant_minus_pioversome_,
                encoded_constant_minus_pioversome_.data(), scale_boot_);
            HEONGPU_CUDA_CHECK(hipGetLastError());

            boot_context_generated_ = true;
        }
        else
        {
            throw std::runtime_error("Bootstrapping parameters is locked after "
                                     "generation and cannot be modified.");
        }

        hipDeviceSynchronize();
    }

    __host__ Ciphertext<Scheme::CKKS>
    HELogicOperator<Scheme::CKKS>::bit_bootstrapping(
        Ciphertext<Scheme::CKKS>& input1, Galoiskey<Scheme::CKKS>& galois_key,
        Relinkey<Scheme::CKKS>& relin_key, const ExecutionOptions& options)
    {
        if (!boot_context_generated_)
        {
            throw std::invalid_argument(
                "Bootstrapping operation can not be performed before "
                "generating Bootstrapping parameters!");
        }

        // Raise modulus
        int current_decomp_count = Q_size_ - input1.depth_;
        if (current_decomp_count != (1 + StoC_piece_))
        {
            throw std::logic_error("Ciphertexts leveled should be at max!");
        }

        ExecutionOptions options_inner =
            ExecutionOptions()
                .set_stream(options.stream_)
                .set_storage_type(storage_type::DEVICE)
                .set_initial_location(true);

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = options.stream_};

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = options.stream_};

        // Slot to coeff
        Ciphertext<Scheme::CKKS> StoC_results =
            solo_slot_to_coeff(input1, galois_key, options_inner);

        DeviceVector<Data64> input_intt_poly(2 * n, options.stream_);
        input_storage_manager(
            StoC_results,
            [&](Ciphertext<Scheme::CKKS>& StoC_results_)
            {
                gpuntt::GPU_NTT(StoC_results.data(), input_intt_poly.data(),
                                intt_table_->data(), modulus_->data(), cfg_intt,
                                2, 1);
            },
            options, false);

        Ciphertext<Scheme::CKKS> c_raised =
            operator_ciphertext(scale_boot_, options_inner.stream_);
        mod_raise_kernel<<<dim3((n >> 8), Q_size_, 2), 256, 0,
                           options_inner.stream_>>>(
            input_intt_poly.data(), c_raised.data(), modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(c_raised.data(), ntt_table_->data(),
                                modulus_->data(), cfg_ntt, 2 * Q_size_,
                                Q_size_);

        // Coeff to slot
        Ciphertext<Scheme::CKKS> CtoS_results =
            solo_coeff_to_slot(c_raised, galois_key, options_inner);

        // Exponentiate
        Ciphertext<Scheme::CKKS> ciph_neg_exp =
            operator_ciphertext(0, options_inner.stream_);
        Ciphertext<Scheme::CKKS> ciph_exp =
            exp_scaled(CtoS_results, relin_key, options_inner);

        // Compute cosine
        Ciphertext<Scheme::CKKS> ciph_cos =
            operator_ciphertext(0, options_inner.stream_);
        conjugate(ciph_exp, ciph_neg_exp, galois_key,
                  options_inner); // conjugate
        add(ciph_exp, ciph_neg_exp, ciph_cos, options_inner);

        // Scale
        current_decomp_count = Q_size_ - ciph_cos.depth_;
        cipherplain_multiplication_kernel<<<dim3((n >> 8), current_decomp_count,
                                                 2),
                                            256, 0, options_inner.stream_>>>(
            ciph_cos.data(), encoded_constant_minus_1over4_.data(),
            ciph_cos.data(), modulus_->data(), n_power);
        ciph_cos.scale_ = ciph_cos.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        ciph_cos.rescale_required_ = true;
        rescale_inplace(ciph_cos, options_inner);

        //

        Ciphertext<Scheme::CKKS> result =
            operator_ciphertext(0, options_inner.stream_);
        current_decomp_count = Q_size_ - ciph_cos.depth_;
        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options_inner.stream_>>>(
            ciph_cos.data(), encoded_constant_1over2_.data(), result.data(),
            modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        result.scheme_ = scheme_;
        result.ring_size_ = n;
        result.coeff_modulus_count_ = Q_size_;
        result.cipher_size_ = 2;
        result.depth_ = ciph_cos.depth_;
        result.scale_ = scale_boot_;
        result.in_ntt_domain_ = ciph_cos.in_ntt_domain_;
        result.rescale_required_ = ciph_cos.rescale_required_;
        result.relinearization_required_ = ciph_cos.relinearization_required_;
        result.ciphertext_generated_ = true;

        //

        return result;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HELogicOperator<Scheme::CKKS>::gate_bootstrapping(
        logic_gate gate_type, Ciphertext<Scheme::CKKS>& input1,
        Ciphertext<Scheme::CKKS>& input2, Galoiskey<Scheme::CKKS>& galois_key,
        Relinkey<Scheme::CKKS>& relin_key, const ExecutionOptions& options)
    {
        if (!boot_context_generated_)
        {
            throw std::invalid_argument(
                "Bootstrapping operation can not be performed before "
                "generating Bootstrapping parameters!");
        }

        // Raise modulus
        int current_decomp_count = Q_size_ - input1.depth_;
        if (current_decomp_count != (1 + StoC_piece_))
        {
            throw std::logic_error("Ciphertexts leveled should be at max!");
        }

        current_decomp_count = Q_size_ - input2.depth_;
        if (current_decomp_count != (1 + StoC_piece_))
        {
            throw std::logic_error("Ciphertexts leveled should be at max!");
        }

        ExecutionOptions options_inner =
            ExecutionOptions()
                .set_stream(options.stream_)
                .set_storage_type(storage_type::DEVICE)
                .set_initial_location(true);

        gpuntt::ntt_rns_configuration<Data64> cfg_intt = {
            .n_power = n_power,
            .ntt_type = gpuntt::INVERSE,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .mod_inverse = n_inverse_->data(),
            .stream = options.stream_};

        gpuntt::ntt_rns_configuration<Data64> cfg_ntt = {
            .n_power = n_power,
            .ntt_type = gpuntt::FORWARD,
            .reduction_poly = gpuntt::ReductionPolynomial::X_N_plus,
            .zero_padding = false,
            .stream = options.stream_};

        Ciphertext<Scheme::CKKS> input_ =
            operator_ciphertext(0, options_inner.stream_);
        add(input1, input2, input_);

        // Slot to coeff
        Ciphertext<Scheme::CKKS> StoC_results =
            solo_slot_to_coeff(input_, galois_key, options_inner);

        DeviceVector<Data64> input_intt_poly(2 * n, options.stream_);
        input_storage_manager(
            StoC_results,
            [&](Ciphertext<Scheme::CKKS>& StoC_results_)
            {
                gpuntt::GPU_NTT(StoC_results.data(), input_intt_poly.data(),
                                intt_table_->data(), modulus_->data(), cfg_intt,
                                2, 1);
            },
            options, false);

        Ciphertext<Scheme::CKKS> c_raised =
            operator_ciphertext(scale_boot_, options_inner.stream_);
        mod_raise_kernel<<<dim3((n >> 8), Q_size_, 2), 256, 0,
                           options_inner.stream_>>>(
            input_intt_poly.data(), c_raised.data(), modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        gpuntt::GPU_NTT_Inplace(c_raised.data(), ntt_table_->data(),
                                modulus_->data(), cfg_ntt, 2 * Q_size_,
                                Q_size_);

        // Coeff to slot
        Ciphertext<Scheme::CKKS> CtoS_results =
            solo_coeff_to_slot(c_raised, galois_key, options_inner);

        Ciphertext<Scheme::CKKS> result =
            operator_ciphertext(0, options_inner.stream_);

        switch (gate_type)
        {
            case logic_gate::AND:
                result = AND_approximation(CtoS_results, galois_key, relin_key,
                                           options_inner);
                break;
            case logic_gate::OR:
                result = OR_approximation(CtoS_results, galois_key, relin_key,
                                          options_inner);
                break;
            case logic_gate::XOR:
                result = XOR_approximation(CtoS_results, galois_key, relin_key,
                                           options_inner);
                break;
            case logic_gate::NAND:
                result = NAND_approximation(CtoS_results, galois_key, relin_key,
                                            options_inner);
                break;
            case logic_gate::NOR:
                result = NOR_approximation(CtoS_results, galois_key, relin_key,
                                           options_inner);
                break;
            case logic_gate::XNOR:
                result = XNOR_approximation(CtoS_results, galois_key, relin_key,
                                            options_inner);
                break;
            default:
                throw std::invalid_argument("Unknown Gate Type!");
        }

        return result;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HELogicOperator<Scheme::CKKS>::AND_approximation(
        Ciphertext<Scheme::CKKS>& cipher, Galoiskey<Scheme::CKKS>& galois_key,
        Relinkey<Scheme::CKKS>& relin_key, const ExecutionOptions& options)
    {
        //////////////////////////////
        // plain add
        Ciphertext<Scheme::CKKS> cipher_add =
            operator_ciphertext(0, options.stream_);
        int current_decomp_count = Q_size_ - cipher.depth_;
        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options.stream_>>>(
            cipher.data(), encoded_constant_pioversome_.data(),
            cipher_add.data(), modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        cipher_add.scheme_ = scheme_;
        cipher_add.ring_size_ = n;
        cipher_add.coeff_modulus_count_ = Q_size_;
        cipher_add.cipher_size_ = 2;
        cipher_add.depth_ = cipher.depth_;
        cipher_add.scale_ = cipher.scale_;
        cipher_add.in_ntt_domain_ = cipher.in_ntt_domain_;
        cipher_add.rescale_required_ = cipher.rescale_required_;
        cipher_add.relinearization_required_ = cipher.relinearization_required_;
        cipher_add.ciphertext_generated_ = true;
        //////////////////////////////

        Ciphertext<Scheme::CKKS> ciph_neg_exp =
            operator_ciphertext(0, options.stream_);
        Ciphertext<Scheme::CKKS> ciph_exp =
            exp_scaled(cipher_add, relin_key, options);

        // Compute sine
        Ciphertext<Scheme::CKKS> ciph_sin =
            operator_ciphertext(0, options.stream_);
        conjugate(ciph_exp, ciph_neg_exp, galois_key,
                  options); // conjugate
        sub(ciph_exp, ciph_neg_exp, ciph_sin, options);

        // Scale
        current_decomp_count = Q_size_ - ciph_sin.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            ciph_sin.data(), encoded_complex_minus_2over6j_.data(),
            ciph_sin.data(), modulus_->data(), n_power);
        ciph_sin.scale_ = ciph_sin.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        ciph_sin.rescale_required_ = true;
        rescale_inplace(ciph_sin, options);

        //////////////////////////////
        // plain add
        Ciphertext<Scheme::CKKS> result =
            operator_ciphertext(0, options.stream_);
        current_decomp_count = Q_size_ - ciph_sin.depth_;
        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options.stream_>>>(
            ciph_sin.data(), encoded_constant_1over3_.data(), result.data(),
            modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        result.scheme_ = scheme_;
        result.ring_size_ = n;
        result.coeff_modulus_count_ = Q_size_;
        result.cipher_size_ = 2;
        result.depth_ = ciph_sin.depth_;
        result.scale_ = scale_boot_;
        result.in_ntt_domain_ = ciph_sin.in_ntt_domain_;
        result.rescale_required_ = ciph_sin.rescale_required_;
        result.relinearization_required_ = ciph_sin.relinearization_required_;
        result.ciphertext_generated_ = true;
        //////////////////////////////

        return result;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HELogicOperator<Scheme::CKKS>::OR_approximation(
        Ciphertext<Scheme::CKKS>& cipher, Galoiskey<Scheme::CKKS>& galois_key,
        Relinkey<Scheme::CKKS>& relin_key, const ExecutionOptions& options)
    {
        Ciphertext<Scheme::CKKS> ciph_neg_exp =
            operator_ciphertext(0, options.stream_);
        Ciphertext<Scheme::CKKS> ciph_exp =
            exp_scaled(cipher, relin_key, options);

        // Compute sine
        Ciphertext<Scheme::CKKS> ciph_sin =
            operator_ciphertext(0, options.stream_);
        conjugate(ciph_exp, ciph_neg_exp, galois_key,
                  options); // conjugate
        add(ciph_exp, ciph_neg_exp, ciph_sin, options);

        // Scale
        int current_decomp_count = Q_size_ - ciph_sin.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            ciph_sin.data(), encoded_constant_minus_2over6_.data(),
            ciph_sin.data(), modulus_->data(), n_power);
        ciph_sin.scale_ = ciph_sin.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        ciph_sin.rescale_required_ = true;
        rescale_inplace(ciph_sin, options);

        //////////////////////////////
        // plain add
        Ciphertext<Scheme::CKKS> result =
            operator_ciphertext(0, options.stream_);
        current_decomp_count = Q_size_ - ciph_sin.depth_;
        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options.stream_>>>(
            ciph_sin.data(), encoded_constant_2over3_.data(), result.data(),
            modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        result.scheme_ = scheme_;
        result.ring_size_ = n;
        result.coeff_modulus_count_ = Q_size_;
        result.cipher_size_ = 2;
        result.depth_ = ciph_sin.depth_;
        result.scale_ = scale_boot_;
        result.in_ntt_domain_ = ciph_sin.in_ntt_domain_;
        result.rescale_required_ = ciph_sin.rescale_required_;
        result.relinearization_required_ = ciph_sin.relinearization_required_;
        result.ciphertext_generated_ = true;
        //////////////////////////////

        return result;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HELogicOperator<Scheme::CKKS>::XOR_approximation(
        Ciphertext<Scheme::CKKS>& cipher, Galoiskey<Scheme::CKKS>& galois_key,
        Relinkey<Scheme::CKKS>& relin_key, const ExecutionOptions& options)
    {
        //////////////////////////////
        // plain add
        Ciphertext<Scheme::CKKS> cipher_add =
            operator_ciphertext(0, options.stream_);
        int current_decomp_count = Q_size_ - cipher.depth_;
        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options.stream_>>>(
            cipher.data(), encoded_constant_minus_pioversome_.data(),
            cipher_add.data(), modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        cipher_add.scheme_ = scheme_;
        cipher_add.ring_size_ = n;
        cipher_add.coeff_modulus_count_ = Q_size_;
        cipher_add.cipher_size_ = 2;
        cipher_add.depth_ = cipher.depth_;
        cipher_add.scale_ = cipher.scale_;
        cipher_add.in_ntt_domain_ = cipher.in_ntt_domain_;
        cipher_add.rescale_required_ = cipher.rescale_required_;
        cipher_add.relinearization_required_ = cipher.relinearization_required_;
        cipher_add.ciphertext_generated_ = true;
        //////////////////////////////

        Ciphertext<Scheme::CKKS> ciph_neg_exp =
            operator_ciphertext(0, options.stream_);
        Ciphertext<Scheme::CKKS> ciph_exp =
            exp_scaled(cipher_add, relin_key, options);

        // Compute sine
        Ciphertext<Scheme::CKKS> ciph_sin =
            operator_ciphertext(0, options.stream_);
        conjugate(ciph_exp, ciph_neg_exp, galois_key,
                  options); // conjugate
        sub(ciph_exp, ciph_neg_exp, ciph_sin, options);

        // Scale
        current_decomp_count = Q_size_ - ciph_sin.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            ciph_sin.data(), encoded_complex_2over6j_.data(), ciph_sin.data(),
            modulus_->data(), n_power);
        ciph_sin.scale_ = ciph_sin.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        ciph_sin.rescale_required_ = true;
        rescale_inplace(ciph_sin, options);

        //////////////////////////////
        // plain add
        Ciphertext<Scheme::CKKS> result =
            operator_ciphertext(0, options.stream_);
        current_decomp_count = Q_size_ - ciph_sin.depth_;
        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options.stream_>>>(
            ciph_sin.data(), encoded_constant_1over3_.data(), result.data(),
            modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        result.scheme_ = scheme_;
        result.ring_size_ = n;
        result.coeff_modulus_count_ = Q_size_;
        result.cipher_size_ = 2;
        result.depth_ = ciph_sin.depth_;
        result.scale_ = scale_boot_;
        result.in_ntt_domain_ = ciph_sin.in_ntt_domain_;
        result.rescale_required_ = ciph_sin.rescale_required_;
        result.relinearization_required_ = ciph_sin.relinearization_required_;
        result.ciphertext_generated_ = true;
        //////////////////////////////

        return result;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HELogicOperator<Scheme::CKKS>::NAND_approximation(
        Ciphertext<Scheme::CKKS>& cipher, Galoiskey<Scheme::CKKS>& galois_key,
        Relinkey<Scheme::CKKS>& relin_key, const ExecutionOptions& options)
    {
        //////////////////////////////
        // plain add
        Ciphertext<Scheme::CKKS> cipher_add =
            operator_ciphertext(0, options.stream_);
        int current_decomp_count = Q_size_ - cipher.depth_;
        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options.stream_>>>(
            cipher.data(), encoded_constant_pioversome_.data(),
            cipher_add.data(), modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        cipher_add.scheme_ = scheme_;
        cipher_add.ring_size_ = n;
        cipher_add.coeff_modulus_count_ = Q_size_;
        cipher_add.cipher_size_ = 2;
        cipher_add.depth_ = cipher.depth_;
        cipher_add.scale_ = cipher.scale_;
        cipher_add.in_ntt_domain_ = cipher.in_ntt_domain_;
        cipher_add.rescale_required_ = cipher.rescale_required_;
        cipher_add.relinearization_required_ = cipher.relinearization_required_;
        cipher_add.ciphertext_generated_ = true;
        //////////////////////////////

        Ciphertext<Scheme::CKKS> ciph_neg_exp =
            operator_ciphertext(0, options.stream_);
        Ciphertext<Scheme::CKKS> ciph_exp =
            exp_scaled(cipher_add, relin_key, options);

        // Compute sine
        Ciphertext<Scheme::CKKS> ciph_sin =
            operator_ciphertext(0, options.stream_);
        conjugate(ciph_exp, ciph_neg_exp, galois_key,
                  options); // conjugate
        sub(ciph_exp, ciph_neg_exp, ciph_sin, options);

        // Scale
        current_decomp_count = Q_size_ - ciph_sin.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            ciph_sin.data(), encoded_complex_2over6j_.data(), ciph_sin.data(),
            modulus_->data(), n_power);
        ciph_sin.scale_ = ciph_sin.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        ciph_sin.rescale_required_ = true;
        rescale_inplace(ciph_sin, options);

        //////////////////////////////
        // plain add
        Ciphertext<Scheme::CKKS> result =
            operator_ciphertext(0, options.stream_);
        current_decomp_count = Q_size_ - ciph_sin.depth_;
        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options.stream_>>>(
            ciph_sin.data(), encoded_constant_2over3_.data(), result.data(),
            modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        result.scheme_ = scheme_;
        result.ring_size_ = n;
        result.coeff_modulus_count_ = Q_size_;
        result.cipher_size_ = 2;
        result.depth_ = ciph_sin.depth_;
        result.scale_ = scale_boot_;
        result.in_ntt_domain_ = ciph_sin.in_ntt_domain_;
        result.rescale_required_ = ciph_sin.rescale_required_;
        result.relinearization_required_ = ciph_sin.relinearization_required_;
        result.ciphertext_generated_ = true;
        //////////////////////////////

        return result;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HELogicOperator<Scheme::CKKS>::NOR_approximation(
        Ciphertext<Scheme::CKKS>& cipher, Galoiskey<Scheme::CKKS>& galois_key,
        Relinkey<Scheme::CKKS>& relin_key, const ExecutionOptions& options)
    {
        Ciphertext<Scheme::CKKS> ciph_neg_exp =
            operator_ciphertext(0, options.stream_);
        Ciphertext<Scheme::CKKS> ciph_exp =
            exp_scaled(cipher, relin_key, options);

        // Compute sine
        Ciphertext<Scheme::CKKS> ciph_sin =
            operator_ciphertext(0, options.stream_);
        conjugate(ciph_exp, ciph_neg_exp, galois_key,
                  options); // conjugate
        add(ciph_exp, ciph_neg_exp, ciph_sin, options);

        // Scale
        int current_decomp_count = Q_size_ - ciph_sin.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            ciph_sin.data(), encoded_constant_2over6_.data(), ciph_sin.data(),
            modulus_->data(), n_power);
        ciph_sin.scale_ = ciph_sin.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        ciph_sin.rescale_required_ = true;
        rescale_inplace(ciph_sin, options);

        //////////////////////////////
        // plain add
        Ciphertext<Scheme::CKKS> result =
            operator_ciphertext(0, options.stream_);
        current_decomp_count = Q_size_ - ciph_sin.depth_;
        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options.stream_>>>(
            ciph_sin.data(), encoded_constant_1over3_.data(), result.data(),
            modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        result.scheme_ = scheme_;
        result.ring_size_ = n;
        result.coeff_modulus_count_ = Q_size_;
        result.cipher_size_ = 2;
        result.depth_ = ciph_sin.depth_;
        result.scale_ = scale_boot_;
        result.in_ntt_domain_ = ciph_sin.in_ntt_domain_;
        result.rescale_required_ = ciph_sin.rescale_required_;
        result.relinearization_required_ = ciph_sin.relinearization_required_;
        result.ciphertext_generated_ = true;
        //////////////////////////////

        return result;
    }

    __host__ Ciphertext<Scheme::CKKS>
    HELogicOperator<Scheme::CKKS>::XNOR_approximation(
        Ciphertext<Scheme::CKKS>& cipher, Galoiskey<Scheme::CKKS>& galois_key,
        Relinkey<Scheme::CKKS>& relin_key, const ExecutionOptions& options)
    {
        //////////////////////////////
        // plain add
        Ciphertext<Scheme::CKKS> cipher_add =
            operator_ciphertext(0, options.stream_);
        int current_decomp_count = Q_size_ - cipher.depth_;
        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options.stream_>>>(
            cipher.data(), encoded_constant_minus_pioversome_.data(),
            cipher_add.data(), modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        cipher_add.scheme_ = scheme_;
        cipher_add.ring_size_ = n;
        cipher_add.coeff_modulus_count_ = Q_size_;
        cipher_add.cipher_size_ = 2;
        cipher_add.depth_ = cipher.depth_;
        cipher_add.scale_ = cipher.scale_;
        cipher_add.in_ntt_domain_ = cipher.in_ntt_domain_;
        cipher_add.rescale_required_ = cipher.rescale_required_;
        cipher_add.relinearization_required_ = cipher.relinearization_required_;
        cipher_add.ciphertext_generated_ = true;
        //////////////////////////////

        Ciphertext<Scheme::CKKS> ciph_neg_exp =
            operator_ciphertext(0, options.stream_);
        Ciphertext<Scheme::CKKS> ciph_exp =
            exp_scaled(cipher_add, relin_key, options);

        // Compute sine
        Ciphertext<Scheme::CKKS> ciph_sin =
            operator_ciphertext(0, options.stream_);
        conjugate(ciph_exp, ciph_neg_exp, galois_key,
                  options); // conjugate
        sub(ciph_exp, ciph_neg_exp, ciph_sin, options);

        // Scale
        current_decomp_count = Q_size_ - ciph_sin.depth_;
        cipherplain_multiplication_kernel<<<
            dim3((n >> 8), current_decomp_count, 2), 256, 0, options.stream_>>>(
            ciph_sin.data(), encoded_complex_minus_2over6j_.data(),
            ciph_sin.data(), modulus_->data(), n_power);
        ciph_sin.scale_ = ciph_sin.scale_ * scale_boot_;
        HEONGPU_CUDA_CHECK(hipGetLastError());
        ciph_sin.rescale_required_ = true;
        rescale_inplace(ciph_sin, options);

        //////////////////////////////
        // plain add
        Ciphertext<Scheme::CKKS> result =
            operator_ciphertext(0, options.stream_);
        current_decomp_count = Q_size_ - ciph_sin.depth_;
        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options.stream_>>>(
            ciph_sin.data(), encoded_constant_2over3_.data(), result.data(),
            modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());

        result.scheme_ = scheme_;
        result.ring_size_ = n;
        result.coeff_modulus_count_ = Q_size_;
        result.cipher_size_ = 2;
        result.depth_ = ciph_sin.depth_;
        result.scale_ = scale_boot_;
        result.in_ntt_domain_ = ciph_sin.in_ntt_domain_;
        result.rescale_required_ = ciph_sin.rescale_required_;
        result.relinearization_required_ = ciph_sin.relinearization_required_;
        result.ciphertext_generated_ = true;
        //////////////////////////////

        return result;
    }

    __host__ void HELogicOperator<Scheme::CKKS>::one_minus_cipher(
        Ciphertext<Scheme::CKKS>& input1, Ciphertext<Scheme::CKKS>& output,
        const ExecutionOptions& options)
    {
        // TODO: make it efficient
        negate_inplace(input1, options);

        int current_decomp_count = Q_size_ - input1.depth_;

        addition_plain_ckks_poly<<<dim3((n >> 8), current_decomp_count, 2), 256,
                                   0, options.stream_>>>(
            input1.data(), encoded_constant_one_.data(), output.data(),
            modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

    __host__ void HELogicOperator<Scheme::CKKS>::one_minus_cipher_inplace(
        Ciphertext<Scheme::CKKS>& input1, const ExecutionOptions& options)
    {
        // TODO: make it efficient
        negate_inplace(input1, options);

        int current_decomp_count = Q_size_ - input1.depth_;

        addition<<<dim3((n >> 8), current_decomp_count, 1), 256, 0,
                   options.stream_>>>(input1.data(),
                                      encoded_constant_one_.data(),
                                      input1.data(), modulus_->data(), n_power);
        HEONGPU_CUDA_CHECK(hipGetLastError());
    }

} // namespace heongpu
