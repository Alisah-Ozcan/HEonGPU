#include "hip/hip_runtime.h"
// Copyright 2024 Alişah Özcan
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0
// Developer: Alişah Özcan

#include "heongpu.cuh"

#include <string>
#include <iomanip>
#include <omp.h>

int main(int argc, char* argv[])
{
    hipSetDevice(0);

    std::vector<size_t> poly_modulus_degrees = {4096, 8192, 16384, 32768};
    std::vector<std::vector<int>> log_Q_bit_sizes = {
        {40, 30, 30},
        {40, 35, 35, 35, 35},
        {50, 40, 40, 40, 40, 40, 40, 40, 40},
        {60, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45, 45}};
    std::vector<std::vector<int>> log_P_bit_sizes = {{40}, {40}, {50}, {60}};
    std::vector<double> scales = {pow(2.0, 30), pow(2.0, 35), pow(2.0, 40),
                                  pow(2.0, 45)};

    // Require High Amount GPU memory
    // std::vector<size_t> poly_modulus_degrees = {65536};
    // std::vector<std::vector<int>> log_Q_bit_sizes =
    // {{60,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50,50}};
    // std::vector<std::vector<int>> log_P_bit_sizes = {{60}};
    // std::vector<double> scales = {pow(2.0, 50)};

    int repeat_count = 10;

    for (int i = 0; i < poly_modulus_degrees.size(); i++)
    {
        heongpu::Parameters context(
            heongpu::scheme_type::ckks,
            heongpu::keyswitching_type::KEYSWITCHING_METHOD_I,
            heongpu::sec_level_type::none);
        context.set_poly_modulus_degree(poly_modulus_degrees[i]);
        context.set_coeff_modulus(log_Q_bit_sizes[i], log_P_bit_sizes[i]);
        context.generate();

        heongpu::HEKeyGenerator keygen(context);
        heongpu::Secretkey secret_key(context);
        keygen.generate_secret_key(secret_key);

        heongpu::Publickey public_key(context);
        keygen.generate_public_key(public_key, secret_key);

        heongpu::Relinkey relin_key(context);
        keygen.generate_relin_key(relin_key, secret_key);

        std::vector<int> custom_key_index = {1};
        heongpu::Galoiskey galois_key(context, custom_key_index);
        keygen.generate_galois_key(galois_key, secret_key);

        heongpu::HEEncoder encoder(context);
        heongpu::HEEncryptor encryptor(context, public_key);
        heongpu::HEDecryptor decryptor(context, secret_key);
        heongpu::HEOperator operators(context);

        const int row_size = poly_modulus_degrees[i] / 2;
        heongpu::HostVector<double> message(row_size, 1);

        float time = 0;
        float time_encode = 0;
        float time_encryption = 0;
        float time_addition = 0;
        float time_subtraction = 0;
        float time_multiplication = 0;
        float time_relinearization = 0;
        float time_plainaddition = 0;
        float time_plainsubtraction = 0;
        float time_plainmultiplication = 0;
        float time_rotaterow = 0;
        float time_rescale = 0;
        float time_decryption = 0;
        float time_decode = 0;

        hipEvent_t start_time, stop_time;
        hipEventCreate(&start_time);
        hipEventCreate(&stop_time);

        for (int j = 0; j < repeat_count; j++)
        {
            double scale = scales[i];
            heongpu::Plaintext P1(context);

            hipEventRecord(start_time);
            encoder.encode(P1, message, scale);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_encode += time;

            heongpu::Ciphertext C1(context);

            hipEventRecord(start_time);
            encryptor.encrypt(C1, P1);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_encryption += time;

            heongpu::Ciphertext C2(context);

            hipEventRecord(start_time);
            operators.add(C1, C1, C2);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_addition += time;

            hipEventRecord(start_time);
            operators.sub(C2, C1, C2);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_subtraction += time;

            hipEventRecord(start_time);
            operators.multiply(C2, C1, C2);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_multiplication += time;

            hipEventRecord(start_time);
            operators.relinearize_inplace(C2, relin_key);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_relinearization += time;

            hipEventRecord(start_time);
            operators.rescale_inplace(C2);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_rescale += time;

            heongpu::Ciphertext C3(context);
            encryptor.encrypt(C3, P1);

            hipEventRecord(start_time);
            operators.rotate_rows(C3, C3, galois_key, 1);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_rotaterow += time;

            hipEventRecord(start_time);
            operators.add_plain_inplace(C3, P1);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_plainaddition += time;

            hipEventRecord(start_time);
            operators.sub_plain_inplace(C3, P1);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_plainsubtraction += time;

            heongpu::Ciphertext C4(context);
            encryptor.encrypt(C4, P1);

            hipEventRecord(start_time);
            operators.multiply_plain(C4, P1, C4);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_plainmultiplication += time;

            heongpu::Plaintext P2(context);

            hipEventRecord(start_time);
            decryptor.decrypt(P2, C3);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_decryption += time;

            heongpu::HostVector<double> message2;
            hipEventRecord(start_time);
            encoder.decode(message2, P2);
            hipEventRecord(stop_time);
            hipEventSynchronize(stop_time);
            hipEventElapsedTime(&time, start_time, stop_time);
            time_decode += time;

            hipDeviceSynchronize();
        }

        std::cout
            << "=================== Benchmark BFV with poly_modulus_degrees: "
            << poly_modulus_degrees[i] << " ===================" << std::endl;
        std::cout << "Average encode timing: " << (time_encode / repeat_count)
                  << " us" << std::endl;
        std::cout << "Average encryption timing: "
                  << (time_encryption / repeat_count) << " us" << std::endl;
        std::cout << "Average addition timing: "
                  << (time_addition / repeat_count) << " us" << std::endl;
        std::cout << "Average subtraction timing: "
                  << (time_subtraction / repeat_count) << " us" << std::endl;
        std::cout << "Average multiplication timing: "
                  << (time_multiplication / repeat_count) << " us" << std::endl;
        std::cout << "Average relinearization timing: "
                  << (time_relinearization / repeat_count) << " us"
                  << std::endl;
        std::cout << "Average plain addition timing: "
                  << (time_plainaddition / repeat_count) << " us" << std::endl;
        std::cout << "Average plain subtraction timing: "
                  << (time_plainsubtraction / repeat_count) << " us"
                  << std::endl;
        std::cout << "Average plain multiplication timing: "
                  << (time_plainmultiplication / repeat_count) << " us"
                  << std::endl;
        std::cout << "Average rotate row timing: "
                  << (time_rotaterow / repeat_count) << " us" << std::endl;
        std::cout << "Average rescale timing: " << (time_rescale / repeat_count)
                  << " us" << std::endl;
        std::cout << "Average decryption timing: "
                  << (time_decryption / repeat_count) << " us" << std::endl;
        std::cout << "Average decode timing: " << (time_decode / repeat_count)
                  << " us" << std::endl;
        std::cout << std::endl << std::endl;
    }

    return EXIT_SUCCESS;
}